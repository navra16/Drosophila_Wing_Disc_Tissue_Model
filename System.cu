#include "hip/hip_runtime.h"
#include <stdio.h>
#include "System.h"
#include "SystemStructures.h"
#include "AreaTriangles.h"
//#include "BendingTriangles.h"
//#include "MemRepulsionSprings_universal.h"
//#include "MemRepulsionSprings_local.h"
//#include "MemRepulsionEnergy.h"
#include "LinearSprings.h"
#include "NodeAdvance.h"
#include "Storage.h"
#include "Utilities.h"
#include "SystemBuilder.h"
#include <vector>
#include "VolumeComp.h"
#include "VolumeSprings.h"
#include <bits/stdc++.h>
#include "LineTensionSprings.h"
#include <math.h>
#include <list>
//#include "TurgorForce.h"
#include "LinearSpringsEnergy.h"
#include "StrainTensor.h"
#include <thrust/iterator/zip_iterator.h>
#include <thrust/functional.h>
#include <thrust/transform_reduce.h>
#include <thrust/tuple.h>
#include "gradientRelax.h"

// somehow the gradient is not being set in my version - Kevin

// Helper function to count elements greater than or equal to zero in a vector.
int count_bigger(const std::vector<int> &elems)
{
    return std::count_if(elems.begin(), elems.end(), [](int c)
                         { return c >= 0; });
}

// Constructor for the System class.
System::System() {};

// Print net force on nodes along a radial line (?  0) from disc center to boundary
void System::PrintForce() {
    // Copy device forces to host
    thrust::host_vector<double> h_fx = coordInfoVecs.nodeForceX;
    thrust::host_vector<double> h_fy = coordInfoVecs.nodeForceY;
    thrust::host_vector<double> h_fz = coordInfoVecs.nodeForceZ;
    thrust::host_vector<double> h_x  = coordInfoVecs.nodeLocX;
    thrust::host_vector<double> h_y  = coordInfoVecs.nodeLocY;

    const double desiredTheta = 0.0;    // along +x axis
    const double eps = 0.01;            // angular tolerance (rad)
    std::vector<std::pair<double,int>> picks;
    int N = static_cast<int>(h_x.size());
    for (int i = 0; i < N; ++i) {
        double r = std::hypot(h_x[i], h_y[i]);
        double theta = std::atan2(h_y[i], h_x[i]);
        double diff = std::fabs(theta - desiredTheta);
        if (diff > M_PI) diff = 2*M_PI - diff;
        if (diff < eps) picks.emplace_back(r, i);
    }
    std::sort(picks.begin(), picks.end());

    std::printf("   r      Fx       Fy       Fz\n");
    for (auto &pr : picks) {
        int idx = pr.second;
        std::printf("%6.3f  %7.3e  %7.3e  %7.3e\n",
                    pr.first,
                    h_fx[idx], h_fy[idx], h_fz[idx]);
    }
}

// Function to solve the forces in the system.
void System::Solve_Forces()
{

    // Reset all forces to zero.
    thrust::fill(coordInfoVecs.nodeForceX.begin(), coordInfoVecs.nodeForceX.end(), 0.0);
    thrust::fill(coordInfoVecs.nodeForceY.begin(), coordInfoVecs.nodeForceY.end(), 0.0);
    thrust::fill(coordInfoVecs.nodeForceZ.begin(), coordInfoVecs.nodeForceZ.end(), 0.0);

    // Compute forces and energy due to linear springs.
    ComputeLinearSprings(
        generalParams,
        coordInfoVecs,
        linearSpringInfoVecs,
        ljInfoVecs);

    // Compute forces and energy due to area springs. Nav commented out to test Active shape programming mesh type 02/27/2025  . Put back in 03/23/25
//      	ComputeAreaTriangleSprings(
//      		generalParams,
//      		coordInfoVecs,
//      		areaTriangleInfoVecs);

    // Compute forces and energy due to turgor pressure springs. (nav - commenting these out for now for flat surface 5/29/24) nav reintroducing the turgor pressure because the eversion wing does have turgor pressure. 8/17/2024
    // ComputeTurgorSprings(
    // generalParams,
    // coordInfoVecs,
    // areaTriangleInfoVecs
    //);

    // Compute forces and energy due to bending springs. Turn this off 10/10/24
//      	ComputeCosTriangleSprings(
//      		generalParams,
//      		coordInfoVecs,
//      		bendingTriangleInfoVecs);

    // Compute forces and energy due to membrane repulsion springs.// Nav commented out to test Active shape programming mesh type 02/27/2025. PUt back in 03/23/25
//      	ComputeMemRepulsionSprings_local(
//      		coordInfoVecs,
//      		linearSpringInfoVecs,
//      		capsidInfoVecs,
//      		generalParams,
//      		auxVecs);

    // Compute forces and energy due to volume springs. //(nav - commenting these out for now for flat surface 5/29/24) Nav had uncommented but she's bringing the comment back because testing out Active shape mesh 02/27/25
//      	ComputeVolume(
//      		generalParams,
//      		coordInfoVecs,
//      		linearSpringInfoVecs,
//      		ljInfoVecs);
    
    // Now print forces along the radial line
   // PrintForce();
};

// Function to solve the entire system.
void System::solveSystem()
{

    // Nav - I dont want to remove these variables. These may come in handy.
    // coordInfoVecs.k_0 = 20.0;
    // coordInfoVecs.k_1 = 25.0;
    // coordInfoVecs.k_2 = 5.0;
    // coordInfoVecs.k_3 = 5.0;
    // coordInfoVecs.k_4 = 1.0;
    // coordInfoVecs.k_ss = 12;//10.75;
    // coordInfoVecs.beta = 1.0/1.0;///1.45;
    // coordInfoVecs.gamma = 1.0;
    // coordInfoVecs.q1 = 10.0;
    // coordInfoVecs.h = 10.0;
    
    generalParams.dt = 0.01;
    
    std::cout<<" Simulating wL3 - 0hAPF "<<std::endl;
    std::cout<<"lambda_iso_center_outDV = -0.12406004 - "<<generalParams.lambda_iso_center_outDV<<std::endl;

    uint mem_prealloc = 4; // Make sure that this number is the same as set in System::initializeSystem found near the bottom of this script. - Kevin. Q. why is this the case? Why does it need to be the same? - Nav.

    // Initial values for determining the region of material insertion.
    double current_edge_to_tip_height_scale = INT_MAX; // 2.0; // The maximum edge to tip height scale, initially set to INT_MAX.
    std::cout << "current_edge_to_tip_height_scale = " << current_edge_to_tip_height_scale << std::endl;

    // Determines how far away from the tip can new material be inserted during edge swap.
    double current_edge_to_tip_height_scale_ES = INT_MAX; // 4.0;//2.0; // Initially set to INT_MAX
    std::cout << "current_edge_to_tip_height_scale for edge-swap = " << current_edge_to_tip_height_scale_ES << std::endl;

    // Determines how far away from the tip can new material be inserted.
    double bdry_to_tip_height_scale = INT_MAX; // 4.0; // Initially set to INT_MAX
    std::cout << "bdry_to_tip_height_scale = " << bdry_to_tip_height_scale << std::endl;

    // Boolean flag to determine if restiffening is being simulated.
    bool isRestiffening = true; // false; // change this to true (nav) - testing for FvK. was false but I have changed to true - nav 8/5/24
    std::cout << "Are we simulating a case where restiffening or the restoration (even just partially) of mechanical properties? " << isRestiffening << " (bool) " << std::endl;

    // Scaling factors for restiffening regions.
    double scale_linear_restiff;
    double scale_bend_restiff;
    double scale_area_restiff;

    // If restiffening is enabled, calculate the scaling factors.
    if (isRestiffening == true)
    {
        // Scale for linear springs in restiffening regions.
        scale_linear_restiff = linearSpringInfoVecs.spring_constant * 0.1; // 0.25;//25.0/2.5;//75.0/15.0; nav changing it to 0.1 8/5/24
        // Scale for bending springs in restiffening regions.
        scale_bend_restiff = bendingTriangleInfoVecs.spring_constant * 0.1; // 0.05;//10.0/1.0;//75.0/7.5; nav changing it to 0.1 8/5/24
                                                                            //  Scale for area springs in restiffening regions.
        scale_area_restiff = areaTriangleInfoVecs.spring_constant * 0.1; // 0.25;//50.0/5.0;//75.0/15.0; nav changing it to 0.1 8/5/24

        std::cout << "restiff region linear = " << scale_linear_restiff << std::endl;
        std::cout << "restiff region bend = " << scale_bend_restiff << std::endl;
        std::cout << "restiff region area = " << scale_area_restiff << std::endl;
        std::cout << "If restiff value is higher than that of the weakened case, something is wrong" << std::endl;
    }

    // Flag to determine if the system has been triggered.
    bool triggered = false;

    // Initialize the current total simulation step.
    generalParams.current_total_sim_step = 0;

    // Maximum relaxation steps before growth and edge swap.
    int relax_max_steps_before_growth_and_edgeswap = 3e3;
    int relax_max_steps_before_strain_tensor_and_gradient_descent = 3e3;
    // std::cout<<"relax max steps before growth and edgeswap = "<<relax_max_steps_before_growth_and_edgeswap<<"*max_runTime"<<std::endl;
    std::cout << "relax max steps before strain tensor application and gradient descent = " << relax_max_steps_before_strain_tensor_and_gradient_descent << std::endl;

    // Create a shared pointer for Utilities.
    auto utilities_ptr = std::make_shared<Utilities>(coordInfoVecs, generalParams);

    // Create a shared pointer for SystemBuilder.
    auto build_ptr = weak_bld_ptr.lock();
    std::cout << "Declaration of rbc and n_rbc complete." << std::endl;
    std::cout << "Utilities_ptr declaration complete." << std::endl;

    /////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    /////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    /////////////////////////Build the "u" vector representing the external or internal influencer for polarization /////////////////////////
    /////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    /////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

    // Part 2

    // Variables to store the maximum and minimum heights of cell triangles.
    double max_cell_triangle_height, min_cell_triangle_height, v1, v2, v3, v4, cell_height;

    // Initialize the maximum and minimum cell triangle heights to extreme values.
    max_cell_triangle_height = -10000.0;
    min_cell_triangle_height = 10000.0;

    // Loop over all triangles to calculate the heights and find the maximum and minimum heights
    for (int i = 0; i < coordInfoVecs.num_triangles; i++)
    {

        // Skip trinagles with invalid node indices.
        if (coordInfoVecs.triangles2Nodes_1[i] >= (INT_MAX - 100) || coordInfoVecs.triangles2Nodes_2[i] >= (INT_MAX - 100) || coordInfoVecs.triangles2Nodes_3[i] >= (INT_MAX - 100))
        {
            continue;
        }
        if (coordInfoVecs.triangles2Nodes_1[i] <= (-INT_MAX + 100) || coordInfoVecs.triangles2Nodes_2[i] <= (-INT_MAX + 100) || coordInfoVecs.triangles2Nodes_3[i] <= (-INT_MAX + 100))
        {
            continue;
        }

        // Get the Z coordinates of the nodes forming the current triangle.
        v1 = coordInfoVecs.nodeLocZ[coordInfoVecs.triangles2Nodes_1[i]];
        v2 = coordInfoVecs.nodeLocZ[coordInfoVecs.triangles2Nodes_2[i]];
        v3 = coordInfoVecs.nodeLocZ[coordInfoVecs.triangles2Nodes_3[i]];

        // Calculate the average Z coordinate of the triangle (cell triangle height).
        v4 = (v1 + v2 + v3) / 3.0;
        // std::cout<<"Max triangle heights (z-coordinates) = "<<v4<<std::endl;

        // Update the maximum and minimum cell triangle heights.
        if (v4 >= max_cell_triangle_height)
        {
            max_cell_triangle_height = v4;
        }
        if (v4 <= min_cell_triangle_height)
        {
            min_cell_triangle_height = v4;
        }
    }

    // Calculate the cell height as the difference between the maximum and minimum cell triangle heights.
    cell_height = (max_cell_triangle_height - min_cell_triangle_height);

    std::cout << "Max_cell_triangle_height = " << max_cell_triangle_height << ", min_cell_triangle_height = " << min_cell_triangle_height << std::endl;

    std::cout << "Cell height = " << cell_height << std::endl;

    // Message printed when max and min height of triangles has been determined.
    std::cout << "Determination of max triangle and min cell height complete." << std::endl;

    /////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    /////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    /////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    /////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    /////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

    // Part 3

    // Setting various simulation parameters and initializing variables.

    // Set the node mass for the simulation.
    generalParams.nodeMass = 1.0;

    // Initialize growth counter to keep track of growth events.
    int GROWTH_COUNTER = 0;

    // Set the minimum number of edge loops for edge-swap during growth events.
    int min_num_edge_loop = 1;
    std::cout << "min_num_edge_loop for edgeswap = " << min_num_edge_loop << std::endl;

    // DEfine max volume and bud area ratios for growth control.
    double MAX_VOLUME_RATIO = 2.0;
    double MAX_BUD_AREA_RATIO = 100.0;
    double MAX_APICAL_AREA_RATIO = 100.0;

    // Set the maximum number of growth events per growth cycle. (INT_MAX means unlimited)
    int MAX_GROWTH_PER_GROWTH_EVENT = 1; // INT_MAX;
    std::cout << "MAX_GROWTH_NUMBER (# of edge to expand) per growth event = " << MAX_GROWTH_PER_GROWTH_EVENT << std::endl;

    // Set the frequency of growth events (how many times Max_Runtime has to be reached to perform growth).
    int GROWTH_FREQUENCY = 25; // 150;//100;//95;//70;//25*3; // E.g., if Max_Runtime = 100, growth will occur every 25 time units.
    std::cout << "GROWTH_FREQ (how many times Max_Runtime has to be reached to perform growth) = " << GROWTH_FREQUENCY << std::endl;

    // Set growth frequency of growth events for variable edge-swap rate cases.
    int GROWTH_FREQUENCY2 = 25; // 150;//100;//95;//70;//25*3;
    std::cout << "GROWTH_FREQ2 (how many times Max_Runtime has to be reached to perform growth, for variable ES rate cases" << GROWTH_FREQUENCY2 << std::endl;

    // Set the point of transition for growth events (describes the ratio of the total simulation time for relaxation (edge-swap) frequency to change).
    double pointOfTransition = 0.10;
    std::cout << "Point of transition describes the ratio of the total time simulation time is reached for relaxation (edge-swap) frequency to change : " << pointOfTransition << std::endl;
    std::cout << "Point of transition is also used to indicate if strain_threshold needs to change or not" << std::endl;

    // Set the energy gradient threshold for growth events.
    double energy_gradient_threshold = 0.02; // 0.01; // Threshold used to trigger growth events based on energy gradients.
    std::cout << "ENERGY_GRADIENT_THRESHOLD = " << energy_gradient_threshold << std::endl;

    // Set kT_growth value for growth events.
    generalParams.kT_growth = 1.0;

    // Set the SCALE_TYPE for weakening during growth events. (0 to 4)
    // 0:= Gaussian-like weakening
    // 1:= a1*(pow(x,b)) + a2*(1-pow(x,b)) type weakening
    // 2:= pure Gaussian weakening
    // 3:= isotropic
    // 4:= hill equation
    // Note that (3) is used in combination with sigma = INT_MAX;
    generalParams.SCALE_TYPE = 3; // Original scale type was 3. Nav changed it to 0 for flat code. 6/2/24. \\ 0 did not work so nav changed it back to 3 8/18/24
    std::cout << "SCALE TYPE = " << generalParams.SCALE_TYPE << std::endl;
    std::cout << "0:= sigmoidal Gaussian-like weakening, 1:= a1*(pow(x,b)) + a2*(1-pow(x,b)) type weakening, 2:= pure Gaussian weakening, 3:= isotropic, 4:= hill equation" << std::endl;

    // Check and set additional parameters based on SCALE_TYPE.
    if (generalParams.SCALE_TYPE == 1)
    {
        generalParams.scaling_pow = 2.0;
        std::cout << "scaling_pow (this is for SCALE_TYPE = 1 case) = " << generalParams.scaling_pow << std::endl;
    }
    if (generalParams.SCALE_TYPE == 0)
    {
        generalParams.gausssigma = 0.1;
        std::cout << "gausssigma (this is for the SCALE_TYPE = 0 case) = " << generalParams.gausssigma << std::endl;
    }

    // Set the display_token to true for displaying additional information during growth events.
    bool display_token = true;

    // Declare variables for Hill function_dependent wall stiffness.
    double dtb_scaler, targetHillEqnPow;
    if (generalParams.SCALE_TYPE == 4)
    {
        generalParams.ratio_for_HillFunctionStiffness = 4.0;
        std::cout << "Hill function dependent wall stiffness triggers when the the distance between tip of the bud and the septin ring is " << generalParams.ratio_for_HillFunctionStiffness << std::endl;
        std::cout << "times larger than the equilibrium length Rmin" << std::endl;
        dtb_scaler = 1.0;
        targetHillEqnPow = 16.0;
        std::cout << "The EC50 position is scaled by " << dtb_scaler << " on the distance from tip to boundary, hence the EC50 occurs on dtb*" << dtb_scaler << "/dtb_max" << std::endl;
        std::cout << "Target hill equation power = " << targetHillEqnPow << std::endl;
    }

    // coordInfoVecs.scaling_per_edge.
    // generalParams.hilleqnconst = 0.9;
    // generalParams.hilleqnpow = 40.0;

    // Declare vectors to store nodes, triangles and edges in the growth region. (nav - unsure if I need to take these out since we will not be performing growth)
    std::vector<int> nodes_in_growth;
    std::vector<int> triangles_in_growth;
    std::vector<int> edges_in_growth;

    // Declare variables for distance to boundary and maximum distance used in Hill equation.
    double dtb;     // dtb := distance to boundary
    double dtb_max; // dtb_max := the max distance used to calculate the distance ratio in the Hill equation.

    // Declare variables for sigma (for gradient distribution variance) and sigma_true (for Gaussian-related distribution variance) used in SCALE_TYPE 0.
    double sigma, sigma_true;

    if (generalParams.SCALE_TYPE == 0)
    {
        sigma = 0.0;            // INT_MAX; //if this is set to be INT_MAX then we assume isotropic weakening.
        sigma_true = sqrt(0.5); // This is the variance used to calculate the scaling of the wall weakening.
        std::cout << "initial sigma (for gradient distribution variance), based on initial distribution of Cdc42, if using true gaussian weakening = " << sigma << std::endl;
        std::cout << "If sigma = INT_MAX, then we have isotropic weakening scenario" << std::endl;
        std::cout << "true sigma (for gaussian-related distribution variance) = " << sigma_true << std::endl;
    }

    // Set the insertion energy cost for material insertion during growth events.
    generalParams.insertion_energy_cost = -log(0.0025); // why is material insertion energy cost -log(0.0025) (question) Ask dr. Chen
    std::cout << "GROWTH: material insertion energy cost (dependent on local chemical concentration) = " << generalParams.insertion_energy_cost << std::endl;

    // Set the strain thresholds for the martial (material?) insertion probability calculation during growth events.
    double strain_threshold1 = 0.05; // 0.01; // Set strain_threshold for initial calculation.
    double strain_threshold2 = 0.05; // Set strain_threshold for subsequent changes if needed.
    generalParams.strain_threshold = strain_threshold1;
    std::cout << "GROWTH: critical strain threshold used for insertion probability calculation = " << generalParams.strain_threshold << ", value loaded = " << strain_threshold1 << std::endl;
    std::cout << "GROWTH: critical strain threshold used for insertion probability calculation if changes are needed= " << strain_threshold2 << std::endl;

    // Set the growth energy scaling for material insertion probability durng growth events.
    generalParams.growth_energy_scaling = 1.0; // 0.01375; // Set the scaling factor for growth energy.
    std::cout << "GROWTH ENERGY SCALING FOR MATERIAL INSERTION PROBABILITY = " << generalParams.growth_energy_scaling << std::endl;

    // Set the neighbour safeguardthreshold (the max number of neighboring nodes a node can have).
    generalParams.safeguardthreshold = 9;
    std::cout << "NEIGHBOR SAFE GUARD THRESHOLD = " << generalParams.safeguardthreshold << std::endl;

    //////////////////////////////////////////////////////////////////////////////////
    //////////////////////////////////////////////////////////////////////////////////
    ////////////////////////// PARAMETER SETTINGS ////////////////////////////////////
    //////////////////////////////////////////////////////////////////////////////////
    //////////////////////////////////////////////////////////////////////////////////

    // Part 4

    // Setting various simulation parameters and initializing variables.

    // Initialize total growth counters to keep track of growth events.
    int TOTAL_GROWTH_COUNTER = 0;
    int TOTAL_GROWTH_ATTEMPT = 0;

    // Set the maximum runtime for the simulation (in time units)
    double Max_Runtime = generalParams.dt * 50.0; //*50.0; nav commented this out for double sheet testing. 9/16/2024. //generalParams.dt*10.0;//generalParams.dt*50.0; Nav commented out this last part for flat code 6/5/24. //50.0; time units //time step is here. Nav has once again commented out the __.dt*10 to make it run longer. 8/2/24

    std::cout << "dt = " << generalParams.dt << std::endl;
    double minimal_run_time_ratio = 1.0;                 // Not used in this section.
    double Max_RunStep = Max_Runtime / generalParams.dt; // Calculate the maximum runstep based on the max runtime.
    std::cout << "Max runtime = " << Max_Runtime << std::endl;
    std::cout << "Max runstep = " << Max_RunStep << std::endl;

    // Boolean flag to determine if the simulation should continue running.
    bool runSim = false; // turn off the budding simulation and edgeswap

    // Declare and initialize variables for growth-related calculations.
    int num_edge_loop;
    double initial_kT;
    initial_kT = generalParams.kT; // Stores the initial kT value for the acceptance of changes after looping through every edge within proximity.

    // double SAMPLE_SIZE = 0.05;//0.025;
    // std::cout<<"Sample ratio: "<<SAMPLE_SIZE<<std::endl;
    // std::cout<<"If the Sample raio is 0, it means we have chosen a fixed number of attempt throughout the simulation"<<std::endl;

    // Set the sample size for testing edges during bondflip remeshing.
    double SAMPLE_SIZE = 2;
    std::cout << "Sample size: " << SAMPLE_SIZE << std::endl;

    // Set the record frequency for the dsaving simulation data (time steps).
    int RECORD_TIME = 1; // round(Max_RunStep/2); Save data every time step.
    std::cout << "Record frequency = " << RECORD_TIME << std::endl;

    // translate_frequency determines the frequency for the mesh to re-center and perform dynamical remeshing.
    int translate_frequency = 10;
    std::cout << "recentering of the model cell frequency = " << translate_frequency << std::endl;

    // Set the total number of growth events and targeted growth events for the simulation.
    int NUMBER_OF_GROWTH_EVENT = 1000;                        // changed by nav on 03/04/2025//200;//2000 - nav changed this for flat code. 6/2/24;//1000;//1000*2; // Total number of growth events. // Nav once again changed it to 1000 from 200. 8/26/24
    int NUMBER_OF_TARGETED_GROWTH_EVENT = 1;                  // Number of targeted growth events.
    int NKBT = GROWTH_FREQUENCY * NUMBER_OF_GROWTH_EVENT * 2; // nav changed this 03/04/2025//10; // GROWTH_FREQUENCY*NUMBER_OF_GROWTH_EVENT*2; Nav changed this last one for the flat code. 6/5/24.//GROWTH_FREQUENCY*NUMBER_OF_GROWTH_EVENT;//10000;//7500; Nav is now changing it back from 10 to turn growth on. 8/26/24
    std::cout << "Number of edge-swap per kBT value (or total number of edge-swap if kBT is fixed) = " << NKBT << std::endl;

    int GROWTH_FREQUENCY_SCALE = 4;
    std::cout << "GROWTH FREQ SCALE: decides how many growth event must be checked before recording the result = " << GROWTH_FREQUENCY_SCALE << std::endl;

    double min_kT = -0.1; // 0.21;
    std::cout << "min kT for simulation termination = " << min_kT << std::endl;

    // Initialize WHEN for conditional checks.
    int WHEN = 0;

    // Initialize the following for energy calculations.
    double old_total_energy = 0.0;
    double new_total_energy = 0.0;
    double energy_gradient = 0.0;
    double energy_rep = 0.0;

    // Initialize the number of simulation steps run.
    int Num_of_step_run = 0;

    // Initialize min_energy.
    double min_energy;

    // Count the total number of true edges (edges connected to valid nodes).
    generalParams.true_num_edges = 0;
    for (int i = 0; i < coordInfoVecs.num_edges; i++)
    {
        if (coordInfoVecs.edges2Nodes_1[i] != INT_MAX && coordInfoVecs.edges2Nodes_2[i] != INT_MAX)
        {
            generalParams.true_num_edges += 1;
        }
    }

    // double COMPRESS = 2.0227;
    //  double COMPRESS2 = -2.0227;

    /////////////////////////////////////////////////////////////////
    /////////////////////// MEMBRANE RELATED ////////////////////////
    /////////////////////////////////////////////////////////////////

    // Part 5

    // Membrane relatde parameters and variables initialization.

    // Initilize the following vectors with zeros.
    std::vector<double> nodenormal_1(generalParams.maxNodeCount, 0.0);
    std::vector<double> nodenormal_2(generalParams.maxNodeCount, 0.0);
    std::vector<double> nodenormal_3(generalParams.maxNodeCount, 0.0);

    // Variable to keep track of how many times the linearSpringsInfoVecs.spring_constant_rep1 is reduced.
    int reduce_counter = 0;

    // Set VOLUME_FACTOR to the maximum volume ratio (target volume = VOLUME_FACTOR * initial_volume).
    double VOLUME_FACTOR = MAX_VOLUME_RATIO; // 1.6;//2.25;

    // double tip_depth = 0.5;
    // tip_depth is currently unused.

    // Line tension threshold for the activation of line tension (currently not used)
    double LINE_TENSION_THRESHOLD = -10000.0;
    std::cout << "LINE TENSION THRESHOLD for activation of line tension = " << LINE_TENSION_THRESHOLD << std::endl;

    // Volume threshold for the activation of weakened membrane (currently not used).
    double VOLUME_THRESHOLD = 0.0;
    std::cout << "VOLUME THRESHOLD for activation of weakened membrane = " << VOLUME_THRESHOLD << std::endl;

    // The minimum height of the z-coordinate of the membrane node to be considered in the area of weakened mechanical properties.
    double weakened = 0.0; // 1.90;//6.0; Nav changed it from 1.90 t0 0.0 to have weakened area increased. 8/26/24

    // double tip_base = 6.0;
    // tip_base currently unused.

    // RULES_OF_EXPAN controls how the EXPAN_THRESHOLD is applied:
    //  // 1:= Both trianglular areas must exceed the threshold value.
    //  // 2:= If one trianglular area exceeds the treshold value while the other exceeds the secondary threshold value.
    //  // 3:= If the combined area of the two triangles exceed 2*EXPAN_THRESHOLD.
    //  // 4:= If a selected edges exceed the threshold value, split the triangles associated with the edge.

    for (int i = 0; i < generalParams.maxNodeCount; i++)
    {
        generalParams.centerX += coordInfoVecs.nodeLocX[i];
        generalParams.centerY += coordInfoVecs.nodeLocY[i];
        generalParams.centerZ += coordInfoVecs.nodeLocZ[i];
    }

    generalParams.centerX = generalParams.centerX / generalParams.maxNodeCount;
    generalParams.centerY = generalParams.centerY / generalParams.maxNodeCount;
    generalParams.centerZ = generalParams.centerZ / generalParams.maxNodeCount;

    // Initialization of newcenterX, newcenterY, newcenterZ for recentering of the mesh.
    double displacementX, displacementY, displacementZ;
    double newcenterX, newcenterY, newcenterZ;

    std::vector<int> VectorShuffleForGrowthLoop;
    std::vector<int> VectorShuffleForFilamentLoop;
    std::vector<int> VectorShuffleForEdgeswapLoop;

    // Find the min and max height of the membrane nodes and their indices.
    double min_height = coordInfoVecs.nodeLocZ[0];
    double max_height = -10000.0;
    int max_height_index = -1;
    for (int k = 0; k < generalParams.maxNodeCount; k++)
    {
        if (coordInfoVecs.nodeLocZ[k] >= max_height)
        {
            max_height = coordInfoVecs.nodeLocZ[k];
            max_height_index = k;
        }
    }

    // Max and min height of the membrane nodes, these have to be changed if the mesh used is changed.

    // Set the equilibrium length of an edge of the triangle.
    generalParams.Rmin = 0.001; // 0.75;//0.5; nav changed this once again. Made it larger 11/7/24 //0.0001; nav changed this on 10/10/24 for the double layer code. This value of 0.0001 works for the circular sheet. //0.3012; changed by nav on 6/5/24 for flat code. //0.15; //equilibrium length (Nav) changed by nav again to 0.5 from 0.15. 8/5/24 11/8/24 5 worked for small number of nodes. < Nav

    generalParams.abs_Rmin = generalParams.Rmin; // 0.15;
    std::cout << "abs_Rmin = " << generalParams.abs_Rmin << std::endl;

    // Equilibrium distance between membrane node for volume exclusion.
    //  Initialize the following which represents the equilibrium triangular area.
    areaTriangleInfoVecs.initial_area = 1.0; // 0.039;nav changed this to make it larger 11/7/24 //2835;//0.009808;//0.039;//0.03927344;//0.009817; 11/8/24 25 worked for small number of nodes. < Nav
    std::cout << "equilibrium triangular area = " << areaTriangleInfoVecs.initial_area << std::endl;

    // Set ljInfoVecs parameters (currently all set to 0.0, indicating no interactions).

    // Equilibrium triangular area.
    ljInfoVecs.Rmin_M = 0.0;

    // Equilibrium distance between the nucleus particle and membrane.
    ljInfoVecs.Rcutoff_M = 0.0;

    // Maximal interaction range between the nucleus and membrane.
    ljInfoVecs.Rmin_LJ = 0.0; // 3.0//1.0;

    // Equilibrium distance between nuclei.
    ljInfoVecs.Rcutoff_LJ = 0.0; // 3.0;//1.0;

    // Maximal interaction range between the nuclei.
    ljInfoVecs.epsilon_M_att1 = 0.0; // 6.0;//16.0;
    ljInfoVecs.epsilon_M_att2 = 0.0; // 1.0;//1.0;
    std::cout << "Morse_NM_D_att = " << ljInfoVecs.epsilon_M_att1 << std::endl;
    std::cout << "Morse_NM_a_att = " << ljInfoVecs.epsilon_M_att2 << std::endl;

    // Coefficient for the attractive interaction between nuclei and membrane.
    ljInfoVecs.epsilon_M_rep1 = 0.0; // 12.5;//16.0;
    ljInfoVecs.epsilon_M_rep2 = 0.0; // 0.5;//1.0;
    std::cout << "Morse_NM_D_rep = " << ljInfoVecs.epsilon_M_rep1 << std::endl;
    std::cout << "Morse_NM_a_rep = " << ljInfoVecs.epsilon_M_rep2 << std::endl;

    // Coefficient for the repulsive interaction between nuclei and membrane.
    ljInfoVecs.epsilon_LJ_rep1 = 0.0; // 10.0;//0.5;// 0.06;//7.5;
    ljInfoVecs.epsilon_LJ_rep2 = 0.0; // 0.5;//1.0;//1.0;//1.0;
    std::cout << "Morse_NN_D = " << ljInfoVecs.epsilon_LJ_rep1 << std::endl;
    std::cout << "Morse_NN_a = " << ljInfoVecs.epsilon_LJ_rep2 << std::endl;
    // Coefficient of the interaction between nuclei.

    linearSpringInfoVecs.spring_constant_rep1 = 0.01; // 0.023;
    linearSpringInfoVecs.spring_constant_rep2 = 9.0;  // 5.0;
    std::cout << "Membrane volume exclusion Morse D = " << linearSpringInfoVecs.spring_constant_rep1 << std::endl;
    std::cout << "Membrane volume exclusion Morse a = " << linearSpringInfoVecs.spring_constant_rep2 << std::endl;
    // The coefficient used for non-neighboring membrane node volume exclusion.
    // rep1 is the "D" and rep2 is the "alpha" in the standard form of Morse potential.

    generalParams.volume_spring_constant = 0.2; //(1.0/3.0)*areaTriangleInfoVecs.initial_area*1.0;
    std::cout << "spring constant for surface normal expansion (pressure within the cell) = " << generalParams.volume_spring_constant << std::endl;

    generalParams.line_tension_constant = 0.0; // 250.0; // Value that generated flat sheet is 0.0. 8/14/24
    std::cout << "spring constant for the septin ring (before budding) = " << generalParams.line_tension_constant << std::endl;

    // Equilibrium length of each segment of the septin ring.
    generalParams.length_scale = 0.0; // 1.0*generalParams.Rmin;//nav changed this from 0 to the current value to test the boundary nodes. 03/06/2025 //0.85;//0.1577;//1.0*generalParams.Rmin;// 0.8333; //nav changed this to be 0.0 from 1.0. 8/5/24; Flat sheet generated when septin ring was 0.0. 8/14/24


    // Set spring constants for apical basal and vertical layers. 
   // generalParams.vertical_spring_constant = 250.0;
    
    // Set weakened region scaling factors.
    generalParams.maxSpringScaler_linear = 1.0;
    generalParams.maxSpringScaler_area = 1.0;
    generalParams.maxSpringScaler_bend = 1.0;
    double scale_linear = linearSpringInfoVecs.spring_constant * 0.25; // 0.25;//25.0/2.5;//75.0/15.0; flat sheet generated when multiplied by 1; 8/15/24; Changing it to 0.25 makes it wrinkle up from before. Same with all three below 8/15/24
    double scale_bend = bendingTriangleInfoVecs.spring_constant * 1;   // 0.05;//10.0/1.0;//75.0/7.5;  flat sheet generated when multiplied by 1; 8/15/24;
    double scale_area = areaTriangleInfoVecs.spring_constant * 0.25;   // 0.25;//50.0/5.0;//75.0/15.0;  flat sheet generated when multiplied by 1; 8/15/24;
    // nav changed all of the above to their original values to see how it affects budding. 8/26/24
    std::cout << "weakened region linear (before budding) = " << scale_linear << std::endl;
    std::cout << "weakened region bend (before budding) = " << scale_bend << std::endl;
    std::cout << "weakened region area (before budding) = " << scale_area << std::endl;

    // Scaling factor of the weakend mechanical properties.
    linearSpringInfoVecs.spring_constant_weak = scale_linear;
    bendingTriangleInfoVecs.spring_constant_weak = scale_bend;
    areaTriangleInfoVecs.spring_constant_weak = scale_area;

    // Set the following for bending angle equilibrium.
   // bendingTriangleInfoVecs.initial_angle = 0.0087;                 // 0.087165870975460;//0.087249;//0.04335; // This is also an angle we need to change to make the code flat. Nav. Make this into some random drastic value.
    bendingTriangleInfoVecs.initial_angle_raft = 0.087165870975460; // 0.087249;//0.04335; //The raft and coat versions are from some weird legacy thing Kevin was testing - Nav.
    bendingTriangleInfoVecs.initial_angle_coat = 0.087165870975460; // 0.087249;//0.04335;
  //  std::cout << "equilibrium bending angle of the membrane = " << bendingTriangleInfoVecs.initial_angle << std::endl;
    // raft and coat are current unused due to the assumption of uniform preferred curvature.

    bendingTriangleInfoVecs.initial_angle_bud = bendingTriangleInfoVecs.initial_angle[1];
    std::cout << "equilibrium bending angle of the bud = " << bendingTriangleInfoVecs.initial_angle_bud << std::endl;

    // following vectors currently empty.
    std::vector<int> pull_nodes_up;   // = {35,    76,    79,   111,   113,   151,   153,   360,   361,   362,   363,   364,   365,   505,   506,   515,   516,   593,   632};//{35, 360,   361,   362,   363,   364,   365};
    std::vector<int> pull_nodes_down; // = {38,    86,    89,   121,   123,   144,   146,   378,   379,   380,   381,   382,   383,   535,   536,   545,   546,   602,   626};//{38, 378,   379,   380,   381,   382,   383};
    std::vector<int> push_nodes_down;
    std::vector<int> push_nodes_up;

    /////////////////////////////////////////////////////////////////
    ////////////////// END OF MEMBRANE RELATED //////////////////////
    /////////////////////////////////////////////////////////////////

    /////////////////////////////////////////////////////////////////
    //////////////////////// NULCEUS RELATED ////////////////////////
    /////////////////////////////////////////////////////////////////

    // Part 6
    // Nucleus related parameters and variables initialization. // Nav - nucleus related stuff should be commented out for flat code. 6/2/24

    // Set beta1 and beta2 to manually push the nucleus tip and the remainder of the nucleus vertically.
    double beta1 = 0.0;
    double beta2 = 0.0;
    std::cout << "manual push speed for the nucleus tip = " << beta1 << std::endl;
    std::cout << "manual push speed for the remainder of the nucleus = " << beta2 << std::endl;
    // beta1 is the vertical speed (0, 0, beta1) applied to the nucleus tip.
    // beta2 is the vertical speed (0, 0, beta2) applied to the remainder of the nucleus.

    // V1, V2 and V3 are vectors representing the (x,y,z)-coordinates of the nucleus particles.
    // Note: These vectors are currently initialized with single values for demonstration purposes.
    std::vector<double> V1 = {-0.0}; /*, 0.0  ,  0.1966  ,  0.5547 ,  -0.4689 ,   0.2422 ,  -0.2229,
                                    -0.4312 ,  -0.0185 ,   0.2887 ,   0.3187 ,   0.7140 ,
                                     0.2231 ,  -0.1921 ,	  -0.5541 ,   -0.1542 ,   -0.1689 ,    0.4391 ,
                                    -0.6661 ,  -0.6381 ,   0.6256 ,   0.0466 ,  -0.0610 ,   0.5134};
                                     */
    std::vector<double> V2 = {0.0};  /*, 0.0 ,  -0.4595 ,  -0.4129 ,   0.0954 ,   0.1764 ,   0.4186 ,
                                    -0.5602 ,  -0.6082 ,  -0.5318 ,   0.3561 ,   0.0753 ,
                                    -0.0917 ,  -0.2596 , 0.2871 ,  -0.3918 ,   0.5195 ,   0.5579 ,
                                    -0.2805 ,   0.0133  , -0.0073 ,   0.7426 ,   0.0614 ,  -0.1506};
                                      */
    std::vector<double> V3 = {0.0};  // initailly 0.6390. changing it to 0.0 (nav)
                                     /*, 0.0 ,  -0.5511 ,   0.0267 ,  -0.5240  , -0.4004 ,   0.2850 ,
                                                                   0.2032 ,  -0.1771 ,   0.4048 ,   0.3461 ,  -0.2034 ,
                                                                   0.5041 ,  -0.4535 ,	-0.1241 ,   0.5722 ,  -0.3748 ,  -0.1335 ,
                                                                   -0.0851 ,   0.3213 ,   0.2389 ,   0.0044 ,  -0.7424 ,  -0.7450};
                                                                   */

    // Push the (x,y,z)-coordinates of the nuclues particles into the ljInfoVecs vectors.
    // These vectors will be used for interactions between the nuclues and other particles.
    for (int i = 0; i < V1.size(); i++)
    {
        ljInfoVecs.LJ_PosX_all.push_back(V1[i]);
        ljInfoVecs.LJ_PosY_all.push_back(V2[i]);
        ljInfoVecs.LJ_PosZ_all.push_back(V3[i]);
    }

    // Set NUCLEUS_UpperHEM_BASE and NUCLEUS_LOWERHEM_BASE, which define the z-coordinate requirement for nucleus particles.
    // to be considered tip-region or base-region. This is used to determine where to apply spring or constant force.
    double NUCLEUS_UPPERHEM_BASE = 0.0; // initially 0.5. Changing it to 0.0 (nav)
    double NUCLEUS_LOWERHEM_BASE = 0.0; // initially -0.6. Changing it to 0.0 (nav)

    //////////////////////////////////////////////////////////////////
    ///////////////// END OF NUCLEUS RELATED /////////////////////////
    //////////////////////////////////////////////////////////////////

    // Part 7

    //////////////////////////////////////////////////////////////////
    /////////// IDENTIFYING REGIONS WITH DIFFERENT MECH PROP /////////
    //////////////////////////////////////////////////////////////////


    /*
    
                    DV BOUNDARY REGION 
    
    */
    
    
    // calculate center of the apical layer of disc
    int max_upperhem_nodes = 0;
    for (int i = 0; i<generalParams.maxNodeCount; i++){
    
      if (generalParams.nodes_in_upperhem[i]==1){
          //std::cout<<"node in upperhem = " << generalParams.nodes_in_upperhem[i] <<std::endl;
          max_upperhem_nodes++;
      }
    
    }
    
    std::cout<< "\n total number of nodes in upperhem = " << max_upperhem_nodes <<std::endl;

    
    double cx_b=0, cy_b=0, cz_b=0;  int nLower=0;
    double cx_a=0, cy_a=0, cz_a=0;  int nUpper=0;
    
    for(int i=0;i<generalParams.maxNodeCount;++i){
        if(generalParams.nodes_in_upperhem[i]==1){
            cx_a += coordInfoVecs.nodeLocX[i];
            cy_a += coordInfoVecs.nodeLocY[i];
            cz_a += coordInfoVecs.nodeLocZ[i];
            ++nUpper;
        }
        else if(generalParams.nodes_in_upperhem[i] == -1){
            cx_b += coordInfoVecs.nodeLocX[i];
            cy_b += coordInfoVecs.nodeLocY[i];
            cz_b += coordInfoVecs.nodeLocZ[i];
            ++nLower;
        }
    }
    cx_a/=nUpper;  cy_a/=nUpper;  cz_a/=nUpper; // Apical center   
    cx_b/=nLower;  cy_b/=nLower;  cz_b/=nLower; // Basal center   
    
    
    // Boundary node along the x-direction -- this is arbitrary. You can choose any axis to go along. 
    
    double R = 0.0, DR = 0.0;
    
    for (int i = 0; i<generalParams.maxNodeCount; i++){
        if (generalParams.nodes_in_upperhem[i]!=1) continue;
        double dx = coordInfoVecs.nodeLocX[i] - cx_a;
        double dy = sqrt((coordInfoVecs.nodeLocY[i] - cy_a)*(coordInfoVecs.nodeLocY[i] - cy_a));
        if (dy<1e-3 && dx>R) R=dx; DR = dy;
    }
    
    
    // for convenience 
    
    generalParams.centerX = cx_a; 
    generalParams.centerY = cy_a;
    generalParams.centerZ = cz_a;
    
    std::cout<< "boundary node in the x direction (apical layer) =  (" <<R<<", "<<DR<< ") \n"<<std::endl;
    
    
    // Identifying regions with different mechanical properties and finding the coundary nodes and edges of the upper hemisphere.

    /////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    //// Define vectors to store information about boundary edges and edge indices.
    // std::vector<int> out;
    ////int ALPHA;
    // std::vector<bool> boundary_edges;
    // boundary_edges.reserve(coordInfoVecs.num_edges);
    //
    //// Populate the boundary_edges vector to identify boundary edges in the mesh.
    // for (int i = 0; i < coordInfoVecs.num_edges; i++){
    //		if (coordInfoVecs.edges2Triangles_1[i] == coordInfoVecs.edges2Triangles_2[i]){
    //			  boundary_edges.push_back(true); // If the edge connects to only one triangle, it's a boundary edge.
    //		}
    //		else {
    //			  boundary_edges.push_back(false);
    //		}
    // }
    //
    //// Create a vector to store edge indices that are not boundary edges.
    // std::vector<int> edgeIndices;
    // edgeIndices.reserve(coordInfoVecs.num_edges);
    // for (int i = 0; i < coordInfoVecs.num_edges; ++i){
    //		if (boundary_edges[i] == false){
    //			  edgeIndices.push_back(i); // Store the indices of non-boundary edges in the edgeIndices vector.
    //		}
    //		else {
    //			  edgeIndices.push_back(-1);
    //		}
    // }
    //
    //
    //// Remove invalid (negative) indices from edgeIndices.
    // auto it = remove_if(edgeIndices.begin(), edgeIndices.end(),  [](const int i) {return i < 0; });
    // edgeIndices.erase(it, edgeIndices.end());
    //// Make sure boundaries_in_upperhem is resized appropriately.
    // generalParams.boundaries_in_upperhem.resize(coordInfoVecs.num_edges);
    //
    //
    //// Moved boundary part 8/26/24 nav
    //  //Find the boundary of the nodes_in_upperhem region
    //	//generalParams.boundaries_in_upperhem.resize(coordInfoVecs.num_edges);
    //	std::vector<int> boundary_node_list;
    //	std::vector<int> boundary_edge_list;
    //	for (int i = 0; i < coordInfoVecs.num_edges; i++){
    //		double T1 = coordInfoVecs.edges2Triangles_1[i];
    //		double T2 = coordInfoVecs.edges2Triangles_2[i];
    //		if (T1 >= (INT_MAX - 1000) || T1 < 0 || T2 >= (INT_MAX-1000) || T2 < 0){
    //			continue;
    //		}
    //		if (generalParams.triangles_in_upperhem[T1] == 1 && generalParams.triangles_in_upperhem[T2] == 1){
    //			generalParams.boundaries_in_upperhem[i] = 1;
    //			//std::cout<<generalParams.boundaries_in_upperhem[i]<<std::endl;
    //		  generalParams.triangles_in_upperhem[T1] = 0;
    //			generalParams.triangles_in_upperhem[T2] = 0;
    //			double bdry_node1 = coordInfoVecs.edges2Nodes_1[i];
    //			double bdry_node2 = coordInfoVecs.edges2Nodes_2[i];
    //			//std::cout<<"septin ring nodes - bdrynode1 = "<<bdry_node1<<std::endl;
    //      boundary_node_list.push_back(bdry_node1);
    //			boundary_node_list.push_back(bdry_node2);
    //			boundary_edge_list.push_back(i);
    //
    //
    //			//generalParams.nodes_in_upperhem[bdry_node1] = 0;
    //			//generalParams.nodes_in_upperhem[bdry_node2] = 0;
    //			//coordInfoVecs.isNodeFixed[bdry_node1] = true;
    //			//coordInfoVecs.isNodeFixed[bdry_node2] = true;
    //		}
    //	/*	else if (generalParams.triangles_in_upperhem[T1] != 1 && generalParams.triangles_in_upperhem[T2] == 1){
    //			generalParams.boundaries_in_upperhem[i] = 1;
    //			std::cout<<generalParams.boundaries_in_upperhem[i]<<std::endl;
    //			generalParams.triangles_in_upperhem[T1] = 0;
    //			generalParams.triangles_in_upperhem[T2] = 0;
    //			double bdry_node1 = coordInfoVecs.edges2Nodes_1[i];
    //			double bdry_node2 = coordInfoVecs.edges2Nodes_2[i];
    //			boundary_node_list.push_back(bdry_node1);
    //			boundary_node_list.push_back(bdry_node2);
    //			boundary_edge_list.push_back(i);
    //			//generalParams.nodes_in_upperhem[bdry_node1] = 0;
    //			//generalParams.nodes_in_upperhem[bdry_node2] = 0;
    //		 coordInfoVecs.isNodeFixed[bdry_node1] = true;
    //		 coordInfoVecs.isNodeFixed[bdry_node2] = true;
    //		}*/
    //		else {
    //			generalParams.boundaries_in_upperhem[i] = -1;
    //		  //std::cout<<generalParams.boundaries_in_upperhem[i]<<std::endl;
    //		}
    //	}
    //
    // std::cout<<"size of boundary_node_list (this is double-counted) = "<<boundary_node_list.size()<<std::endl;

    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

    // Ensure boundaries_in_upperhem is resized.
    // generalParams.boundaries_in_upperhem.resize(coordInfoVecs.num_edges);
    // Ensure boundaries_in_upperhem is resized.
    generalParams.boundaries_in_upperhem.resize(coordInfoVecs.num_edges);

    std::cout << "boundaries in upperhem = " << generalParams.boundaries_in_upperhem.size() << std::endl;

    std::vector<int> boundary_edge_list;
    std::vector<int> boundary_node_list;

    std::cout << "edges2Triangles_1 = " << coordInfoVecs.edges2Triangles_1.size() << std::endl;
    std::cout << "edges2Triangles_2 = " << coordInfoVecs.edges2Triangles_2.size() << std::endl;

    //std::cout << "generalParams.nodes in upperhem size = " << generalParams.nodes_in_upperhem.size() << std::endl;

    for (int i = 0; i < coordInfoVecs.num_edges; i++)
    {
        int T1 = static_cast<int>(coordInfoVecs.edges2Triangles_1[i]);
        int T2 = static_cast<int>(coordInfoVecs.edges2Triangles_2[i]);

        // std::cout<<"it got till here - nav "<< std::endl;// it got till here.

        // Optionally check if the triangle indices are valid.
        if (T1 < 0 || T2 < 0 || T1 >= (INT_MAX - 1000) || T2 >= (INT_MAX - 1000))
        {
            continue;
        }

        // std::cout<<"it got till here - nav 1 "<< std::endl; // it got till here.

        // Mark edge as boundary if the two adjacent triangle IDs are identical.
        // std::cout<<"T1 = "<<T1<<std::endl;
        // std::cout<<"T2 = "<<T2<<std::endl;
        // std::cout<<"generalParams.edges_in_upperhem["<<i<<"] = "<< generalParams.edges_in_upperhem[i]<<std::endl;

        if (T1 == T2 && generalParams.edges_in_upperhem[i] == -1)
        {
            generalParams.boundaries_in_lowerhem[i] = 1;
            boundary_edge_list.push_back(i); // This is to store the total number of boundary edges.
            int bdry_node1 = static_cast<int>(coordInfoVecs.edges2Nodes_1[i]);
            int bdry_node2 = static_cast<int>(coordInfoVecs.edges2Nodes_2[i]);
            boundary_node_list.push_back(bdry_node1);
            boundary_node_list.push_back(bdry_node2);

            // mark these nodes as boundary or (fixed).
            generalParams.nodes_in_upperhem[bdry_node1] = -1;
            generalParams.nodes_in_upperhem[bdry_node2] = -1;
            coordInfoVecs.isNodeFixed[bdry_node1] = false;
            coordInfoVecs.isNodeFixed[bdry_node2] = false;
            
        }
        // This is for apical boundary nodes.
        if (T1 == T2 && generalParams.edges_in_upperhem[i] == 1)
        { // nav added the second conditional && generalParams.nodes_in_upperhem[i]==1 so that the new apical model would work.
            generalParams.boundaries_in_upperhem[i] = 1;
            boundary_edge_list.push_back(i);

            // std::cout<<"it got till here - nav 3 "<< std::endl;
            int bdry_node1 = static_cast<int>(coordInfoVecs.edges2Nodes_1[i]);
            int bdry_node2 = static_cast<int>(coordInfoVecs.edges2Nodes_2[i]);
            boundary_node_list.push_back(bdry_node1);
            boundary_node_list.push_back(bdry_node2);

            // Optionally mark these nodes as boundary (or fixed).
            generalParams.nodes_in_upperhem[bdry_node1] = 1; // nav changed this from 0 to 10.
            generalParams.nodes_in_upperhem[bdry_node2] = 1; // nav changed this from 0 to 10.
            coordInfoVecs.isNodeFixed[bdry_node1] = false;
            coordInfoVecs.isNodeFixed[bdry_node2] = false;

            // std::cout<<"it got till here - nav 4 "<< std::endl;
        }
        else
        {
            generalParams.boundaries_in_upperhem[i] = -1;
        }
    }

    std::cout << "size of boundary_edge_list = " << boundary_edge_list.size() << std::endl;
    std::cout << "size of boundary_node_list (double-counted) = " << boundary_node_list.size() << std::endl;

    // Count the true number of edges in the upper hemisphere.
    int true_num_edges_in_upperhem = 0;
    int edges_in_upperhem_COUNT = 0;
    for (int i = 0; i < coordInfoVecs.num_edges; i++)
    {
        if (generalParams.edges_in_upperhem_list[i] != INT_MAX && generalParams.edges_in_upperhem_list[i] >= 0)
        {
            true_num_edges_in_upperhem += 1;
            edges_in_upperhem_COUNT += 1;
        }
    }

    // Define a row2 vector to store specific node indices. These are the specific nodes in the septin ring. Nav replaced the following hard coded row with a boundary node list from the segment of code she moved. 8/26/24
    // std::vector<int> row2 = boundary_node_list;//nav commenting out on 03/09/2025. //{35 ,   76 ,   79 ,  111 ,  113 ,  151 ,  153 ,  360 ,  361 ,  362 ,  363 ,  364 ,  365 ,  505 ,  506 ,  515 ,  516 ,  593 ,  632};
    // nav commenting the above out to see if row2 can be defined later for the septin ring. 8/19/24 nav putting these back in.

    // Identify nodes in the upper hemisphere based on their z-coordinates.
    // for (int i = 0; i < generalParams.maxNodeCount; i++){
    //   // generalParams.nodes_in_upperhem[i] = -1; (nav commented this out for flat virus code 5/29/24)
    //  // generalParams.nodes_in_upperhem[i] = 1; // (nav uncommented this for flat virus code 5/29/24)
    //  std::cout<<"nodes in upperhem = "<<generalParams.nodes_in_upperhem[i]<<std::endl;
    //}

    // Nav commented out the following for flat virus code 5/29/24. nav put it back in 8/19/24. nav commented out again 03/09/2025
    // for (int i = 0; i < row2.size(); i++){
    //		generalParams.nodes_in_upperhem[row2[i]] = 1;
    // }

    // nav commented out 03/09/2025. Calculate the minimum z-coordinate of the nodes in row2.
    // double min_septin_z = 1000.0;
    // for (int i = 0; i < row2.size(); i++){
    //		if (coordInfoVecs.nodeLocZ[row2[i]] < min_septin_z){
    //			min_septin_z = coordInfoVecs.nodeLocZ[row2[i]];
    //		}
    //}

    // nav commented out again 03/09/2025. Identify additional nodes in the upper hemisphere based on their z-coordinates.
    // for (int i = 0; i < generalParams.maxNodeCount; i++){
    //		if (coordInfoVecs.nodeLocZ[i] > (min_septin_z)){
    //			generalParams.nodes_in_upperhem[i] = 1;
    //		}
    //}

    //// Identify triangles in the upper hemisaphere based on their nodes. nav uncommented till line 745 for tests 8/19/24. nav commented out 03/09/2025
    for (int i = 0; i < coordInfoVecs.num_triangles; i++)
    {
        if (coordInfoVecs.triangles2Nodes_1[i] >= (INT_MAX - 1000) || coordInfoVecs.triangles2Nodes_1[i] < 0)
        {
            generalParams.triangles_in_upperhem[i] = -1;
            continue;
        }
        else if (coordInfoVecs.triangles2Nodes_2[i] >= (INT_MAX - 1000) || coordInfoVecs.triangles2Nodes_2[i] < 0)
        {
            generalParams.triangles_in_upperhem[i] = -1;
            continue;
        }
        else if (coordInfoVecs.triangles2Nodes_3[i] >= (INT_MAX - 1000) || coordInfoVecs.triangles2Nodes_3[i] < 0)
        {
            generalParams.triangles_in_upperhem[i] = -1;
            continue;
        }

        int aaa = generalParams.nodes_in_upperhem[coordInfoVecs.triangles2Nodes_1[i]];
        int bbb = generalParams.nodes_in_upperhem[coordInfoVecs.triangles2Nodes_2[i]];
        int ccc = generalParams.nodes_in_upperhem[coordInfoVecs.triangles2Nodes_3[i]];

        if ((aaa + bbb + ccc) == 3)
        {
            generalParams.triangles_in_upperhem[i] = 1;
        }
        else
        {
            generalParams.triangles_in_upperhem[i] = -1;
        }
    }

    // Identify edges in the upper hemisphere based on their triangles.
    // Store the indices of edges in the upper hemisphere in the edges_in_upperhem_list vector.
    // Count the number of edges in the upper hemisphere.

    // int edges_in_upperhem_COUNT = 0;
    //
    // for (int i = 0; i < coordInfoVecs.num_edges; i++){
    //// NEW: Compute the edges midpoint z coordinate
    // double avg_z = (coordInfoVecs.nodeLocZ[coordInfoVecs.edges2Nodes_1[i]] +
    //                 coordInfoVecs.nodeLocZ[coordInfoVecs.edges2Nodes_2[i]]) / 2.0;
    // if (avg_z <= 0.5) {
    //     // If the midpoint is not above z=0.5, mark the edge as not in upper hemisphere.
    //     generalParams.edges_in_upperhem[i] = -1;
    //     generalParams.edges_in_upperhem_list[i] = -INT_MAX;
    //     continue;
    // }
    //
    // else{
    //     int aaa = generalParams.triangles_in_upperhem[coordInfoVecs.edges2Triangles_1[i]];
    //     int bbb = generalParams.triangles_in_upperhem[coordInfoVecs.edges2Triangles_2[i]];
    //     if (aaa == 1 && bbb == 1){
    //         generalParams.edges_in_upperhem[i] = 1;
    //         generalParams.edges_in_upperhem_list[i] = i;
    //         edges_in_upperhem_COUNT += 1;
    //     }
    //     else if (aaa == 1 || bbb == 1){
    //         generalParams.edges_in_upperhem[i] = 1;
    //         generalParams.edges_in_upperhem_list[i] = -INT_MAX;
    //         edges_in_upperhem_COUNT += 1;
    //     }
    //     else{
    //         generalParams.edges_in_upperhem[i] = -1;
    //         generalParams.edges_in_upperhem_list[i] = -INT_MAX;
    //     }
    // }
    //
    // }
    //// nav commented out the following and added the part above. 03/10/2025
    // for (int i = 0; i < coordInfoVecs.num_edges; i++){
    //		if (coordInfoVecs.edges2Triangles_1[i] >= (INT_MAX-1000) || coordInfoVecs.edges2Triangles_1[i] < 0){
    //			generalParams.edges_in_upperhem[i] = -1;
    //			generalParams.edges_in_upperhem_list[i] = -INT_MAX;
    //			continue;
    //		}
    //		else if (coordInfoVecs.edges2Triangles_2[i] >= (INT_MAX-1000) || coordInfoVecs.edges2Triangles_2[i] < 0){
    //   			generalParams.edges_in_upperhem[i] = -1;
    //   			generalParams.edges_in_upperhem_list[i] = -INT_MAX;
    //   			continue;
    //		}
    //		else{
    //   			int aaa = generalParams.triangles_in_upperhem[coordInfoVecs.edges2Triangles_1[i]];
    //   			int bbb = generalParams.triangles_in_upperhem[coordInfoVecs.edges2Triangles_2[i]];
    //   			if (aaa == 1 && bbb == 1){
    //   				generalParams.edges_in_upperhem[i] = 1;
    //   				generalParams.edges_in_upperhem_list[i] = i;
    //   				edges_in_upperhem_COUNT += 1;
    //   			}
    //   			else if (aaa == 1 || bbb == 1){
    //   				generalParams.edges_in_upperhem[i] = 1;
    //   				generalParams.edges_in_upperhem_list[i] = -INT_MAX;
    //   				edges_in_upperhem_COUNT += 1;
    //   			}
    //   			else{
    //   				generalParams.edges_in_upperhem[i] = -1;
    //   				generalParams.edges_in_upperhem_list[i] = -INT_MAX;
    //   			}
    //        // std::cout<< "Edges in upperhem = "<<generalParams.edges_in_upperhem[i]<<std::endl;
    //		}
    // }

    std::cout << "INITIAL EDGES IN UPPERHEM = " << edges_in_upperhem_COUNT << std::endl;

    int COUNTING_EDGE = 0;
    for (int y = 0; y < coordInfoVecs.num_edges; y++)
    {
        if (generalParams.edges_in_upperhem_list[y] >= 0)
        {
            COUNTING_EDGE += 1;
        }
        generalParams.edges_in_upperhem_list_length = COUNTING_EDGE;
    }

    /*

    //Find the boundary of the nodes_in_upperhem region
    // Define vectors to store the indices of boundary nodes and edges of the upper hemisphere.
    std::vector<int> boundary_node_list;
    std::vector<int> boundary_edge_list;

    // Find the boundary nodes and edges of the upper hemisphere.
    for (int i = 0; i < coordInfoVecs.num_edges; i++){
            double T1 = coordInfoVecs.edges2Triangles_1[i];
            double T2 = coordInfoVecs.edges2Triangles_2[i];
            if (T1 >= (INT_MAX - 1000) || T1 < 0 || T2 >= (INT_MAX-1000) || T2 < 0){
                continue;
            }
       // Have to change this to reflect what the boundary condition is.
       // This condition is checking to if you take a sphere and circle out a region then every triangle in the boundary of that region will have the condition that one triangle is within the region and the other is not.

       //The following is a nav added part 6/4/24:

       double node1 = coordInfoVecs.edges2Nodes_1[i];
        double node2 = coordInfoVecs.edges2Nodes_2[i];

        // Check if the two nodes associated with the edge are the same
        if (node1 == node2) {
            generalParams.boundaries_in_upperhem[i] = 1;

            boundary_node_list.push_back(node1);
            boundary_node_list.push_back(node2);
            boundary_edge_list.push_back(i);
        } else {
            generalParams.boundaries_in_upperhem[i] = -1;
        }
    }*/

    /*		if (generalParams.triangles_in_upperhem[T1] == generalParams.triangles_in_upperhem[T2]){ //1 && generalParams.triangles_in_upperhem[T2] ==1){ //!= 1){ this is what it was before. Nav is changing it for the flat code.
                generalParams.boundaries_in_upperhem[i] = 1;

          double bdry_node1 = coordInfoVecs.edges2Nodes_1[i];
                double bdry_node2 = coordInfoVecs.edges2Nodes_2[i];

          boundary_node_list.push_back(bdry_node1);
                boundary_node_list.push_back(bdry_node2);
                boundary_edge_list.push_back(i);
            }
            /*else if (generalParams.triangles_in_upperhem[T1] ==1 /*!= 1 This is what it was before nav changed it  && /*generalParams.triangles_in_upperhem[T2] == 1){
                generalParams.boundaries_in_upperhem[i] = 1;
                double bdry_node1 = coordInfoVecs.edges2Nodes_1[i];
                double bdry_node2 = coordInfoVecs.edges2Nodes_2[i];
                boundary_node_list.push_back(bdry_node1);
                boundary_node_list.push_back(bdry_node2);
                boundary_edge_list.push_back(i);

            }*/
    // else {
    // generalParams.boundaries_in_upperhem[i] = -1;
    //	}
    //}*///Nav commented the above out to test her own version 6/4/24
    // This is where the boundary part labelled (moved boundary part) was originally. 8/26/24
    // Nav commented this out to restore to original version to make changes once more. Let's see! 8/5/24

    // If conditions on 815 and 830 in the original code need to be changed to reflect boundary condition ie T1==T2 - nav

    // Initialize the generalParams.edge_to_ljparticle vector to store the connection between an edge and LJ particle (nucleus particle).
    // for (int i = 0; i < coordInfoVecs.num_edges; i++){
    //		generalParams.edge_to_ljparticle.push_back(-1);
    //};

    /////////////////////////////////////////////////////////////////////
    ////////////// END OF IDENTIFYING REG. WITH DIFF. MECH PROP /////////
    /////////////////////////////////////////////////////////////////////

    // Part 9

    // Compute the initial volume of the system.
    ComputeVolume(
        generalParams,
        coordInfoVecs,
        linearSpringInfoVecs,
        ljInfoVecs);
    double initial_volume;

    std::cout << "Initial volume = " << initial_volume << std::endl;

    //////////////////////////////////////////////////////////////////////////////////////
    /////////////////////////////////////////////////////////////////////////////////////////////
    ///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    ////////////////////////////// START OF ACTUAL SIMULATION /////////////////////////////////////////////////////////////////
    /////////////////////////////////////////////////////////////////////////////////////////////
    //////////////////////////////////////////////////////////////////////////////////////

    /* Build the initial gradient weakend scale */

    // Initialize variables for gradient weakening.
    dtb = 0.0;                         // dtb := distance to boundary
    generalParams.septin_ring_z = 0.0; // was 0.0, nav changed it to test 8/5/24
    generalParams.boundary_z = 0.0;

    // Loop through all boundary nodes to calculate the distance to the cell tip node.
    for (int k = 0; k < boundary_node_list.size(); k++)
    {
        double n1 = boundary_node_list[k];
        double dist_x = coordInfoVecs.nodeLocX[max_height_index] - coordInfoVecs.nodeLocX[n1];
        double dist_y = coordInfoVecs.nodeLocY[max_height_index] - coordInfoVecs.nodeLocY[n1];
        double dist_z = coordInfoVecs.nodeLocZ[max_height_index] - coordInfoVecs.nodeLocZ[n1];
        double temp_dist = sqrt(dist_x * dist_x + dist_y * dist_y + dist_z * dist_z);
        generalParams.septin_ring_z += coordInfoVecs.nodeLocZ[n1];
        if (temp_dist >= dtb)
        {
            dtb = temp_dist;
            /* "dtb" will be used to identify where the septin ring is located, and used to determine the Hill coefficient*/
        }
    }
    std::cout << "dtb = " << dtb << std::endl;

    // dtb will be only calculated once so we can effectively keep the Hill eqn curve consistent with only horizontal shift
    dtb_max = dtb + (generalParams.Rmin); // Calculate dtb_max, which is dtb plus the equilibrium length of an edge (Rmin).

    std::cout << "initial distance between cell tip and the boundary of weakened area = " << dtb << std::endl;
    std::cout << "Notice that here, the distance from the tip to the boundary is slightly extended by half of the equilibrium length of an edge" << std::endl;

    // Calculate the hill equation constant K using dtb and dtb_max, and set the hill (equation) coefficient.
    generalParams.hilleqnconst = (dtb * dtb_scaler) / dtb_max;
    generalParams.hilleqnpow = targetHillEqnPow;

    if (generalParams.SCALE_TYPE == 4)
    {
        std::cout << "hill equation constant K = " << generalParams.hilleqnconst << std::endl;
        std::cout << "hill (equation) coefficient = " << generalParams.hilleqnpow << std::endl;
    }
    // NOTE: IN THIS SIMULATION, THE LOCATION WHERE 50% WEAKENING IS EXPERIENCED IS LOCATED SLIGHTLY AWAY FROM THE SEPTIN RING. THIS IS DUE TO THE FACT THAT IN ISOTROPIC CASE, SEPTIN RING LOCATION MUST BE SUFFICIENTLY WEAKENED TO INDUCE BUDDING.

    // Transfer host vectors to device memory and perform gradient weakening update.
    utilities_ptr->transferDtoH(generalParams, coordInfoVecs, build_ptr->hostSetInfoVecs);
    utilities_ptr->gradient_weakening_update_host_vecs(sigma,
                                                       coordInfoVecs.nodeLocX[max_height_index],
                                                       coordInfoVecs.nodeLocY[max_height_index],
                                                       coordInfoVecs.nodeLocZ[max_height_index],
                                                       dtb,
                                                       dtb_max,
                                                       generalParams,
                                                       coordInfoVecs,
                                                       build_ptr->hostSetInfoVecs);

    // Calculate the boundary elements for each node.
    for (int u = 0; u < generalParams.maxNodeCount; u++)
    {
        int BETA = utilities_ptr->nodes2Triangles_host_vecs(
            u,
            build_ptr->hostSetInfoVecs,
            coordInfoVecs,
            generalParams,
            auxVecs);
    }

    // Transfer updated host vectors back to device memory.
    utilities_ptr->transferHtoD(generalParams, coordInfoVecs, build_ptr->hostSetInfoVecs);

    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    /////////////////////////////////// DSP EVERSION SIMULATION LOOP ///////////////////////////////////////////////////////
    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

    std::cout << "STARTING THE EVERSION ACTUAL SIMULATION" << std::endl;

    // Print VTK file for visualization.

    bool runSim_Eversion = true;

    // EVERSION SIMULATION LOOP
    while (runSim_Eversion == true)
    {
        double current_time = 0.0;
        int translate_counter = 0;

        // Simulate until the specified relaxation time is reached.
        while (current_time < relax_max_steps_before_strain_tensor_and_gradient_descent * (Max_Runtime))
        {

            translate_counter += 1;

            // Solve forces and update positions of nodes
            Solve_Forces(); // check if the force modules take in individual rest lengths or just a single one (Rmin) Also remove Rmin from the code.
            AdvancePositions(coordInfoVecs,
                             generalParams,
                             domainParams);

            // Calculate the new energy of the system.
            new_total_energy = linearSpringInfoVecs.linear_spring_energy +
                               areaTriangleInfoVecs.area_triangle_energy +
                               bendingTriangleInfoVecs.bending_triangle_energy;

            old_total_energy = new_total_energy;
            current_time += generalParams.dt;

        } // end of time loop

        // Print simulation results for "steady state" initial condition before strain application and gradient relaxation.
        std::cout << "Time used for 'steady state' initial condition before strain application and gradient relaxation = " << current_time << std::endl;
        std::cout << "Current total energy (before strain application and gradient relaxation) = " << new_total_energy << std::endl;
        std::cout << "LINEAR ENERGY = " << linearSpringInfoVecs.linear_spring_energy << std::endl;
        std::cout << "BEND ENERGY = " << bendingTriangleInfoVecs.bending_triangle_energy << std::endl;
        std::cout << "AREA ENERGY = " << areaTriangleInfoVecs.area_triangle_energy << std::endl;
        std::cout << "True current total volume (before strain tensor and gradient relaxation) = " << generalParams.true_current_total_volume << std::endl;
        std::cout << "Current KBT = " << generalParams.kT << std::endl;

        if (std::isnan(new_total_energy))
        {
            std::cout << "Total energy is NaN. Exit code (-1) in System.cu main function." << std::endl;
            exit(-1);
        }

        // Calculate area of apical and basal layers and print them out.

        double current_apical_area = 0.0;
        // double current_basal_area = 0.0; basal layer can be done after apical layer.

        // check total number of triangles and then calculate area.

        std::cout << "INT_MAX = " << INT_MAX << std::endl;

        for (int k = 0; k < coordInfoVecs.num_triangles; k++)
        {

            if (coordInfoVecs.triangles2Nodes_1[k] >= (INT_MAX - 1000.0) || coordInfoVecs.triangles2Nodes_1[k] <= (-INT_MAX + 1000.0) ||
                coordInfoVecs.triangles2Nodes_2[k] >= (INT_MAX - 1000.0) || coordInfoVecs.triangles2Nodes_2[k] <= (-INT_MAX + 1000.0) ||
                coordInfoVecs.triangles2Nodes_3[k] >= (INT_MAX - 1000.0) || coordInfoVecs.triangles2Nodes_3[k] <= (-INT_MAX + 1000.0))
            {
                continue;
            }
            else
            {
                if (generalParams.triangles_in_upperhem[k] == 1)
                {
                    double r1x = coordInfoVecs.nodeLocX[coordInfoVecs.triangles2Nodes_1[k]];
                    double r1y = coordInfoVecs.nodeLocY[coordInfoVecs.triangles2Nodes_1[k]];
                    double r1z = coordInfoVecs.nodeLocZ[coordInfoVecs.triangles2Nodes_1[k]];
                    double r2x = coordInfoVecs.nodeLocX[coordInfoVecs.triangles2Nodes_2[k]];
                    double r2y = coordInfoVecs.nodeLocY[coordInfoVecs.triangles2Nodes_2[k]];
                    double r2z = coordInfoVecs.nodeLocZ[coordInfoVecs.triangles2Nodes_2[k]];
                    double r3x = coordInfoVecs.nodeLocX[coordInfoVecs.triangles2Nodes_3[k]];
                    double r3y = coordInfoVecs.nodeLocY[coordInfoVecs.triangles2Nodes_3[k]];
                    double r3z = coordInfoVecs.nodeLocZ[coordInfoVecs.triangles2Nodes_3[k]];
                    double norm_r1r2 = sqrt((r2x - r1x) * (r2x - r1x) + (r2y - r1y) * (r2y - r1y) + (r2z - r1z) * (r2z - r1z));
                    double norm_r2r3 = sqrt((r3x - r2x) * (r3x - r2x) + (r3y - r2y) * (r3y - r2y) + (r3z - r2z) * (r3z - r2z));
                    double norm_r3r1 = sqrt((r3x - r1x) * (r3x - r1x) + (r3y - r1y) * (r3y - r1y) + (r3z - r1z) * (r3z - r1z));
                    //std::cout << "norm_r1r2 = " << norm_r1r2 << std::endl;
                    //std::cout << "norm_r2r3 = " << norm_r2r3 << std::endl;
                    //std::cout << "norm_r3r1 = " << norm_r3r1 << std::endl;
                    double s = (norm_r1r2 + norm_r2r3 + norm_r3r1) / 2.0;
                    double area = sqrt(s * (s - norm_r1r2) * (s - norm_r2r3) * (s - norm_r3r1));
                    current_apical_area += area;
                }
            }
        }

        // Below is your initial apical surface area.
        double Initial_Apical_Area = current_apical_area;
        std::cout << "Initial apical surface area (before growth and edge swaps) = " << Initial_Apical_Area << std::endl;

        // Spring constants for volume interactions. At the moment I dont think these are needed so I'm commenting it out. We shall see if they're used in other computations in different places.
        // generalParams.volume_spring_constant = 0.2;
        // std::cout<<"Spring constant for surface normal expansion (volume spring constant) = "<<generalParams.volume_spring_constant<<std::endl;

        if (generalParams.SCALE_TYPE == 4)
        {
            // Set scaling factors for different mechanical properties in the hill function. It's 1 if you want the top and bottom to have the same mechanical properties. Otherwise change them. Originally
            // used for the bud vs mother cell.
            generalParams.maxSpringScaler_linear = 1.0;
            generalParams.maxSpringScaler_area = 1.0;
            generalParams.maxSpringScaler_bend = 1.0;
        }

        std::cout << "maxSpringScaler_linear (not 1.0 if we want max linear spring in the hill function scaling not equal to mother cell) = " << generalParams.maxSpringScaler_linear << std::endl;
        std::cout << "maxSpringScaler_area (not 1.0 if we want max area spring in the hill function scaling not equal to mother cell) = " << generalParams.maxSpringScaler_area << std::endl;
        std::cout << "maxSpringScaler_bend (not 1.0 if we want max bend spring in the hill function scaling not equal to mother cell) = " << generalParams.maxSpringScaler_bend << std::endl;

        double scale_linear = linearSpringInfoVecs.spring_constant * 0.75;
        double scale_bend = bendingTriangleInfoVecs.spring_constant * 0.135;
        double scale_area = areaTriangleInfoVecs.spring_constant * 0.75;

        std::cout << "weakened region linear = " << scale_linear << std::endl;
        std::cout << "weakened region bend = " << scale_bend << std::endl;
        std::cout << "weakened region area = " << scale_area << std::endl;

        // Update the weakened spring constants.

        linearSpringInfoVecs.spring_constant_weak = scale_linear;
        bendingTriangleInfoVecs.spring_constant_weak = scale_bend;
        areaTriangleInfoVecs.spring_constant_weak = scale_area;

        // Initial volume of the system.
        initial_volume = generalParams.true_current_total_volume;
        generalParams.eq_total_volume = generalParams.true_current_total_volume * VOLUME_FACTOR; // This is for setting different equilibrium volume to mimic growth or shirnkage.
        std::cout << "true current total volume = " << generalParams.true_current_total_volume << std::endl;
        std::cout << "eq total volume = " << generalParams.eq_total_volume << std::endl;

        // Print VTK file for visualization.
        // storage->print_VTK_File();

        // Weakened start for simulating perhaps some chemical properties? Undetermined for now.
        int gradient_descent_iteration = 2; // this number needs to be changed.
        bool WEAKENED_STRAT = false;
        bool GRADIENT_DESCENT_ALGORITHM_TRIGGERED;
        bool NeedToRebuildDifStructAfterGradientDescent = false;
        int number_of_simulation_step = 0;

        // Main simulation loop.
        while (initial_kT > 0)
        {
            if (gradient_descent_iteration >= NKBT)
            {
                runSim_Eversion = false;
                initial_kT = -1;
                gradient_descent_iteration+=1;
                break;
            }

            // Now relax mesh (initial relaxation before strain tensor application)

            current_time = 0.0;
            translate_counter = 0;
            double VOLUME_RATIO = generalParams.true_current_total_volume / generalParams.eq_total_volume; // this is for volume measurements.

            GRADIENT_DESCENT_ALGORITHM_TRIGGERED = true;
            bool end_of_relaxation = false;

            // The following loop is set to calclulate the total number of simulation steps.
            while (current_time < Max_Runtime)
            {
                number_of_simulation_step += 1;
                if (Max_Runtime <= 0.0)
                {
                    std::cout << "Max_Runtime is set to be 0 or negative! " << std::endl;
                    break;
                }

                Solve_Forces();

                AdvancePositions(
                    coordInfoVecs,
                    generalParams,
                    domainParams);

                // Calculate the new total energy of the system.
                new_total_energy = linearSpringInfoVecs.linear_spring_energy +
                                   areaTriangleInfoVecs.area_triangle_energy +
                                   bendingTriangleInfoVecs.bending_triangle_energy; // +0.5*energy_rep;

                energy_gradient = sqrt((new_total_energy - old_total_energy) * (new_total_energy - old_total_energy)) / old_total_energy;
                old_total_energy = new_total_energy;
                current_time += generalParams.dt;

// nav commented out below because we are no longer recentering the mesh. 
//                // Computing center of just the apical layer.
//                if (translate_counter % translate_frequency == 0)
//                {
//
//                    // Compute the new center of the system.
//                    newcenterX = 0.0;
//                    newcenterY = 0.0;
//                    newcenterZ = 0.0;
//
//                    for (int i = 0; i < generalParams.maxNodeCount; i++)
//                    {
//                        if (generalParams.nodes_in_upperhem[i] == 1)
//                        {
//                            newcenterX += coordInfoVecs.nodeLocX[i];
//                            newcenterY += coordInfoVecs.nodeLocY[i];
//                            newcenterZ += coordInfoVecs.nodeLocZ[i];
//                        }
//                    }
//                    newcenterX = newcenterX / generalParams.maxNodeCount;
//                    newcenterY = newcenterY / generalParams.maxNodeCount;
//                    newcenterZ = newcenterZ / generalParams.maxNodeCount;
//
//                    // Compute the displacement vector.
//                    displacementX = newcenterX - generalParams.centerX;
//                    displacementY = newcenterY - generalParams.centerY;
//                    displacementZ = newcenterZ - generalParams.centerZ;
//
//                    // Update the positions of all nodes and LJ particles.
//                    for (int i = 0; i < generalParams.maxNodeCount; i++)
//                    {
//                        coordInfoVecs.nodeLocX[i] += -displacementX;
//                        coordInfoVecs.nodeLocY[i] += -displacementY;
//                        coordInfoVecs.nodeLocZ[i] += -displacementZ;
//                    }
//                    // you can use this part to compute the positions of the LJ_particles too if you choose to include that energy.
//
//                    // Recompute the volume of the system after the translation
//                    ComputeVolume(
//                        generalParams,
//                        coordInfoVecs,
//                        linearSpringInfoVecs,
//                        ljInfoVecs);
//                }

            } // end of main simulation loop eversion

            // Print VTK file for visualization.
            storage->print_VTK_File();

            end_of_relaxation = true;
            double current_center_x = 0.0;
            double current_center_y = 0.0;
            double bdry_to_tip_height = 0.0; // this here is now for the total height of the apical layer of the tissue.

            if (generalParams.SCALE_TYPE == 4)
            {
                max_height = -10000.0;
                for (int k = 0; k < generalParams.maxNodeCount; k++)
                {
                    if (generalParams.nodes_in_upperhem[k] == 1)
                    {
                        current_center_x += coordInfoVecs.nodeLocX[k];
                        current_center_y += coordInfoVecs.nodeLocX[k];
                    }

                    if (coordInfoVecs.nodeLocZ[k] >= max_height)
                    {
                        max_height = coordInfoVecs.nodeLocZ[k];
                        max_height_index = k;
                    }
                }
                current_center_x = current_center_x / generalParams.maxNodeCount;
                current_center_y = current_center_y / generalParams.maxNodeCount;

//                if (generalParams.nonuniform_wall_weakening_bend == false && generalParams.nonuniform_wall_weakening_linear == false && generalParams.nonuniform_wall_weakening_area == false)
//                {
//                    bdry_to_tip_height = 0.0;
//
//                    for (int y = 0; y < boundary_edge_list.size(); y++)
//                    {
//                        double edge_mdpt_z = (coordInfoVecs.nodeLocZ[coordInfoVecs.edges2Nodes_1[boundary_edge_list[y]]] +
//                                              coordInfoVecs.nodeLocZ[coordInfoVecs.edges2Nodes_2[boundary_edge_list[y]]]) /
//                                             2.0;
//                        bdry_to_tip_height += sqrt(pow(coordInfoVecs.nodeLocZ[max_height_index] - edge_mdpt_z, 2.0));
//                    }
//                    bdry_to_tip_height = bdry_to_tip_height / boundary_edge_list.size();
//
//                    for (int y = 0; y < coordInfoVecs.num_edges; y++)
//                    {
//                        if (generalParams.edges_in_upperhem_list[y] >= 0 &&
//                            generalParams.edges_in_upperhem_list[y] != INT_MAX &&
//                            generalParams.edges_in_upperhem_list[y] <= (INT_MAX - 1000) &&
//                            generalParams.edges_in_upperhem_list[y] >= (-INT_MAX + 1000) &&
//                            generalParams.boundaries_in_upperhem[y] != 1)
//                        {
//                            if (coordInfoVecs.edges2Nodes_1[y] < 0 || coordInfoVecs.edges2Nodes_1[y] >= (INT_MAX - 1000))
//                            {
//                                continue;
//                            }
//                            else if (coordInfoVecs.edges2Nodes_2[y] < 0 || coordInfoVecs.edges2Nodes_2[y] >= (INT_MAX - 1000))
//                            {
//                                continue;
//                            }
//                            double edge_mdpt_z = (coordInfoVecs.nodeLocZ[coordInfoVecs.edges2Nodes_1[y]] + coordInfoVecs.nodeLocZ[coordInfoVecs.edges2Nodes_2[y]]) / 2.0;
//                            double current_edge_to_tip_height = sqrt(pow(coordInfoVecs.nodeLocZ[max_height_index] - edge_mdpt_z, 2.0));
//                            // generalParams.Rmin = linearSpringInfoVecs.edge_initial_length[y];
//
//                            if (bdry_to_tip_height >= (generalParams.Rmin * generalParams.ratio_for_HillFunctionStiffness))
//                            {
//                                if (generalParams.nonuniform_wall_weakening_bend == false && generalParams.nonuniform_wall_weakening_area == false && generalParams.nonuniform_wall_weakening_linear == false && display_token == true)
//                                {
//                                    std::cout << "generalParams.nonuniform_wall_weakening_XXXX is set to be true from this point" << std::endl;
//                                    display_token = false;
//                                }
//                                generalParams.nonuniform_wall_weakening_bend = true;
//                                generalParams.nonuniform_wall_weakening_area = true;
//                                generalParams.nonuniform_wall_weakening_linear = true;
//                            }
//                            else if (bdry_to_tip_height < (generalParams.Rmin * generalParams.ratio_for_HillFunctionStiffness))
//                            {
//                                generalParams.nonuniform_wall_weakening_bend = false;
//                                generalParams.nonuniform_wall_weakening_linear = false;
//                                generalParams.nonuniform_wall_weakening_area = false;
//                            }
//                        }
//                    }
//                }
            }

            //// Here is where you would insert chemical diffusion. We dont have that yet so no need to worry about it.

            // Check if relaxation loop has ended.
            if (end_of_relaxation == true)
            {

                // Compute volume of system
                ComputeVolume(generalParams,
                              coordInfoVecs,
                              linearSpringInfoVecs,
                              ljInfoVecs);

                // Check if mesh overcompression is happening for volume related termination.
                if ((generalParams.true_current_total_volume / initial_volume) < 0.6 || (generalParams.true_current_total_volume / initial_volume) >= MAX_VOLUME_RATIO)
                {

                    // Update true_num_edges and store data before terminating the simulation
                    generalParams.true_num_edges = 0;
                    for (int i = 0; i < coordInfoVecs.num_edges; i++)
                    {
                        if (coordInfoVecs.edges2Nodes_1[i] != INT_MAX && coordInfoVecs.edges2Nodes_2[i] != INT_MAX)
                        {
                            generalParams.true_num_edges += 1;
                        }
                    }

                    storage->print_VTK_File();

                    // Print appropriate message based on the termination reason.
                    if (generalParams.true_current_total_volume / initial_volume < 0.6)
                    {
                        std::cout << "Cell over compression 60%" << std::endl;
                    }
                    else if (generalParams.true_current_total_volume / initial_volume >= MAX_VOLUME_RATIO)
                    {
                        std::cout << "Target volume ratio exceeded. Current volume ratio = " << generalParams.true_current_total_volume / initial_volume << std::endl;
                    }

                    // Print relevant simulation statistics
                    std::cout << "Current number of gradient descent iteration performed at volume-related termination = " << gradient_descent_iteration << std::endl;
                    std::cout << "Current number of simulation step at volume-related termination = " << number_of_simulation_step << std::endl;

                    // Termination simulation. Nav commented out for testing without volume thresholds 8/26/24
                    Max_Runtime = 0.0;
                    runSim_Eversion = false;
                    initial_kT = -1;
                    break;
                }
                else {// This is just added in for the sake of it. Remove it later - nav 05-13-25 the yeast budding model chose a volume related termination, you should have a curvature related termination
                    Max_Runtime = 0.0;
                    runSim_Eversion = false;
                    initial_kT = -1;
                    break;
                }
            }

            // Calculate current apical surface area
            double current_apical_area = 0.0;

            for (int k = 0; k < coordInfoVecs.num_triangles; k++)
            {

                // Check if triangle data is valid
                if (coordInfoVecs.triangles2Nodes_1[k] >= (INT_MAX - 1000.0) || coordInfoVecs.triangles2Nodes_1[k] <= (-INT_MAX + 1000.0) ||
                    coordInfoVecs.triangles2Nodes_2[k] >= (INT_MAX - 1000.0) || coordInfoVecs.triangles2Nodes_2[k] <= (-INT_MAX + 1000.0) ||
                    coordInfoVecs.triangles2Nodes_3[k] >= (INT_MAX - 1000.0) || coordInfoVecs.triangles2Nodes_3[k] <= (-INT_MAX + 1000.0))
                {
                    continue;
                }
                else
                {
                    // Check if triangle is in the upper hemisphere
                    if (generalParams.triangles_in_upperhem[k] == 1)
                    {

                        // Calculate triangle area and update current_bud_area
                        double r1x = coordInfoVecs.nodeLocX[coordInfoVecs.triangles2Nodes_1[k]];
                        double r1y = coordInfoVecs.nodeLocY[coordInfoVecs.triangles2Nodes_1[k]];
                        double r1z = coordInfoVecs.nodeLocZ[coordInfoVecs.triangles2Nodes_1[k]];
                        double r2x = coordInfoVecs.nodeLocX[coordInfoVecs.triangles2Nodes_2[k]];
                        double r2y = coordInfoVecs.nodeLocY[coordInfoVecs.triangles2Nodes_2[k]];
                        double r2z = coordInfoVecs.nodeLocZ[coordInfoVecs.triangles2Nodes_2[k]];
                        double r3x = coordInfoVecs.nodeLocX[coordInfoVecs.triangles2Nodes_3[k]];
                        double r3y = coordInfoVecs.nodeLocY[coordInfoVecs.triangles2Nodes_3[k]];
                        double r3z = coordInfoVecs.nodeLocZ[coordInfoVecs.triangles2Nodes_3[k]];
                        double norm_r1r2 = sqrt((r2x - r1x) * (r2x - r1x) + (r2y - r1y) * (r2y - r1y) + (r2z - r1z) * (r2z - r1z));
                        double norm_r2r3 = sqrt((r3x - r2x) * (r3x - r2x) + (r3y - r2y) * (r3y - r2y) + (r3z - r2z) * (r3z - r2z));
                        double norm_r3r1 = sqrt((r3x - r1x) * (r3x - r1x) + (r3y - r1y) * (r3y - r1y) + (r3z - r1z) * (r3z - r1z));
                        double s = (norm_r1r2 + norm_r2r3 + norm_r3r1) / 2.0;
                        double area = sqrt(s * (s - norm_r1r2) * (s - norm_r2r3) * (s - norm_r3r1));
                        current_apical_area += area;
                    }
                }
            }

            // Check if bud surface area ratio conditions are met for termination
            if (current_apical_area / Initial_Apical_Area >= MAX_APICAL_AREA_RATIO)
            {

                // Print relevant message and statistics
                std::cout << "Target apical surface area ratio exceeded. Current apical surface area ratio = " << current_apical_area / Initial_Apical_Area << std::endl;
                std::cout << "Current number of relaxation iterations performed at area-related termination = " << gradient_descent_iteration << std::endl;
                std::cout << "Current number of simulation step at area-related termination = " << number_of_simulation_step << std::endl;

                // Update true_num_edges and store data before terminating the simulation
                generalParams.true_num_edges = 0;
                for (int i = 0; i < coordInfoVecs.num_edges; i++)
                {
                    if (coordInfoVecs.edges2Nodes_1[i] != INT_MAX && coordInfoVecs.edges2Nodes_2[i] != INT_MAX)
                    {
                        generalParams.true_num_edges += 1;
                    }
                }
                storage->print_VTK_File();

                // Terminate the simulation
                Max_Runtime = 0.0;
                runSim = false;
                initial_kT = -1;
                break;
            }
        }
    
    // here recenter the mesh first.


// nav commented out because we are no longer recentering the mesh. 
//    // Compute the new center of the system.
    newcenterX = 0.0;
    newcenterY = 0.0;
    newcenterZ = 0.0;

    for (int i = 0; i < generalParams.maxNodeCount; i++)
    {
        if (generalParams.nodes_in_upperhem[i] == 1)
        {
            newcenterX += coordInfoVecs.nodeLocX[i];
            newcenterY += coordInfoVecs.nodeLocY[i];
            newcenterZ += coordInfoVecs.nodeLocZ[i];
        }
    }
    newcenterX = newcenterX / generalParams.maxNodeCount;
    newcenterY = newcenterY / generalParams.maxNodeCount;
    newcenterZ = newcenterZ / generalParams.maxNodeCount;
    
    generalParams.centerX = newcenterX; //sumX / nNodes;
        generalParams.centerY =newcenterY;// sumY / nNodes;
        generalParams.centerZ = newcenterZ;

//    // Compute the displacement vector.
//    displacementX = newcenterX - generalParams.centerX;
//    displacementY = newcenterY - generalParams.centerY;
//    displacementZ = newcenterZ - generalParams.centerZ;
//
//    // Update the positions of all nodes and LJ particles.
//    for (int i = 0; i < generalParams.maxNodeCount; i++)
//    {
//        coordInfoVecs.nodeLocX[i] += -displacementX;
//        coordInfoVecs.nodeLocY[i] += -displacementY;
//        coordInfoVecs.nodeLocZ[i] += -displacementZ;
//    }
    storage->print_VTK_File();

    //// commenting out the following for some tests.  Nav 05/04/2025
    std::cout << "number of simulation steps = " << generalParams.Tf << std::endl;


    
        double sumX = 0.0, sumY = 0.0, sumZ = 0.0;
        int nNodes = coordInfoVecs.nodeLocX.size();
        //int layerflag = 0; // this is the layer that will be excluded. 
        for (int i = 0; i < nNodes; i++)
        {  
            //if (layerflag == 1 && i >= nNodes/2) break;
            //if (layerflag == -1 && i < nNodes/2) continue;
                sumX += coordInfoVecs.nodeLocX[i];
                sumY += coordInfoVecs.nodeLocY[i];
                sumZ += coordInfoVecs.nodeLocZ[i];
        }
        generalParams.centerX = sumX / nNodes;
        generalParams.centerY = sumY / nNodes;
        generalParams.centerZ = sumZ / nNodes;
        
        //std::cout<<"center at "<<layerflag<<" = ("<<generalParams.centerX<<", "<<generalParams.centerY<<", "<<generalParams.centerZ<<") "<< std::endl;

    
    std::cout<<"Tf = "<< generalParams.Tf<<std::endl;
//    
//    for (int i = 0; i<coordInfoVecs.num_edges; i++){
//        if (i >= 10) break;
//        std::cout<< "initial rest length before strain tensor, at edge = "<< i << " = " <<linearSpringInfoVecs.edge_rest_length[i]<<std::endl;
//        
//    }

    //double tol = 1e-8;
    double iter =100;// 1/(generalParams.dt*generalParams.tol);
    
    // if you leave ^ this to be 100 it'll take wayy longer for your simulations. Make it at least 10000.
    
    ///////////////////////////////////////////////////////////////////
    // Okay so let's start the simulation loop for the strain tensor //
    ///////////////////////////////////////////////////////////////////
    
    int stages = generalParams.Tf;
    //generalParams.tol = 1e-4;
    
 //   int layerflag = 0;
    
    LambdaField lambda;
    
//    for (int stage = 0; stage<=stages; stage++){
    
        double frac = 1.0;///generalParams.Tf; // this fraction should remain what it is. Dont change it. 
        
      sumX = 0.0, sumY = 0.0, sumZ = 0.0;
        
        
         nNodes = coordInfoVecs.nodeLocX.size();
        int layerflag = 0; // this is the layer that will be excluded. 
        for (int i = 0; i < nNodes; i++)
        {  
            //if (layerflag == 1 && i >= nNodes/2) break;
            //if (layerflag == -1 && i < nNodes/2) continue;
                sumX += coordInfoVecs.nodeLocX[i];
                sumY += coordInfoVecs.nodeLocY[i];
                sumZ += coordInfoVecs.nodeLocZ[i];
        }
        generalParams.centerX = sumX / nNodes;
        generalParams.centerY = sumY / nNodes;
        generalParams.centerZ = sumZ / nNodes;
    
        StrainTensorGPU::buildVertexLambda(generalParams, coordInfoVecs, lambda, frac);
        
          //std::cout << "\n[Stage " << stage << "] spring lengths BEFORE strain:\n";
//    for (int e = 0; e < coordInfoVecs.num_edges; ++e) {
//        if (generalParams.edges_in_upperhem[e] == 0) continue;  // skip vertical
//        double L0 = linearSpringInfoVecs.edge_initial_length[e];
//        std::cout
//            << "  edge[" << e << "]: initial_length = "
//            << L0 << "\n";
//    }
        StrainTensorGPU::updateEdgeRestLengths(coordInfoVecs,generalParams, lambda, linearSpringInfoVecs, layerflag);
        
        //Solve_Forces();
        //AdvancePositions(coordInfoVecs, generalParams, domainParams);
        
        
        // === PRINT AFTER ===
   // std::cout << "[Stage " << stage << "] spring lengths AFTER strain:\n";
//    for (int e = 0; e < coordInfoVecs.num_edges; ++e) {
//        if (generalParams.edges_in_upperhem[e] == 0) continue;  // skip vertical
//        double Lstar = linearSpringInfoVecs.edge_final_length[e];
//        std::cout
//            << "  edge[" << e << "]: strained_length = "
//            << Lstar << "\n";
//    }
generalParams.tol = 1e-4;
              
              ///////////////////////////////////////////////////////////////////////  
        // resize edge_rest_length to match the size of the steps between edge_initial and edge_final 
        // loop for changing the rest lengths between stages. Here the spring length is broken down into steps and passed into the force kernels. 
        for (int step = 0; step<= iter; step++){
        
            for(int i = 0; i<coordInfoVecs.num_edges; i++){ // this can be done from inside the strain tensor file. 
                
                double dl = (linearSpringInfoVecs.edge_final_length[i] - linearSpringInfoVecs.edge_initial_length[i])/iter;
                linearSpringInfoVecs.edge_rest_length[i] += dl;
            }
            
           // double avg_mvt = 999999.0;
            // generalParams.dx = 0.0;
            
           // std::cout<< "Starting relaxation loop."<<std::endl;
            //int k = 0;
            
            int k = relaxUntilConverged(*this);
            
            new_total_energy = linearSpringInfoVecs.linear_spring_energy;
                                   //areaTriangleInfoVecs.area_triangle_energy +
                                   //bendingTriangleInfoVecs.bending_triangle_energy; 
                                   
            std::cout<<"Energy = "<<new_total_energy<<", iterations = "<<k<<", = Average Movement = "<<generalParams.dx<<std::endl;
                        
           // std::cout<<"Exiting relaxation loop. Average movement = "<<generalParams.dx<<", and k = "<< k<<" at step = "<<step<<std::endl;//" and stage = "<<stage<<std::endl;     
            if (step > 0 && step % 2 == 0) {
                
                storage->print_VTK_File(); // this is just for now. To get better simulation results we're gonna view them more frequently later. 
            }

          }
//////////////////////////////////////
//            
           // do{
             //   generalParams.dx = 0.0; 
                
//                double x1[coordInfoVecs.nodeLocX.size()];
//                double y1[coordInfoVecs.nodeLocX.size()];
//                double z1[coordInfoVecs.nodeLocX.size()];
//                for (int n = 0; n<coordInfoVecs.nodeLocX.size(); n++){ // this can be handled from inside the nodeAdvance.cu
//                    x1[n] = coordInfoVecs.nodeLocX[n];
//                    y1[n] = coordInfoVecs.nodeLocY[n];
//                    z1[n] = coordInfoVecs.nodeLocZ[n];
//                } 
                
                //avg_mvt = 0.0;
               // Solve_Forces();
                // AdvancePositions(coordInfoVecs, generalParams, domainParams);
                
                
//                double x2[coordInfoVecs.nodeLocX.size()];
//                double y2[coordInfoVecs.nodeLocX.size()];
//                double z2[coordInfoVecs.nodeLocX.size()];
//                
//                for (int n = 0; n<coordInfoVecs.nodeLocX.size(); n++){ // same as above. 
//                    x2[n] = coordInfoVecs.nodeLocX[n];
//                    y2[n] = coordInfoVecs.nodeLocY[n];
//                    z2[n] = coordInfoVecs.nodeLocZ[n];
//                    
//                    generalParams.dx+= sqrt((x1[n]-x2[n])*(x1[n]-x2[n])+(y1[n]-y2[n])*(y1[n]-y2[n])+(z1[n]-z2[n])*(z1[n]-z2[n]));
//                }
//                
//                
                //avg_mvt=generalParams.dx;   
              //  k++;
                
               // if (k == 4500) break;
            //}while (generalParams.dx>generalParams.tol);
            
            
            //if (step == 100 || step == 200 || step == 300 || step == 400 || step == 500 || step == 600 || step == 700 || step == 800 || step == 900 || step == 1000 || step == 2000 || step == 5000 || step == 7000 || step == 9000 ){ // badly done ik but I'll fix it later. 
            
      //  }
        
        storage->print_VTK_File();
        
  //  }
    
    // strain tensor acts on edge_inital to give edge_final 

}
};




// Function to assign the shared pointer to storage.
void System::assignStorage(std::shared_ptr<Storage> _storage)
{
    storage = _storage;
};

// Function to set the weak pointer to the SystemBuilder.
void System::set_weak_builder(std::weak_ptr<SystemBuilder> _weak_bld_ptr)
{
    weak_bld_ptr = _weak_bld_ptr;
};

// Function to initialize memory for thrust vectors and set coordInfoVecs values from input.
void System::initializeSystem(HostSetInfoVecs & hostSetInfoVecs)
{
    std::cout << "Initializing" << std::endl;

    // Set the max node count, edge count and triangle count.
    generalParams.maxNodeCount = hostSetInfoVecs.nodeLocX.size();
    coordInfoVecs.num_edges = hostSetInfoVecs.edges2Nodes_1.size();
    coordInfoVecs.num_triangles = hostSetInfoVecs.triangles2Nodes_1.size();

    std::cout << "num nodes: " << generalParams.maxNodeCount << std::endl;
    std::cout << "num edges: " << coordInfoVecs.num_edges << std::endl;
    std::cout << "num elems: " << coordInfoVecs.num_triangles << std::endl;
    // Allocate memory for various vectors using preallocated memory size.
    int mem_prealloc = 4;

    // Resize and initialize the following various vectors.
    coordInfoVecs.scaling_per_edge.resize(mem_prealloc * coordInfoVecs.num_edges, 0.0);
    hostSetInfoVecs.scaling_per_edge.resize(coordInfoVecs.scaling_per_edge.size(), 0.0);

    coordInfoVecs.soln_per_triangle.resize(mem_prealloc * coordInfoVecs.num_triangles, INT_MAX);
    coordInfoVecs.b_per_triangle.resize(mem_prealloc * coordInfoVecs.num_triangles, INT_MAX);

    coordInfoVecs.isNodeFixed.resize(mem_prealloc * hostSetInfoVecs.nodeLocX.size(), false);
    coordInfoVecs.prevNodeLocX.resize(mem_prealloc * hostSetInfoVecs.nodeLocX.size());
    coordInfoVecs.prevNodeLocY.resize(mem_prealloc * hostSetInfoVecs.nodeLocX.size());
    coordInfoVecs.prevNodeLocZ.resize(mem_prealloc * hostSetInfoVecs.nodeLocX.size());

    coordInfoVecs.prevNodeForceX.resize(mem_prealloc * hostSetInfoVecs.nodeLocX.size());
    coordInfoVecs.prevNodeForceY.resize(mem_prealloc * hostSetInfoVecs.nodeLocX.size());
    coordInfoVecs.prevNodeForceZ.resize(mem_prealloc * hostSetInfoVecs.nodeLocX.size());

    coordInfoVecs.nodeLocX.resize(mem_prealloc * hostSetInfoVecs.nodeLocX.size());
    coordInfoVecs.nodeLocY.resize(mem_prealloc * hostSetInfoVecs.nodeLocX.size());
    coordInfoVecs.nodeLocZ.resize(mem_prealloc * hostSetInfoVecs.nodeLocX.size());

    // coordInfoVecs.nodeVelX.resize(mem_prealloc*hostSetInfoVecs.nodeVelX.size(), 0.0);
    // coordInfoVecs.nodeVelY.resize(mem_prealloc*hostSetInfoVecs.nodeVelY.size(), 0.0);
    // coordInfoVecs.nodeVelZ.resize(mem_prealloc*hostSetInfoVecs.nodeVelZ.size(), 0.0);

    coordInfoVecs.nodeForceX.resize(mem_prealloc * hostSetInfoVecs.nodeLocX.size(), 0.0);
    coordInfoVecs.nodeForceY.resize(mem_prealloc * hostSetInfoVecs.nodeLocX.size(), 0.0);
    coordInfoVecs.nodeForceZ.resize(mem_prealloc * hostSetInfoVecs.nodeLocX.size(), 0.0);

    coordInfoVecs.triangles2Nodes_1.resize(mem_prealloc * coordInfoVecs.num_triangles);
    coordInfoVecs.triangles2Nodes_2.resize(mem_prealloc * coordInfoVecs.num_triangles);
    coordInfoVecs.triangles2Nodes_3.resize(mem_prealloc * coordInfoVecs.num_triangles);

    coordInfoVecs.triangles2Edges_1.resize(mem_prealloc * coordInfoVecs.num_triangles);
    coordInfoVecs.triangles2Edges_2.resize(mem_prealloc * coordInfoVecs.num_triangles);
    coordInfoVecs.triangles2Edges_3.resize(mem_prealloc * coordInfoVecs.num_triangles);

    coordInfoVecs.triangles2Triangles_1.resize(mem_prealloc * coordInfoVecs.num_triangles, -INT_MAX);
    coordInfoVecs.triangles2Triangles_2.resize(mem_prealloc * coordInfoVecs.num_triangles, -INT_MAX);
    coordInfoVecs.triangles2Triangles_3.resize(mem_prealloc * coordInfoVecs.num_triangles, -INT_MAX);

    hostSetInfoVecs.triangles2Triangles_1.resize(mem_prealloc * coordInfoVecs.num_triangles, -INT_MAX);
    hostSetInfoVecs.triangles2Triangles_2.resize(mem_prealloc * coordInfoVecs.num_triangles, -INT_MAX);
    hostSetInfoVecs.triangles2Triangles_3.resize(mem_prealloc * coordInfoVecs.num_triangles, -INT_MAX);

    coordInfoVecs.edges2Nodes_1.resize(mem_prealloc * coordInfoVecs.num_edges);
    coordInfoVecs.edges2Nodes_2.resize(mem_prealloc * coordInfoVecs.num_edges);

    coordInfoVecs.edges2Triangles_1.resize(mem_prealloc * coordInfoVecs.num_edges);
    coordInfoVecs.edges2Triangles_2.resize(mem_prealloc * coordInfoVecs.num_edges);

    coordInfoVecs.nndata1.resize(mem_prealloc * generalParams.maxNodeCount);
    coordInfoVecs.nndata2.resize(mem_prealloc * generalParams.maxNodeCount);
    coordInfoVecs.nndata3.resize(mem_prealloc * generalParams.maxNodeCount);
    coordInfoVecs.nndata4.resize(mem_prealloc * generalParams.maxNodeCount);
    coordInfoVecs.nndata5.resize(mem_prealloc * generalParams.maxNodeCount);
    coordInfoVecs.nndata6.resize(mem_prealloc * generalParams.maxNodeCount);
    coordInfoVecs.nndata7.resize(mem_prealloc * generalParams.maxNodeCount);
    coordInfoVecs.nndata8.resize(mem_prealloc * generalParams.maxNodeCount);
    coordInfoVecs.nndata9.resize(mem_prealloc * generalParams.maxNodeCount);
    coordInfoVecs.SurfaceNormalX.resize(mem_prealloc * generalParams.maxNodeCount);
    coordInfoVecs.SurfaceNormalY.resize(mem_prealloc * generalParams.maxNodeCount);
    coordInfoVecs.SurfaceNormalZ.resize(mem_prealloc * generalParams.maxNodeCount);

    generalParams.nodes_in_upperhem.resize(mem_prealloc * generalParams.maxNodeCount);
    generalParams.triangles_in_upperhem.resize(mem_prealloc * coordInfoVecs.num_triangles);
    generalParams.edges_in_upperhem.resize(mem_prealloc * coordInfoVecs.num_edges);
    generalParams.edges_in_upperhem_list.resize(mem_prealloc * coordInfoVecs.num_edges);
    generalParams.boundaries_in_upperhem.resize(mem_prealloc * coordInfoVecs.num_edges, -1);
    generalParams.boundaries_in_lowerhem.resize(mem_prealloc * coordInfoVecs.num_edges, -1);

    hostSetInfoVecs.nodes_in_upperhem.resize(generalParams.nodes_in_upperhem.size());
    generalParams.nodes_in_upperhem = hostSetInfoVecs.nodes_in_upperhem;
    hostSetInfoVecs.triangles_in_upperhem.resize(generalParams.triangles_in_upperhem.size());
    hostSetInfoVecs.edges_in_upperhem.resize(generalParams.edges_in_upperhem.size());
    generalParams.edges_in_upperhem = hostSetInfoVecs.edges_in_upperhem;
    hostSetInfoVecs.edges_in_upperhem_list.resize(mem_prealloc * coordInfoVecs.num_edges);
    hostSetInfoVecs.boundaries_in_upperhem.resize(mem_prealloc * coordInfoVecs.num_edges, -1);
    hostSetInfoVecs.boundaries_in_lowerhem.resize(mem_prealloc * coordInfoVecs.num_edges, -1);

    hostSetInfoVecs.nodes2Triangles_1.resize(mem_prealloc * generalParams.maxNodeCount, -INT_MAX);
    hostSetInfoVecs.nodes2Triangles_2.resize(mem_prealloc * generalParams.maxNodeCount, -INT_MAX);
    hostSetInfoVecs.nodes2Triangles_3.resize(mem_prealloc * generalParams.maxNodeCount, -INT_MAX);
    hostSetInfoVecs.nodes2Triangles_4.resize(mem_prealloc * generalParams.maxNodeCount, -INT_MAX);
    hostSetInfoVecs.nodes2Triangles_5.resize(mem_prealloc * generalParams.maxNodeCount, -INT_MAX);
    hostSetInfoVecs.nodes2Triangles_6.resize(mem_prealloc * generalParams.maxNodeCount, -INT_MAX);
    hostSetInfoVecs.nodes2Triangles_7.resize(mem_prealloc * generalParams.maxNodeCount, -INT_MAX);
    hostSetInfoVecs.nodes2Triangles_8.resize(mem_prealloc * generalParams.maxNodeCount, -INT_MAX);
    hostSetInfoVecs.nodes2Triangles_9.resize(mem_prealloc * generalParams.maxNodeCount, -INT_MAX);

    // Part 17

    // Resize vectors to allocate memory for nodes-to-triangles mapping.
    coordInfoVecs.nodes2Triangles_1.resize(mem_prealloc * generalParams.maxNodeCount, -INT_MAX);
    coordInfoVecs.nodes2Triangles_2.resize(mem_prealloc * generalParams.maxNodeCount, -INT_MAX);
    coordInfoVecs.nodes2Triangles_3.resize(mem_prealloc * generalParams.maxNodeCount, -INT_MAX);
    coordInfoVecs.nodes2Triangles_4.resize(mem_prealloc * generalParams.maxNodeCount, -INT_MAX);
    coordInfoVecs.nodes2Triangles_5.resize(mem_prealloc * generalParams.maxNodeCount, -INT_MAX);
    coordInfoVecs.nodes2Triangles_6.resize(mem_prealloc * generalParams.maxNodeCount, -INT_MAX);
    coordInfoVecs.nodes2Triangles_7.resize(mem_prealloc * generalParams.maxNodeCount, -INT_MAX);
    coordInfoVecs.nodes2Triangles_8.resize(mem_prealloc * generalParams.maxNodeCount, -INT_MAX);
    coordInfoVecs.nodes2Triangles_9.resize(mem_prealloc * generalParams.maxNodeCount, -INT_MAX);

    // Copy nodes-to-triangles mapping information from hostSetInfoVecs to coodInfoVecs and others.
    thrust::copy(coordInfoVecs.nodes2Triangles_1.begin(), coordInfoVecs.nodes2Triangles_1.end(), hostSetInfoVecs.nodes2Triangles_1.begin());
    thrust::copy(coordInfoVecs.nodes2Triangles_2.begin(), coordInfoVecs.nodes2Triangles_2.end(), hostSetInfoVecs.nodes2Triangles_2.begin());
    thrust::copy(coordInfoVecs.nodes2Triangles_3.begin(), coordInfoVecs.nodes2Triangles_3.end(), hostSetInfoVecs.nodes2Triangles_3.begin());
    thrust::copy(coordInfoVecs.nodes2Triangles_4.begin(), coordInfoVecs.nodes2Triangles_4.end(), hostSetInfoVecs.nodes2Triangles_4.begin());
    thrust::copy(coordInfoVecs.nodes2Triangles_5.begin(), coordInfoVecs.nodes2Triangles_5.end(), hostSetInfoVecs.nodes2Triangles_5.begin());
    thrust::copy(coordInfoVecs.nodes2Triangles_6.begin(), coordInfoVecs.nodes2Triangles_6.end(), hostSetInfoVecs.nodes2Triangles_6.begin());
    thrust::copy(coordInfoVecs.nodes2Triangles_7.begin(), coordInfoVecs.nodes2Triangles_7.end(), hostSetInfoVecs.nodes2Triangles_7.begin());
    thrust::copy(coordInfoVecs.nodes2Triangles_8.begin(), coordInfoVecs.nodes2Triangles_8.end(), hostSetInfoVecs.nodes2Triangles_8.begin());
    thrust::copy(coordInfoVecs.nodes2Triangles_9.begin(), coordInfoVecs.nodes2Triangles_9.end(), hostSetInfoVecs.nodes2Triangles_9.begin());

    // copy info to GPU
    std::cout << "Copying" << std::endl;
    thrust::copy(hostSetInfoVecs.isNodeFixed.begin(), hostSetInfoVecs.isNodeFixed.end(), coordInfoVecs.isNodeFixed.begin());

    // Print information about fixed nodes in hostSetInfoVecs and coordInfoVecs.
    std::cout << "fixed_node_in_host: " << std::endl;
    for (int k = 0; k < hostSetInfoVecs.isNodeFixed.size(); k++)
    {
    }
    std::cout << "end_of_fixed_node_host_printout" << std::endl;
    std::cout << "fixed_node_in_device: " << std::endl;
    for (int k = 0; k < coordInfoVecs.isNodeFixed.size(); k++)
    {
    }
    std::cout << "end_of_fixed_node_device_printout" << std::endl;
    std::cout << "size of host fixed " << hostSetInfoVecs.isNodeFixed.size() << std::endl;
    std::cout << "size of device fixed " << coordInfoVecs.isNodeFixed.size() << std::endl;

    // initialize various vectors with zeros or values from hostSetInfoVecs.
    //  Fill operations for other nodeForce and prevNodeForce vectors.
    thrust::fill(coordInfoVecs.nodeForceX.begin(), coordInfoVecs.nodeForceX.end(), 0.0);
    thrust::fill(coordInfoVecs.nodeForceY.begin(), coordInfoVecs.nodeForceY.end(), 0.0);
    thrust::fill(coordInfoVecs.nodeForceZ.begin(), coordInfoVecs.nodeForceZ.end(), 0.0);

    thrust::fill(coordInfoVecs.prevNodeForceX.begin(), coordInfoVecs.prevNodeForceX.end(), 0.0);
    thrust::fill(coordInfoVecs.prevNodeForceY.begin(), coordInfoVecs.prevNodeForceY.end(), 0.0);
    thrust::fill(coordInfoVecs.prevNodeForceZ.begin(), coordInfoVecs.prevNodeForceZ.end(), 0.0);

    // Copy node locations and other related information from hostSetInfoVecs to coordInfoVecs and other copy operations for triangles, edges and other related vectors.
    thrust::copy(hostSetInfoVecs.nodeLocX.begin(), hostSetInfoVecs.nodeLocX.end(), coordInfoVecs.prevNodeLocX.begin());
    thrust::copy(hostSetInfoVecs.nodeLocY.begin(), hostSetInfoVecs.nodeLocY.end(), coordInfoVecs.prevNodeLocY.begin());
    thrust::copy(hostSetInfoVecs.nodeLocZ.begin(), hostSetInfoVecs.nodeLocZ.end(), coordInfoVecs.prevNodeLocZ.begin());

    thrust::copy(hostSetInfoVecs.nodeLocX.begin(), hostSetInfoVecs.nodeLocX.end(), coordInfoVecs.nodeLocX.begin());
    thrust::copy(hostSetInfoVecs.nodeLocY.begin(), hostSetInfoVecs.nodeLocY.end(), coordInfoVecs.nodeLocY.begin());
    thrust::copy(hostSetInfoVecs.nodeLocZ.begin(), hostSetInfoVecs.nodeLocZ.end(), coordInfoVecs.nodeLocZ.begin());

    thrust::copy(hostSetInfoVecs.triangles2Nodes_1.begin(), hostSetInfoVecs.triangles2Nodes_1.end(), coordInfoVecs.triangles2Nodes_1.begin());
    thrust::copy(hostSetInfoVecs.triangles2Nodes_2.begin(), hostSetInfoVecs.triangles2Nodes_2.end(), coordInfoVecs.triangles2Nodes_2.begin());
    thrust::copy(hostSetInfoVecs.triangles2Nodes_3.begin(), hostSetInfoVecs.triangles2Nodes_3.end(), coordInfoVecs.triangles2Nodes_3.begin());

    thrust::copy(hostSetInfoVecs.triangles2Edges_1.begin(), hostSetInfoVecs.triangles2Edges_1.end(), coordInfoVecs.triangles2Edges_1.begin());
    thrust::copy(hostSetInfoVecs.triangles2Edges_2.begin(), hostSetInfoVecs.triangles2Edges_2.end(), coordInfoVecs.triangles2Edges_2.begin());
    thrust::copy(hostSetInfoVecs.triangles2Edges_3.begin(), hostSetInfoVecs.triangles2Edges_3.end(), coordInfoVecs.triangles2Edges_3.begin());

    thrust::copy(hostSetInfoVecs.edges2Nodes_1.begin(), hostSetInfoVecs.edges2Nodes_1.end(), coordInfoVecs.edges2Nodes_1.begin());
    thrust::copy(hostSetInfoVecs.edges2Nodes_2.begin(), hostSetInfoVecs.edges2Nodes_2.end(), coordInfoVecs.edges2Nodes_2.begin());

    thrust::copy(hostSetInfoVecs.edges2Triangles_1.begin(), hostSetInfoVecs.edges2Triangles_1.end(), coordInfoVecs.edges2Triangles_1.begin());
    thrust::copy(hostSetInfoVecs.edges2Triangles_2.begin(), hostSetInfoVecs.edges2Triangles_2.end(), coordInfoVecs.edges2Triangles_2.begin());

    thrust::copy(hostSetInfoVecs.nndata1.begin(), hostSetInfoVecs.nndata1.end(), coordInfoVecs.nndata1.begin());
    thrust::copy(hostSetInfoVecs.nndata2.begin(), hostSetInfoVecs.nndata2.end(), coordInfoVecs.nndata2.begin());
    thrust::copy(hostSetInfoVecs.nndata3.begin(), hostSetInfoVecs.nndata3.end(), coordInfoVecs.nndata3.begin());
    thrust::copy(hostSetInfoVecs.nndata4.begin(), hostSetInfoVecs.nndata4.end(), coordInfoVecs.nndata4.begin());
    thrust::copy(hostSetInfoVecs.nndata5.begin(), hostSetInfoVecs.nndata5.end(), coordInfoVecs.nndata5.begin());
    thrust::copy(hostSetInfoVecs.nndata6.begin(), hostSetInfoVecs.nndata6.end(), coordInfoVecs.nndata6.begin());
    thrust::copy(hostSetInfoVecs.nndata7.begin(), hostSetInfoVecs.nndata7.end(), coordInfoVecs.nndata7.begin());
    thrust::copy(hostSetInfoVecs.nndata8.begin(), hostSetInfoVecs.nndata8.end(), coordInfoVecs.nndata8.begin());
    thrust::copy(hostSetInfoVecs.nndata9.begin(), hostSetInfoVecs.nndata9.end(), coordInfoVecs.nndata9.begin());

    // Resize and initialize the 'u' vector.
    coordInfoVecs.u.resize(mem_prealloc * coordInfoVecs.num_triangles);

    // Part 18

    // Allocate memory for additiional data structures.

    // Area triangle info vec.
    // Number of area springs is the number of triangles
    std::cout << "Mem" << std::endl;
    // Allocate memory for temporary node information in unreduced form for area springs
    areaTriangleInfoVecs.tempNodeIdUnreduced.resize(mem_prealloc * areaTriangleInfoVecs.factor * coordInfoVecs.num_triangles);
    areaTriangleInfoVecs.tempNodeForceXUnreduced.resize(mem_prealloc * areaTriangleInfoVecs.factor * coordInfoVecs.num_triangles);
    areaTriangleInfoVecs.tempNodeForceYUnreduced.resize(mem_prealloc * areaTriangleInfoVecs.factor * coordInfoVecs.num_triangles);
    areaTriangleInfoVecs.tempNodeForceZUnreduced.resize(mem_prealloc * areaTriangleInfoVecs.factor * coordInfoVecs.num_triangles);

    // Allocate memory for temporary node information in reduced form for area springs.
    areaTriangleInfoVecs.tempNodeIdReduced.resize(mem_prealloc * areaTriangleInfoVecs.factor * coordInfoVecs.num_triangles);
    areaTriangleInfoVecs.tempNodeForceXReduced.resize(mem_prealloc * areaTriangleInfoVecs.factor * coordInfoVecs.num_triangles);
    areaTriangleInfoVecs.tempNodeForceYReduced.resize(mem_prealloc * areaTriangleInfoVecs.factor * coordInfoVecs.num_triangles);
    areaTriangleInfoVecs.tempNodeForceZReduced.resize(mem_prealloc * areaTriangleInfoVecs.factor * coordInfoVecs.num_triangles);

    // beinding triangle info vec
    // num bending springs is the number of times each edge is between two triangles.
    bendingTriangleInfoVecs.numBendingSprings = coordInfoVecs.num_edges;

    // Allocate memory for temporary node information in unreduced form for bending springs.
    bendingTriangleInfoVecs.tempNodeIdUnreduced.resize(mem_prealloc * bendingTriangleInfoVecs.factor * bendingTriangleInfoVecs.numBendingSprings);
    bendingTriangleInfoVecs.tempNodeForceXUnreduced.resize(mem_prealloc * bendingTriangleInfoVecs.factor * bendingTriangleInfoVecs.numBendingSprings);
    bendingTriangleInfoVecs.tempNodeForceYUnreduced.resize(mem_prealloc * bendingTriangleInfoVecs.factor * bendingTriangleInfoVecs.numBendingSprings);
    bendingTriangleInfoVecs.tempNodeForceZUnreduced.resize(mem_prealloc * bendingTriangleInfoVecs.factor * bendingTriangleInfoVecs.numBendingSprings);

    // Allocate memory for temporary node information in reduced form for bending springs.
    bendingTriangleInfoVecs.tempNodeIdReduced.resize(mem_prealloc * bendingTriangleInfoVecs.factor * bendingTriangleInfoVecs.numBendingSprings);
    bendingTriangleInfoVecs.tempNodeForceXReduced.resize(mem_prealloc * bendingTriangleInfoVecs.factor * bendingTriangleInfoVecs.numBendingSprings);
    bendingTriangleInfoVecs.tempNodeForceYReduced.resize(mem_prealloc * bendingTriangleInfoVecs.factor * bendingTriangleInfoVecs.numBendingSprings);
    bendingTriangleInfoVecs.tempNodeForceZReduced.resize(mem_prealloc * bendingTriangleInfoVecs.factor * bendingTriangleInfoVecs.numBendingSprings);
    
    bendingTriangleInfoVecs.initial_angle.resize(coordInfoVecs.num_triangles);
    thrust::fill(bendingTriangleInfoVecs.initial_angle.begin(), bendingTriangleInfoVecs.initial_angle.end(), 0.0087);
        
    // linear springs info vectors.
    //  Allocate memory for temporary node information in unreduced form for linear springs.
    linearSpringInfoVecs.tempNodeIdUnreduced.resize(mem_prealloc * linearSpringInfoVecs.factor * coordInfoVecs.num_edges);
    linearSpringInfoVecs.tempNodeForceXUnreduced.resize(mem_prealloc * linearSpringInfoVecs.factor * coordInfoVecs.num_edges);
    linearSpringInfoVecs.tempNodeForceYUnreduced.resize(mem_prealloc * linearSpringInfoVecs.factor * coordInfoVecs.num_edges);
    linearSpringInfoVecs.tempNodeForceZUnreduced.resize(mem_prealloc * linearSpringInfoVecs.factor * coordInfoVecs.num_edges);

    // Allocate memory for temporary node information in reduced form for bending springs.
    linearSpringInfoVecs.tempNodeIdReduced.resize(mem_prealloc * linearSpringInfoVecs.factor * coordInfoVecs.num_edges);
    linearSpringInfoVecs.tempNodeForceXReduced.resize(mem_prealloc * linearSpringInfoVecs.factor * coordInfoVecs.num_edges);
    linearSpringInfoVecs.tempNodeForceYReduced.resize(mem_prealloc * linearSpringInfoVecs.factor * coordInfoVecs.num_edges);
    linearSpringInfoVecs.tempNodeForceZReduced.resize(mem_prealloc * linearSpringInfoVecs.factor * coordInfoVecs.num_edges);

    // Clear edge_initial_length vector for linear springs.
    // linearSpringInfoVecs.edge_initial_length.clear();
    // linearSpringInfoVecs.edge_rest_length.clear();

    // linearSpringInfoVecs.edge_rest_length.resize(hostSetInfoVecs.edge_rest_length.size());
    linearSpringInfoVecs.edge_final_length.resize(hostSetInfoVecs.edge_initial_length.size());
    linearSpringInfoVecs.edge_initial_length = hostSetInfoVecs.edge_initial_length;
    linearSpringInfoVecs.edge_final_length = hostSetInfoVecs.edge_initial_length;
    
    
    std::cout << "host edge_initial_length size = " << hostSetInfoVecs.edge_initial_length.size() << std::endl;
    std::cout << "device edge_initial_length size = " << linearSpringInfoVecs.edge_initial_length.size() << std::endl;

    //  for (int e = 0; e < coordInfoVecs.num_edges; ++e) {
    //    int i = coordInfoVecs.edges2Nodes_1[e];
    //    int j = coordInfoVecs.edges2Nodes_2[e];
    //    double dx = hostSetInfoVecs.nodeLocX[j] - hostSetInfoVecs.nodeLocX[i];
    //    double dy = hostSetInfoVecs.nodeLocY[j] - hostSetInfoVecs.nodeLocY[i];
    //    double dz = hostSetInfoVecs.nodeLocZ[j] - hostSetInfoVecs.nodeLocZ[i];
    //    double dist = sqrt(dx*dx + dy*dy + dz*dz);
    //    //hostSetInfoVecs.edge_initial_length.push_back(dist);    // already done for initial
    //    hostSetInfoVecs.edge_rest_length.push_back(dist);
    //    //std::cout<< "edge_rest_length = " << hostSetInfoVecs.edge_initial_length[i]<<std::endl; in the current data structure it gave me 1021 edges. That's good. Now that they have been initialized I should start changing the rest lengths.
    //    }

    // thrust::copy(linearSpringInfoVecs.edge_rest_length.begin(),
    //              linearSpringInfoVecs.edge_rest_length.end(),
    //              hostSetInfoVecs.edge_rest_length.begin());

    linearSpringInfoVecs.edge_rest_length = hostSetInfoVecs.edge_initial_length;

    //linearSpringInfoVecs.edge_rest_length.resize(1/(generalParams.dt*generalParams.tol)) //= hostSetInfoVecs.edge_rest_length;

    std::cout << "host edge_rest_length size = " << hostSetInfoVecs.edge_rest_length.size() << std::endl;
    std::cout << "device edge_rest_length size = " << linearSpringInfoVecs.edge_rest_length.size() << std::endl;

    //linearSpringInfoVecs.edge_final_length.resize(coordInfoVecs.num_edges);

    //linearSpringInfoVecs.edge_final_length = linearSpringInfoVecs.edge_initial_length;

    // This loop is to test out whether the edge_rest_length is being initialized properly.
    // for (int i = 0; i < hostSetInfoVecs.edge_rest_length.size(); i++) {
    //    std::cout<< "edge_rest_length # "<< i <<" = "<< hostSetInfoVecs.edge_rest_length[i]<<std::endl;
    //    std::cout<< "edge_initial_length # "<< i <<" = "<< hostSetInfoVecs.edge_initial_length[i]<<std::endl;
    //}

    // Resize the hostSetInfoVecs for data transfer between host and device.
    hostSetInfoVecs.isNodeFixed.resize(mem_prealloc * hostSetInfoVecs.nodeLocX.size());

    hostSetInfoVecs.nodeLocX.resize(coordInfoVecs.nodeLocX.size());
    hostSetInfoVecs.nodeLocY.resize(coordInfoVecs.nodeLocX.size());
    hostSetInfoVecs.nodeLocZ.resize(coordInfoVecs.nodeLocX.size());
    std::cout << "Host_nodeLocX size = " << hostSetInfoVecs.nodeLocX.size() << std::endl;

    // hostSetInfoVecs.nodeVelX.resize(coordInfoVecs.nodeVelX.size());
    // hostSetInfoVecs.nodeVelY.resize(coordInfoVecs.nodeVelY.size());
    // hostSetInfoVecs.nodeVelZ.resize(coordInfoVecs.nodeVelZ.size());

    hostSetInfoVecs.nodeForceX.resize(coordInfoVecs.nodeLocX.size());
    hostSetInfoVecs.nodeForceY.resize(coordInfoVecs.nodeLocX.size());
    hostSetInfoVecs.nodeForceZ.resize(coordInfoVecs.nodeLocX.size());
    std::cout << "Host_nodeForceX size = " << hostSetInfoVecs.nodeLocX.size() << std::endl;

    hostSetInfoVecs.triangles2Nodes_1.resize(coordInfoVecs.triangles2Nodes_1.size());
    hostSetInfoVecs.triangles2Nodes_2.resize(coordInfoVecs.triangles2Nodes_2.size());
    hostSetInfoVecs.triangles2Nodes_3.resize(coordInfoVecs.triangles2Nodes_3.size());
    std::cout << "Host_triangles2Nodes size = " << hostSetInfoVecs.triangles2Nodes_1.size() << std::endl;

    hostSetInfoVecs.triangles2Edges_1.resize(coordInfoVecs.triangles2Edges_1.size());
    hostSetInfoVecs.triangles2Edges_2.resize(coordInfoVecs.triangles2Edges_2.size());
    hostSetInfoVecs.triangles2Edges_3.resize(coordInfoVecs.triangles2Edges_3.size());
    std::cout << "Host_triangles2Edges size = " << hostSetInfoVecs.triangles2Edges_1.size() << std::endl;

    hostSetInfoVecs.edges2Nodes_1.resize(coordInfoVecs.edges2Nodes_1.size());
    hostSetInfoVecs.edges2Nodes_2.resize(coordInfoVecs.edges2Nodes_2.size());
    std::cout << "Host_edges2Nodes size = " << hostSetInfoVecs.edges2Nodes_1.size() << std::endl;

    hostSetInfoVecs.edges2Triangles_1.resize(coordInfoVecs.edges2Triangles_1.size());
    hostSetInfoVecs.edges2Triangles_2.resize(coordInfoVecs.edges2Triangles_2.size());
    std::cout << "Host_edges2Triangles size = " << hostSetInfoVecs.edges2Triangles_1.size() << std::endl;

    hostSetInfoVecs.nndata1.resize(mem_prealloc * generalParams.maxNodeCount);
    hostSetInfoVecs.nndata2.resize(mem_prealloc * generalParams.maxNodeCount);
    hostSetInfoVecs.nndata3.resize(mem_prealloc * generalParams.maxNodeCount);
    hostSetInfoVecs.nndata4.resize(mem_prealloc * generalParams.maxNodeCount);
    hostSetInfoVecs.nndata5.resize(mem_prealloc * generalParams.maxNodeCount);
    hostSetInfoVecs.nndata6.resize(mem_prealloc * generalParams.maxNodeCount);
    hostSetInfoVecs.nndata7.resize(mem_prealloc * generalParams.maxNodeCount);
    hostSetInfoVecs.nndata8.resize(mem_prealloc * generalParams.maxNodeCount);
    hostSetInfoVecs.nndata9.resize(mem_prealloc * generalParams.maxNodeCount);

    // Print message indicating the system is ready.
    std::cout << "System Ready" << std::endl;

    // Allocate memory for buckets.
    auxVecs.id_bucket.resize(generalParams.maxNodeCount);
    auxVecs.id_value.resize(generalParams.maxNodeCount);
    auxVecs.id_bucket_expanded.resize(27 * (generalParams.maxNodeCount));
    auxVecs.id_value_expanded.resize(27 * (generalParams.maxNodeCount));
};





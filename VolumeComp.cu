#include "hip/hip_runtime.h"

#include "System.h"
#include "SystemStructures.h"
#include "VolumeComp.h"

void ComputeVolume(
    GeneralParams& generalParams,
    CoordInfoVecs& coordInfoVecs,
    LinearSpringInfoVecs& linearSpringInfoVecs,
    LJInfoVecs& ljInfoVecs) {  
    
    // Create a counting iterator for triangle IDs from 0 to num_triangles.
    thrust::counting_iterator<int> triangleIdBegin(0);
    // Note: The triangleIdEnd is not used in the code.

    // Calculate the current total volume of the system using thrust::transform_reduce.
    // The VolumeCompFunctor is used to compute the volume for each triangle.
    // It takes the counting iterator and the indices of three nodes (r1, r2, r3) of each triangle as input.
    // The output of this transform_reduce is the sum of all the computed volumes, which represents the current total volume.
    generalParams.current_total_volume = thrust::transform_reduce(  
        thrust::make_zip_iterator(
            thrust::make_tuple(
                triangleIdBegin,
                coordInfoVecs.triangles2Nodes_1.begin(),
                coordInfoVecs.triangles2Nodes_2.begin(),
                coordInfoVecs.triangles2Nodes_3.begin()
                )
            ),
        thrust::make_zip_iterator( 
            thrust::make_tuple(
                triangleIdBegin,
                coordInfoVecs.triangles2Nodes_1.begin(),
                coordInfoVecs.triangles2Nodes_2.begin(), 
                coordInfoVecs.triangles2Nodes_3.begin()
                )
            ) + coordInfoVecs.num_triangles,
        // VolumeCompFunctor is used to calculate the volume of each triangle.
        VolumeCompFunctor(
            linearSpringInfoVecs.spring_constant, 
            thrust::raw_pointer_cast(coordInfoVecs.nodeLocX.data()),
            thrust::raw_pointer_cast(coordInfoVecs.nodeLocY.data()),
            thrust::raw_pointer_cast(coordInfoVecs.nodeLocZ.data()) 
            ),
        0.0, 
        thrust::plus<double>() 
        ); 
        // This sum is the part without the absolute value and factor of (1/6) in the formula.

    // Calculate the true current total volume by taking the absolute value of the computed current total volume.
    generalParams.true_current_total_volume = sqrt(generalParams.current_total_volume*generalParams.current_total_volume)/6.0;
    
    // Calculate the volume energy using the volume spring constant and the deviation of the current total volume from the equilibrium volume.
    // The result is stored in generalParams.volume_energy.
    generalParams.volume_energy = generalParams.volume_spring_constant*(generalParams.true_current_total_volume - generalParams.eq_total_volume)*
                                        (generalParams.true_current_total_volume - generalParams.eq_total_volume)/
                                        (2.0*generalParams.Rmin*generalParams.Rmin*generalParams.Rmin*generalParams.eq_total_volume);

};


//Note: The code computes the total volume of a system using the VolumeCompFunctor and performs additional calculations related to the system's volume energy.
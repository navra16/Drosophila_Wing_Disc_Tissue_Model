#include "hip/hip_runtime.h"
#include "gradientRelax.h"
#include <vector>            // for std::vector
#include <thrust/copy.h>     // for thrust::copy
#include <limits>            // for std::numeric_limits
#include <cmath>             // for std::sqrt

int relaxUntilConverged(System& system)
{
    // Pull references to the pieces we need
    auto& coordInfoVecs = system.coordInfoVecs;
    auto& generalParams = system.generalParams;
    auto& domainParams  = system.domainParams;

    // Number of nodes
    const int N = static_cast<int>(coordInfoVecs.nodeLocX.size());

    // Host-side buffers for before/after positions
    std::vector<double> x_old(N), y_old(N), z_old(N);
    std::vector<double> x_new(N), y_new(N), z_new(N);

    // Force-movement accumulator
    generalParams.dx = std::numeric_limits<double>::infinity();
    int iter = 0;

    while (generalParams.dx > generalParams.tol) {
        // 1) Snapshot old positions (device ? host)
        thrust::copy(
            coordInfoVecs.nodeLocX.begin(),
            coordInfoVecs.nodeLocX.end(),
            x_old.begin());
        thrust::copy(
            coordInfoVecs.nodeLocY.begin(),
            coordInfoVecs.nodeLocY.end(),
            y_old.begin());
        thrust::copy(
            coordInfoVecs.nodeLocZ.begin(),
            coordInfoVecs.nodeLocZ.end(),
            z_old.begin());

        // 2) Build forces, then move nodes
        system.Solve_Forces();  // member in System
        AdvancePositions(
            coordInfoVecs,
            generalParams,
            domainParams);       // free function

        // 3) Snapshot new positions 
        thrust::copy(
            coordInfoVecs.nodeLocX.begin(),
            coordInfoVecs.nodeLocX.end(),
            x_new.begin());
        thrust::copy(
            coordInfoVecs.nodeLocY.begin(),
            coordInfoVecs.nodeLocY.end(),
            y_new.begin());
        thrust::copy(
            coordInfoVecs.nodeLocZ.begin(),
            coordInfoVecs.nodeLocZ.end(),
            z_new.begin());

        // 4) Compute total L2-movement across all nodes
        double dx_sum = 0.0;
        for (int i = 0; i < N; ++i) {
            double dx = x_new[i] - x_old[i];
            double dy = y_new[i] - y_old[i];
            double dz = z_new[i] - z_old[i];
            dx_sum += std::sqrt(dx*dx + dy*dy + dz*dz);
        }
        generalParams.dx = dx_sum/77.66;

        ++iter;
    }

    return iter;
}

#include "hip/hip_runtime.h"

#include "System.h"
#include "SystemStructures.h"
#include "BendingTriangles.h"

void ComputeCosTriangleSprings(
    GeneralParams& generalParams,
    CoordInfoVecs& coordInfoVecs,
    BendingTriangleInfoVecs& bendingTriangleInfoVecs) {
/*
if (generalParams.iteration % 1 == 0) {
	
	std::cout<<" iteration: "<< generalParams.iteration << std::endl;
        unsigned id_l, id_j;
        
        unsigned T1 = coordInfoVecs.edges2Triangles_1[0];
        unsigned T2 = coordInfoVecs.edges2Triangles_2[0];

		//these id's are accurate
		unsigned id_k = coordInfoVecs.edges2Nodes_1[0];
        unsigned id_i = coordInfoVecs.edges2Nodes_2[0];
        
		if (T1 != T2) {
			//we need to compute rl and rj from the two involved triangles. 

			
			//one of these is j, but it cannot be equal to id_i or id_k
			unsigned n1T1 = coordInfoVecs.triangles2Nodes_1[T1];
			unsigned n2T1 = coordInfoVecs.triangles2Nodes_2[T1];
			unsigned n3T1 = coordInfoVecs.triangles2Nodes_3[T1];
			if ((n1T1 != id_i) && (n1T1 != id_k)) {
				id_j = n1T1;
			}
			else if ((n2T1 != id_i) && (n2T1 != id_k)) {
				id_j = n2T1;
			}
			else if ((n3T1 != id_i) && (n3T1 != id_k)) {
				id_j = n3T1;
			}

			//one of these is l, find it
			unsigned n1T2 = coordInfoVecs.triangles2Nodes_1[T2];
			unsigned n2T2 = coordInfoVecs.triangles2Nodes_2[T2];
			unsigned n3T2 = coordInfoVecs.triangles2Nodes_3[T2];
			if ((n1T2 != id_i) && (n1T2 != id_k)) {
				id_l = n1T2;
			}
			else if ((n2T2 != id_i) && (n2T2 != id_k)) {
				id_l = n2T2;
			}
			else if ((n3T2 != id_i) && (n3T2 != id_k)) {
				id_l = n3T2;
			}

			std::cout<<"i: "<< id_i << std::endl;
			std::cout<<"k: "<< id_k << std::endl;
			std::cout<<"l: "<< id_l << std::endl;
			std::cout<<"j: "<< id_j << std::endl;
			CVec3 ri = thrust::make_tuple<double>(coordInfoVecs.nodeLocX[id_i], coordInfoVecs.nodeLocY[id_i], coordInfoVecs.nodeLocZ[id_i]);
			CVec3 rj = thrust::make_tuple<double>(coordInfoVecs.nodeLocX[id_j], coordInfoVecs.nodeLocY[id_j], coordInfoVecs.nodeLocZ[id_j]);
			CVec3 rk = thrust::make_tuple<double>(coordInfoVecs.nodeLocX[id_k], coordInfoVecs.nodeLocY[id_k], coordInfoVecs.nodeLocZ[id_k]);
			CVec3 rl = thrust::make_tuple<double>(coordInfoVecs.nodeLocX[id_l], coordInfoVecs.nodeLocY[id_l], coordInfoVecs.nodeLocZ[id_l]);
			std::cout<<"ri: "<< thrust::get<0>(ri)<<" "<< thrust::get<1>(ri)<<" "<< thrust::get<2>(ri)<<std::endl;
            std::cout<<"rj: "<< thrust::get<0>(rj)<<" "<< thrust::get<1>(rj)<<" "<< thrust::get<2>(rj)<<std::endl;
            std::cout<<"rk: "<< thrust::get<0>(rk)<<" "<< thrust::get<1>(rk)<<" "<< thrust::get<2>(rk)<<std::endl;
            std::cout<<"rl: "<< thrust::get<0>(rl)<<" "<< thrust::get<1>(rl)<<" "<< thrust::get<2>(rl)<<std::endl;
            
			CVec3 rjk = CVec3_plus(rk, CVec3_scalermult(-1.0, rj) );
			CVec3 rji = CVec3_plus(ri, CVec3_scalermult(-1.0, rj) );
			CVec3 rli = CVec3_plus(ri, CVec3_scalermult(-1.0, rl) );		
			CVec3 rlk = CVec3_plus(rk, CVec3_scalermult(-1.0, rl) );		
			CVec3 rki = CVec3_plus(ri, CVec3_scalermult(-1.0, rk) );

            std::cout<<"rjk: "<< thrust::get<0>(rjk)<<" "<< thrust::get<1>(rjk)<<" "<< thrust::get<2>(rjk)<<std::endl;
            std::cout<<"rji: "<< thrust::get<0>(rji)<<" "<< thrust::get<1>(rji)<<" "<< thrust::get<2>(rji)<<std::endl;
            std::cout<<"rli: "<< thrust::get<0>(rli)<<" "<< thrust::get<1>(rli)<<" "<< thrust::get<2>(rli)<<std::endl;
            std::cout<<"rlk: "<< thrust::get<0>(rlk)<<" "<< thrust::get<1>(rlk)<<" "<< thrust::get<2>(rlk)<<std::endl;
            std::cout<<"rki: "<< thrust::get<0>(rki)<<" "<< thrust::get<1>(rki)<<" "<< thrust::get<2>(rki)<<std::endl;
			double nrki = sqrt(CVec3_dot(rki, rki));
			//nrki = sqrt(sum(rki.^2));

			CVec3 unitDir =  thrust::make_tuple<double>(thrust::get<0>(rki)/nrki,
														thrust::get<1>(rki)/nrki,
														thrust::get<2>(rki)/nrki);

			//UD is the unit direction we use to check if the cross product is pointing
			//in the right direction.
			double inv_nrki_sq = 1.0/ (CVec3_dot(rki, rki)); //CHANGE(9/13): removing sqrt, this corresponds to norm^2
			CVec3 zero_vec = thrust::make_tuple<double>(0.0, 0.0, 0.0);
			CVec3 unitX = thrust::make_tuple<double>(1.0,0.0,0.0);
			CVec3 unitY = thrust::make_tuple<double>(0.0,1.0,0.0);
			CVec3 unitZ = thrust::make_tuple<double>(0.0,0.0,1.0);
			
			Mat_3x3 dUD_rj = thrust::make_tuple<CVec3>(
				CVec3_scalermult( inv_nrki_sq,
					CVec3_plus( 
						CVec3_scalermult( nrki, zero_vec) , 
						CVec3_scalermult( ((0.0 + 0.0 + 0.0) * (-1.0/nrki)), rki) ) ),
				CVec3_scalermult( inv_nrki_sq,
					CVec3_plus( 
						CVec3_scalermult( nrki, zero_vec) , 
						CVec3_scalermult( ((0.0 + 0.0 + 0.0) * (-1.0/nrki)), rki)) ),
				CVec3_scalermult( inv_nrki_sq,
					CVec3_plus( 
						CVec3_scalermult( nrki, zero_vec) , 
						CVec3_scalermult( ((0.0 + 0.0 + 0.0) * (-1.0/nrki)), rki)) ) );
			std::cout<< "dUD_rj_1"<< thrust::get<0>(thrust::get<0>(dUD_rj))<< " "<< thrust::get<1>(thrust::get<0>(dUD_rj))<< " "<< thrust::get<2>(thrust::get<0>(dUD_rj)) <<std::endl;
			std::cout<< "dUD_rj_2"<< thrust::get<0>(thrust::get<1>(dUD_rj))<< " "<< thrust::get<1>(thrust::get<1>(dUD_rj))<< " "<< thrust::get<2>(thrust::get<1>(dUD_rj)) <<std::endl;
			std::cout<< "dUD_rj_3"<< thrust::get<0>(thrust::get<2>(dUD_rj))<< " "<< thrust::get<1>(thrust::get<2>(dUD_rj))<< " "<< thrust::get<2>(thrust::get<2>(dUD_rj)) <<std::endl;
						
				//CVec3_minus( CVec3_scalermult( inv_nrki_sq, zero_vec), CVec3_scalermult( (0 + 0 + 0) * (1/nrki) ,rki )),
				//CVec3_minus( CVec3_scalermult( inv_nrki_sq, zero_vec), CVec3_scalermult( (0 + 0 + 0) * (1/nrki) ,rki )),
				//CVec3_minus( CVec3_scalermult( inv_nrki_sq, zero_vec), CVec3_scalermult( (0 + 0 + 0) * (1/nrki) ,rki ))); 
				//CHANGE(9/13): rewriting the computation to match the original matlab version

				//(1/norm(rki)^2)*[
				//	nrki*[0,0,0] - rki*(1/nrki)*(0+0+0);...
				//  nrki*[0,0,0] - rki*(1/nrki)*(0+0+0);...
				//  nrki*[0,0,0] - rki*(1/nrki)*(0+0+0)];

			//While dUD_rj can be listed as zero vectors directly since it has no
			//dependence on rj, it is written out fully for double-checking.
			Mat_3x3 dUD_rk = thrust::make_tuple<CVec3>(
				CVec3_scalermult( inv_nrki_sq,
					CVec3_plus( 
						CVec3_scalermult( -nrki, unitX) , 
						CVec3_scalermult(-1.0*(thrust::get<0>(rki) * (-1.0) + 0.0 + 0.0) * (1.0/nrki), rki) )),
				CVec3_scalermult( inv_nrki_sq,
					CVec3_plus( 
						CVec3_scalermult( -nrki, unitY) , 
						CVec3_scalermult(-1.0*(0.0 + thrust::get<1>(rki) * (-1.0) + 0.0) * (1.0/nrki), rki) )),
				CVec3_scalermult( inv_nrki_sq,
					CVec3_plus( 
						CVec3_scalermult( -nrki, unitZ) , 
						CVec3_scalermult(-1.0*(0.0 + 0.0 + thrust::get<2>(rki) * (-1.0)) * (1.0/nrki), rki) )) );
				
					std::cout<< "dUD_rk_1"<< thrust::get<0>(thrust::get<0>(dUD_rk))<< " "<< thrust::get<1>(thrust::get<0>(dUD_rk))<< " "<< thrust::get<2>(thrust::get<0>(dUD_rk)) <<std::endl;
					std::cout<< "dUD_rk_2"<< thrust::get<0>(thrust::get<1>(dUD_rk))<< " "<< thrust::get<1>(thrust::get<1>(dUD_rk))<< " "<< thrust::get<2>(thrust::get<1>(dUD_rk)) <<std::endl;
					std::cout<< "dUD_rk_3"<< thrust::get<0>(thrust::get<2>(dUD_rk))<< " "<< thrust::get<1>(thrust::get<2>(dUD_rk))<< " "<< thrust::get<2>(thrust::get<2>(dUD_rk)) <<std::endl;
					
			
			Mat_3x3 dUD_ri = thrust::make_tuple<CVec3>(
				CVec3_scalermult(inv_nrki_sq, 
					CVec3_plus( 
						CVec3_scalermult(nrki, unitX),  
						CVec3_scalermult(-1.0*(thrust::get<0>(rki)*(1.0) + 0.0 + 0.0) * (1.0/nrki), rki) )),
				CVec3_scalermult(inv_nrki_sq, 
					CVec3_plus(
						CVec3_scalermult(nrki, unitY),  
						CVec3_scalermult(-1.0*(0.0 + thrust::get<1>(rki) * (1.0) + 0.0) * (1.0/nrki), rki) )),
				CVec3_scalermult(inv_nrki_sq, 
					CVec3_plus(
						CVec3_scalermult(nrki, unitZ),  
						CVec3_scalermult(-1.0*(0.0 + 0.0 + thrust::get<2>(rki) * (1.0)) * (1.0/nrki), rki) )) );
			
			std::cout<< "dUD_ri_1"<< thrust::get<0>(thrust::get<0>(dUD_ri))<< " "<< thrust::get<1>(thrust::get<0>(dUD_ri))<< " "<< thrust::get<2>(thrust::get<0>(dUD_ri)) <<std::endl;
			std::cout<< "dUD_ri_2"<< thrust::get<0>(thrust::get<1>(dUD_ri))<< " "<< thrust::get<1>(thrust::get<1>(dUD_ri))<< " "<< thrust::get<2>(thrust::get<1>(dUD_ri)) <<std::endl;
			std::cout<< "dUD_ri_3"<< thrust::get<0>(thrust::get<2>(dUD_ri))<< " "<< thrust::get<1>(thrust::get<2>(dUD_ri))<< " "<< thrust::get<2>(thrust::get<2>(dUD_ri)) <<std::endl;
					
			
				//CVec3_minus( CVec3_scalermult( inv_nrki_sq, unitX), CVec3_scalermult( (thrust::get<0>(rki)*(1) + 0 + 0) * (1/nrki) ,rki )),
				//CVec3_minus( CVec3_scalermult( inv_nrki_sq, unitY), CVec3_scalermult( (0 + thrust::get<1>(rki)*(1) + 0) * (1/nrki) ,rki )),
				//CVec3_minus( CVec3_scalermult( inv_nrki_sq, unitZ), CVec3_scalermult( (0 + 0 + thrust::get<2>(rki)*(1)) * (1/nrki) ,rki ))); 
				//CHANGE(9/13): rewriting the computation to match the original matlab version

				//(1/nrki^2)*[
				//	nrki*[1,0,0] - rki*(1/nrki)*(rki(1)*1+0+0);...
				//	nrki*[0,1,0] - rki*(1/nrki)*(0+rki(2)*1+0);...
				//	nrki*[0,0,1] - rki*(1/nrki)*(0+0+rki(3)*1)];

			Mat_3x3 dUD_rl = thrust::make_tuple<CVec3>(
				CVec3_scalermult(inv_nrki_sq, 
					CVec3_plus(
						CVec3_scalermult(nrki, zero_vec),  
						CVec3_scalermult(-1.0*(0.0 + 0.0 + 0.0) * (1.0/nrki), rki) )),
				CVec3_scalermult(inv_nrki_sq, 
					CVec3_plus(
						CVec3_scalermult(nrki, zero_vec),  
						CVec3_scalermult(-1.0*(0.0 + 0.0 + 0.0) * (1.0/nrki), rki) )),
				CVec3_scalermult(inv_nrki_sq, 
					CVec3_plus(
						CVec3_scalermult(nrki, zero_vec),  
						CVec3_scalermult(-1.0*(0.0 + 0.0 + 0.0) * (1.0/nrki), rki) )) );
						std::cout<< "dUD_rl_1"<< thrust::get<0>(thrust::get<0>(dUD_rl))<< " "<< thrust::get<1>(thrust::get<0>(dUD_rl))<< " "<< thrust::get<2>(thrust::get<0>(dUD_rl)) <<std::endl;
						std::cout<< "dUD_rl_2"<< thrust::get<0>(thrust::get<1>(dUD_rl))<< " "<< thrust::get<1>(thrust::get<1>(dUD_rl))<< " "<< thrust::get<2>(thrust::get<1>(dUD_rl)) <<std::endl;
						std::cout<< "dUD_rl_3"<< thrust::get<0>(thrust::get<2>(dUD_rl))<< " "<< thrust::get<1>(thrust::get<2>(dUD_rl))<< " "<< thrust::get<2>(thrust::get<2>(dUD_rl)) <<std::endl;
					

			
			CVec3 N1 = CVec3_cross(rjk,rji);
			CVec3 N2 = CVec3_cross(rli, rlk);
			double nN1 = sqrt(CVec3_dot(N1,N1));
			double nN2 = sqrt(CVec3_dot(N2,N2));

			//N1 = cross(rjk, rji);
			//N2 = cross(rli, rlk);
			//nN1 = sqrt(sum(N1.^2)); %norm of N1
			//nN2 = sqrt(sum(N2.^2)); %norm of N2

			double A1 = thrust::get<0>(N1);
			double B1 = thrust::get<1>(N1);
			double C1 = thrust::get<2>(N1);
			double A2 = thrust::get<0>(N2);
			double B2 = thrust::get<1>(N2);
			double C2 = thrust::get<2>(N2);

			
			std::cout<<"N1: "<< thrust::get<0>(N1)<<" "<< thrust::get<1>(N1)<<" "<< thrust::get<2>(N1)<<std::endl;
            std::cout<<"N2: "<< thrust::get<0>(N2)<<" "<< thrust::get<1>(N2)<<" "<< thrust::get<2>(N2)<<std::endl;
            

			//Derivative of 1st component in N1 with respect to rj
			CVec3 A1_rj = thrust::make_tuple<double>(0.0, -thrust::get<2>(rji) + thrust::get<2>(rjk), -thrust::get<1>(rjk) +thrust::get<1>(rji));
			//A1_rj = [0 , -rji(3)+rjk(3) , -rjk(2)+rji(2)];
			CVec3 A1_rk = thrust::make_tuple<double>(0.0, thrust::get<2>(rji), -thrust::get<1>(rji) );
			//A1_rk = [0 , rji(3) , -rji(2)];
			CVec3 A1_ri = thrust::make_tuple<double>(0.0, -thrust::get<2>(rjk), thrust::get<1>(rjk) );
			//A1_ri = [0 , -rjk(3) , rjk(2)];
			CVec3 A1_rl = thrust::make_tuple<double>(0.0, 0.0, 0.0 );
			//A1_rl = [0 , 0 , 0];

			CVec3 B1_rj = thrust::make_tuple<double>(thrust::get<2>(rji) - thrust::get<2>(rjk), 0.0, thrust::get<0>(rjk) - thrust::get<0>(rji));
			//B1_rj = [rji(3)-rjk(3) , 0 , rjk(1)-rji(1)];
			CVec3 B1_rk = thrust::make_tuple<double>(-thrust::get<2>(rji), 0.0, thrust::get<0>(rji));
			//B1_rk = [-rji(3) , 0 , rji(1)];
			CVec3 B1_ri = thrust::make_tuple<double>(thrust::get<2>(rjk), 0.0, -thrust::get<0>(rjk));
			//B1_ri = [rjk(3) , 0 , -rjk(1)];
			CVec3 B1_rl = thrust::make_tuple<double>(0.0,0.0,0.0);
			//B1_rl = [0 , 0 , 0];

			CVec3 C1_rj = thrust::make_tuple<double>(-thrust::get<1>(rji) + thrust::get<1>(rjk), -thrust::get<0>(rjk) + thrust::get<0>(rji), 0.0);
			//C1_rj = [-rji(2)+rjk(2), -rjk(1)+rji(1) , 0];
			CVec3 C1_rk = thrust::make_tuple<double>(thrust::get<1>(rji), -thrust::get<0>(rji), 0.0);
			//C1_rk = [rji(2), -rji(1) , 0];
            CVec3 C1_ri = thrust::make_tuple<double>(-thrust::get<1>(rjk), thrust::get<0>(rjk), 0.0);
            //C1_ri = [-rjk(2), rjk(1) , 0];
			CVec3 C1_rl = thrust::make_tuple<double>(0.0,0.0,0.0);
			//C1_rl = [0 , 0 , 0];

			CVec3 A2_rj = thrust::make_tuple<double>(0.0,0.0,0.0);
			//A2_rj = [0 , 0 , 0];
			CVec3 A2_rk = thrust::make_tuple<double>( 0.0, -thrust::get<2>(rli), thrust::get<1>(rli) );
			//A2_rk = [0 , -rli(3) , rli(2)];
			CVec3 A2_ri = thrust::make_tuple<double>( 0.0, thrust::get<2>(rlk), -thrust::get<1>(rlk) );
			//A2_ri = [0 , rlk(3) , -rlk(2)];
			CVec3 A2_rl = thrust::make_tuple<double>( 0.0, -thrust::get<2>(rlk) + thrust::get<2>(rli), -thrust::get<1>(rli) + thrust::get<1>(rlk) );
			//A2_rl = [0 , -rlk(3)+rli(3) , -rli(2)+rlk(2)];

			CVec3 B2_rj = thrust::make_tuple<double>(0.0,0.0,0.0);
			//B2_rj = [0 , 0 , 0];
			CVec3 B2_rk = thrust::make_tuple<double>(thrust::get<2>(rli), 0.0, -thrust::get<0>(rli));
			//B2_rk = [rli(3) , 0 , -rli(1)];
			CVec3 B2_ri = thrust::make_tuple<double>(-thrust::get<2>(rlk), 0.0, thrust::get<0>(rlk));
			//B2_ri = [-rlk(3) , 0 , rlk(1)];
			CVec3 B2_rl = thrust::make_tuple<double>(thrust::get<2>(rlk) - thrust::get<2>(rli), 0.0, thrust::get<0>(rli) - thrust::get<0>(rlk));
			//B2_rl = [rlk(3)-rli(3) , 0 , rli(1)-rlk(1)];

			CVec3 C2_rj = thrust::make_tuple<double>(0.0,0.0,0.0);
			//C2_rj = [0 , 0 , 0];
			CVec3 C2_rk = thrust::make_tuple<double>(-thrust::get<1>(rli), thrust::get<0>(rli), 0.0);
			//C2_rk = [-rli(2) , rli(1) , 0];
			CVec3 C2_ri = thrust::make_tuple<double>(thrust::get<1>(rlk), -thrust::get<0>(rlk), 0.0);
			//C2_ri = [rlk(2) , -rlk(1) , 0];
			CVec3 C2_rl = thrust::make_tuple<double>(-thrust::get<1>(rlk) + thrust::get<1>(rli), -thrust::get<0>(rli) + thrust::get<0>(rlk), 0.0);
			//C2_rl = [-rlk(2)+rli(2) , -rli(1)+rlk(1) , 0];

            std::cout<<"A1_rk: "<< thrust::get<0>(A1_rk)<<" "<< thrust::get<1>(A1_rk)<<" "<< thrust::get<2>(A1_rk)<<std::endl;
            std::cout<<"A1_rj: "<< thrust::get<0>(A1_rj)<<" "<< thrust::get<1>(A1_rj)<<" "<< thrust::get<2>(A1_rj)<<std::endl;
            std::cout<<"A1_ri: "<< thrust::get<0>(A1_ri)<<" "<< thrust::get<1>(A1_ri)<<" "<< thrust::get<2>(A1_ri)<<std::endl;
            std::cout<<"A1_rl: "<< thrust::get<0>(A1_rl)<<" "<< thrust::get<1>(A1_rl)<<" "<< thrust::get<2>(A1_rl)<<std::endl;
            std::cout<<"B1_rk: "<< thrust::get<0>(B1_rk)<<" "<< thrust::get<1>(B1_rk)<<" "<< thrust::get<2>(B1_rk)<<std::endl;
            std::cout<<"B1_rj: "<< thrust::get<0>(B1_rj)<<" "<< thrust::get<1>(B1_rj)<<" "<< thrust::get<2>(B1_rj)<<std::endl;
            std::cout<<"B1_ri: "<< thrust::get<0>(B1_ri)<<" "<< thrust::get<1>(B1_ri)<<" "<< thrust::get<2>(B1_ri)<<std::endl;
            std::cout<<"B1_rl: "<< thrust::get<0>(B1_rl)<<" "<< thrust::get<1>(B1_rl)<<" "<< thrust::get<2>(B1_rl)<<std::endl;
            std::cout<<"C1_rk: "<< thrust::get<0>(C1_rk)<<" "<< thrust::get<1>(C1_rk)<<" "<< thrust::get<2>(C1_rk)<<std::endl;
            std::cout<<"C1_rj: "<< thrust::get<0>(C1_rj)<<" "<< thrust::get<1>(C1_rj)<<" "<< thrust::get<2>(C1_rj)<<std::endl;
            std::cout<<"C1_ri: "<< thrust::get<0>(C1_ri)<<" "<< thrust::get<1>(C1_ri)<<" "<< thrust::get<2>(C1_ri)<<std::endl;
            std::cout<<"C1_rl: "<< thrust::get<0>(C1_rl)<<" "<< thrust::get<1>(C1_rl)<<" "<< thrust::get<2>(C1_rl)<<std::endl;
            std::cout<<"A2_rk: "<< thrust::get<0>(A2_rk)<<" "<< thrust::get<1>(A2_rk)<<" "<< thrust::get<2>(A2_rk)<<std::endl;
            std::cout<<"A2_rj: "<< thrust::get<0>(A2_rj)<<" "<< thrust::get<1>(A2_rj)<<" "<< thrust::get<2>(A2_rj)<<std::endl;
            std::cout<<"A2_ri: "<< thrust::get<0>(A2_ri)<<" "<< thrust::get<1>(A2_ri)<<" "<< thrust::get<2>(A2_ri)<<std::endl;
            std::cout<<"A2_rl: "<< thrust::get<0>(A2_rl)<<" "<< thrust::get<1>(A2_rl)<<" "<< thrust::get<2>(A2_rl)<<std::endl;
            std::cout<<"B2_rk: "<< thrust::get<0>(B2_rk)<<" "<< thrust::get<1>(B2_rk)<<" "<< thrust::get<2>(B2_rk)<<std::endl;
            std::cout<<"B2_rj: "<< thrust::get<0>(B2_rj)<<" "<< thrust::get<1>(B2_rj)<<" "<< thrust::get<2>(B2_rj)<<std::endl;
            std::cout<<"B2_ri: "<< thrust::get<0>(B2_ri)<<" "<< thrust::get<1>(B2_ri)<<" "<< thrust::get<2>(B2_ri)<<std::endl;
            std::cout<<"B2_rl: "<< thrust::get<0>(B2_rl)<<" "<< thrust::get<1>(B2_rl)<<" "<< thrust::get<2>(B2_rl)<<std::endl;
            std::cout<<"C2_rk: "<< thrust::get<0>(C2_rk)<<" "<< thrust::get<1>(C2_rk)<<" "<< thrust::get<2>(C2_rk)<<std::endl;
            std::cout<<"C2_rj: "<< thrust::get<0>(C2_rj)<<" "<< thrust::get<1>(C2_rj)<<" "<< thrust::get<2>(C2_rj)<<std::endl;
            std::cout<<"C2_ri: "<< thrust::get<0>(C2_ri)<<" "<< thrust::get<1>(C2_ri)<<" "<< thrust::get<2>(C2_ri)<<std::endl;
            std::cout<<"C2_rl: "<< thrust::get<0>(C2_rl)<<" "<< thrust::get<1>(C2_rl)<<" "<< thrust::get<2>(C2_rl)<<std::endl;
             

			//Derivative of the dot product of normal vectors

			
			CVec3 At =	CVec3_plus(
					CVec3_scalermult( A1 , A2_rj ),
					CVec3_scalermult( A2 , A1_rj ));
			CVec3 Bt =	CVec3_plus(
					CVec3_scalermult( B1 , B2_rj ),
					CVec3_scalermult( B2 , B1_rj ));
			CVec3 Ct =	CVec3_plus(
					CVec3_scalermult( C1 , C2_rj ),
					CVec3_scalermult( C2 , C1_rj )) ;
			
			CVec3 ABt = CVec3_plus(At,Bt);
			
			CVec3 DN1N2_rj = CVec3_plus(ABt,Ct);
					//A1*A2_rj + A2*A1_rj + B1*B2_rj + B2*B1_rj + C1*C2_rj + C2*C1_rj;

			//DN1N2 := dot product dot(N1,N2), and "_rj" represents the partial 
			//derivative with respect to 1st, 2nd, and 3rd component of rj;
			//i.e. rj(1), rj(2), rj(3) being the x,y,z component of rj.

			CVec3 DN1N2_rk = 
				CVec3_plus(
					CVec3_scalermult( A1 , A2_rk ),
					CVec3_scalermult( A2 , A1_rk ),
					CVec3_scalermult( B1 , B2_rk ),
					CVec3_scalermult( B2 , B1_rk ),
					CVec3_scalermult( C1 , C2_rk ),
					CVec3_scalermult( C2 , C1_rk ) );
				//A1*A2_rk + A2*A1_rk + B1*B2_rk + B2*B1_rk + C1*C2_rk + C2*C1_rk;

			CVec3 DN1N2_ri = 
				CVec3_plus(
					CVec3_scalermult( A1 , A2_ri ),
					CVec3_scalermult( A2 , A1_ri ),
					CVec3_scalermult( B1 , B2_ri ),
					CVec3_scalermult( B2 , B1_ri ),
					CVec3_scalermult( C1 , C2_ri ),
					CVec3_scalermult( C2 , C1_ri ) );
				//A1*A2_ri + A2*A1_ri + B1*B2_ri + B2*B1_ri + C1*C2_ri + C2*C1_ri;

			CVec3 DN1N2_rl =
				CVec3_plus(
					CVec3_scalermult( A1 , A2_rl ),
					CVec3_scalermult( A2 , A1_rl ),
					CVec3_scalermult( B1 , B2_rl ),
					CVec3_scalermult( B2 , B1_rl ),
					CVec3_scalermult( C1 , C2_rl ),
					CVec3_scalermult( C2 , C1_rl ) );
				//A1*A2_rl + A2*A1_rl + B1*B2_rl + B2*B1_rl + C1*C2_rl + C2*C1_rl;
                std::cout<<"DN1N2_rj:    "<< thrust::get<0>(DN1N2_rj)<<" "<< thrust::get<1>(DN1N2_rj)<<" "<< thrust::get<2>(DN1N2_rj)<<std::endl;
                std::cout<<"DN1N2_rk:    "<< thrust::get<0>(DN1N2_rk)<<" "<< thrust::get<1>(DN1N2_rk)<<" "<< thrust::get<2>(DN1N2_rk)<<std::endl;
                std::cout<<"DN1N2_ri:    "<< thrust::get<0>(DN1N2_ri)<<" "<< thrust::get<1>(DN1N2_ri)<<" "<< thrust::get<2>(DN1N2_ri)<<std::endl;
                std::cout<<"DN1N2_rl:    "<< thrust::get<0>(DN1N2_rl)<<" "<< thrust::get<1>(DN1N2_rl)<<" "<< thrust::get<2>(DN1N2_rl)<<std::endl;
                
			// Derivative of the product of norms of normal vectors
			CVec3 PnN1nN2_rj = CVec3_plus(
				CVec3_scalermult(nN1/nN2, 
					CVec3_plus(
						CVec3_scalermult( A2 ,A2_rj ), 
						CVec3_scalermult( B2, B2_rj), 
						CVec3_scalermult( C2, C2_rj) ) ),
				CVec3_scalermult(nN2/nN1, 
					CVec3_plus(
						CVec3_scalermult( A1 ,A1_rj ), 
						CVec3_scalermult( B1, B1_rj), 
						CVec3_scalermult( C1, C1_rj) ) ) );
				//PnN1nN2_rj = [
				//nN1*(1/nN2)*(A2*A2_rj(1)+B2*B2_rj(1)+C2*C2_rj(1))+nN2*(1/nN1)*(A1*A1_rj(1)+B1*B1_rj(1)+C1*C1_rj(1));...
    			//nN1*(1/nN2)*(A2*A2_rj(2)+B2*B2_rj(2)+C2*C2_rj(2))+nN2*(1/nN1)*(A1*A1_rj(2)+B1*B1_rj(2)+C1*C1_rj(2));...
    			//nN1*(1/nN2)*(A2*A2_rj(3)+B2*B2_rj(3)+C2*C2_rj(3))+nN2*(1/nN1)*(A1*A1_rj(3)+B1*B1_rj(3)+C1*C1_rj(3))]; 

			CVec3 PnN1nN2_rk = CVec3_plus(
				CVec3_scalermult(nN1/nN2, 
					CVec3_plus(
						CVec3_scalermult( A2, A2_rk ), 
						CVec3_scalermult( B2, B2_rk),
						CVec3_scalermult( C2, C2_rk) ) ),
				CVec3_scalermult(nN2/nN1, 
					CVec3_plus(
						CVec3_scalermult( A1, A1_rk ), 
						CVec3_scalermult( B1, B1_rk), 
						CVec3_scalermult( C1, C1_rk) ) ) );
				//PnN1nN2_rk = [
				//nN1*(1/nN2)*(A2*A2_rk(1)+B2*B2_rk(1)+C2*C2_rk(1))+nN2*(1/nN1)*(A1*A1_rk(1)+B1*B1_rk(1)+C1*C1_rk(1));...
    			//nN1*(1/nN2)*(A2*A2_rk(2)+B2*B2_rk(2)+C2*C2_rk(2))+nN2*(1/nN1)*(A1*A1_rk(2)+B1*B1_rk(2)+C1*C1_rk(2));...
    			//nN1*(1/nN2)*(A2*A2_rk(3)+B2*B2_rk(3)+C2*C2_rk(3))+nN2*(1/nN1)*(A1*A1_rk(3)+B1*B1_rk(3)+C1*C1_rk(3))]; 


			CVec3 PnN1nN2_ri = CVec3_plus(
				CVec3_scalermult(nN1/nN2, 
					CVec3_plus(
						CVec3_scalermult( A2 ,A2_ri ), 
						CVec3_scalermult( B2, B2_ri), 
						CVec3_scalermult( C2, C2_ri) ) ),
				CVec3_scalermult(nN2/nN1, 
					CVec3_plus(
						CVec3_scalermult( A1, A1_ri ), 
						CVec3_scalermult( B1, B1_ri), 
						CVec3_scalermult( C1, C1_ri) ) ) );
				//PnN1nN2_ri=[
				//nN1*(1/nN2)*(A2*A2_ri(1)+B2*B2_ri(1)+C2*C2_ri(1))+nN2*(1/nN1)*(A1*A1_ri(1)+B1*B1_ri(1)+C1*C1_ri(1));...
    			//nN1*(1/nN2)*(A2*A2_ri(2)+B2*B2_ri(2)+C2*C2_ri(2))+nN2*(1/nN1)*(A1*A1_ri(2)+B1*B1_ri(2)+C1*C1_ri(2));...
    			//nN1*(1/nN2)*(A2*A2_ri(3)+B2*B2_ri(3)+C2*C2_ri(3))+nN2*(1/nN1)*(A1*A1_ri(3)+B1*B1_ri(3)+C1*C1_ri(3))]; 

			CVec3 PnN1nN2_rl = CVec3_plus(
				CVec3_scalermult(nN1/nN2, 
					CVec3_plus(
						CVec3_scalermult( A2, A2_rl ), 
						CVec3_scalermult( B2, B2_rl), 
						CVec3_scalermult( C2, C2_rl) ) ),
				CVec3_scalermult(nN2/nN1, 
					CVec3_plus(
						CVec3_scalermult( A1, A1_rl ), 
						CVec3_scalermult( B1, B1_rl), 
                        CVec3_scalermult( C1, C1_rl) ) ) );
					
						CVec3 dcN1N2_rj_1 = thrust::make_tuple<double>(
							(B1 * thrust::get<0>(C2_rj) + C2 * thrust::get<0>(B1_rj)) -(B2 * thrust::get<0>(C1_rj) + C1 * thrust::get<0>(B2_rj)), 
							//B1*C2_rj(1) + C2*B1_rj(1))-(B2*C1_rj(1) + C1*B2_rj(1)),
							-(A1 * thrust::get<0>(C2_rj) + C2 * thrust::get<0>(A1_rj) ) + (A2 * thrust::get<0>(C1_rj) + C1 * thrust::get<0>(A2_rj)) , //CHANGE(9/19): misplaced parenthesis
							// -(A1*C2_rj(1)+C2*A1_rj(1))+(A2*C1_rj(1)+C1*A2_rj(1)), 
							(A1 * thrust::get<0>(B2_rj) + B2 * thrust::get<0>(A1_rj) - (A1 * thrust::get<0>(B1_rj) + B1 * thrust::get<0>(A1_rj) ) ) );
							//(A1*B2_rj(1)+B2*A1_rj(1))-(A1*B1_rj(1)+B1*A1_rj(1));
				
						CVec3 dcN1N2_rj_2 = thrust::make_tuple<double>(
							(B2 * thrust::get<1>(C2_rj) + C2 * thrust::get<1>(B1_rj)) - (B2 * thrust::get<1>(C1_rj) + C1 * thrust::get<1>(B2_rj)),
							//(B1*C2_rj(2) + C2*B1_rj(2))-(B2*C1_rj(2) + C1*B2_rj(2)),
							-(A1 * thrust::get<1>(C2_rj)  + C2 * thrust::get<1>(A1_rj))  + ( A2 * thrust::get<1>(C1_rj) + C1 * thrust::get<1>(A2_rj)) , //CHANGE(9/19): misplaced parenthesis
							// -(A1*C2_rj(2)+C2*A1_rj(2))+(A2*C1_rj(2)+C1*A2_rj(2)), 
							 (A1 * thrust::get<1>(B2_rj) + B2 * thrust::get<1>(A1_rj) - ( A1 * thrust::get<1>(B1_rj) + B1 * thrust::get<1>(A1_rj) ) ) );
							 //(A1*B2_rj(2)+B2*A1_rj(2))-(A1*B1_rj(2)+B1*A1_rj(2));
				
						CVec3 dcN1N2_rj_3 = thrust::make_tuple<double>(
							(B1 * thrust::get<2>(C2_rj) + C2 * thrust::get<2>(B1_rj)) - (B2 * thrust::get<2>(C1_rj) + C1 * thrust::get<2>(B2_rj)),
							// (B1*C2_rj(3) + C2*B1_rj(3))-(B2*C1_rj(3) + C1*B2_rj(3)), 
							-(A1 * thrust::get<2>(C2_rj)  + C2 * thrust::get<2>(A1_rj))  + ( A2 * thrust::get<2>(C1_rj) + C1 * thrust::get<2>(A2_rj)) , //CHANGE(9/19):misplaced parenthesis 
							//-(A1*C2_rj(3)+C2*A1_rj(3))+(A2*C1_rj(3)+C1*A2_rj(3)),
							 (A1 * thrust::get<2>(B2_rj) + B2 * thrust::get<2>(A1_rj) - ( A1 * thrust::get<2>(B1_rj) + B1 * thrust::get<2>(A1_rj) ) ) );
							// (A1*B2_rj(3)+B2*A1_rj(3))-(A1*B1_rj(3)+B1*A1_rj(3))];
				
						Mat_3x3	dcN1N2_rj = thrust::make_tuple<CVec3>(dcN1N2_rj_1, dcN1N2_rj_2, dcN1N2_rj_3);
				
				
				
						CVec3 dcN1N2_rk_1 = thrust::make_tuple<double>(
							(B1 * thrust::get<0>(C2_rk) + C2 * thrust::get<0>(B1_rk)) - (B2 * thrust::get<0>(C1_rk) + C1 * thrust::get<0>(B2_rk)),
							//(B1*C2_rk(1) + C2*B1_rk(1))-(B2*C1_rk(1) + C1*B2_rk(1)), 
							-(A1 * thrust::get<0>(C2_rk) + C2 * thrust::get<0>(A1_rk)) + (A2 * thrust::get<0>(C1_rk) + C1 * thrust::get<0>(A2_rk)) , //CHANGE(9/19): misplaced parenthesis
							//-(A1*C2_rk(1)+C2*A1_rk(1))+(A2*C1_rk(1)+C1*A2_rk(1)), 
							(A1 * thrust::get<0>(B2_rk) + B2 * thrust::get<0>(A1_rk)) - (A1 * thrust::get<0>(B1_rk) + B1 * thrust::get<0>(A1_rk)));
							//(A1*B2_rk(1)+B2*A1_rk(1))-(A1*B1_rk(1)+B1*A1_rk(1));...
						
						CVec3 dcN1N2_rk_2 = thrust::make_tuple<double>(
							(B1 * thrust::get<1>(C2_rk) + C2 * thrust::get<1>(B1_rk)) - (B2 * thrust::get<1>(C1_rk) + C1 * thrust::get<1>(B2_rk)),
							//  (B1*C2_rk(2) + C2*B1_rk(2))-(B2*C1_rk(2) + C1*B2_rk(2)),
							-(A1 * thrust::get<1>(C2_rk) + C2 * thrust::get<1>(A1_rk)) + (A2 * thrust::get<1>(C1_rk) + C1 * thrust::get<1>(A2_rk)) , //CHANGE(9/19): misplaced parenthesis
							//-(A1*C2_rk(2)+C2*A1_rk(2))+(A2*C1_rk(2)+C1*A2_rk(2)),
							(A1 * thrust::get<1>(B2_rk) + B2 * thrust::get<1>(A1_rk)) - (A1 * thrust::get<1>(B1_rk) + B1 * thrust::get<1>(A1_rk)));	
							// (A1*B2_rk(2)+B2*A1_rk(2))-(A1*B1_rk(2)+B1*A1_rk(2));...
						
						CVec3 dcN1N2_rk_3 = thrust::make_tuple<double>(
							(B1 * thrust::get<2>(C2_rk) + C2 * thrust::get<2>(B1_rk)) - (B2 * thrust::get<2>(C1_rk) + C1 * thrust::get<2>(B2_rk)),
							//  (B1*C2_rk(3) + C2*B1_rk(3))-(B2*C1_rk(3) + C1*B2_rk(3)),
							-(A1 * thrust::get<2>(C2_rk) + C2 * thrust::get<2>(A1_rk)) + (A2 * thrust::get<2>(C1_rk) + C1 * thrust::get<2>(A2_rk)) , //CHANGE(9/19): misplaced parenthesis
							//-(A1*C2_rk(3)+C2*A1_rk(3))+(A2*C1_rk(3)+C1*A2_rk(3)),
							(A1 * thrust::get<2>(B2_rk) + B2 * thrust::get<2>(A1_rk)) - (A1 * thrust::get<2>(B1_rk) + B1 * thrust::get<2>(A1_rk)));	
							// (A1*B2_rk(3)+B2*A1_rk(3))-(A1*B1_rk(3)+B1*A1_rk(3))];
				
						Mat_3x3	dcN1N2_rk = thrust::make_tuple<CVec3>(dcN1N2_rk_1, dcN1N2_rk_2, dcN1N2_rk_3);
				
				
				
						CVec3 dcN1N2_ri_1 = thrust::make_tuple<double>(
							(B1 * thrust::get<0>(C2_ri) + C2 * thrust::get<0>(B1_ri)) - (B2 * thrust::get<0>(C1_ri) + C1 * thrust::get<0>(B2_ri)),	
							//[(B1*C2_ri(1) + C2*B1_ri(1))-(B2*C1_ri(1) + C1*B2_ri(1)),
							-(A1 * thrust::get<0>(C2_ri) + C2 * thrust::get<0>(A1_ri)) + (A2 * thrust::get<0>(C1_ri) + C1 * thrust::get<0>(A2_ri)) , //CHANGE(9/19): misplaced parenthesis
							//-(A1*C2_ri(1)+C2*A1_ri(1))+(A2*C1_ri(1)+C1*A2_ri(1)),
							(A1 * thrust::get<0>(B2_ri) + B2 * thrust::get<0>(A1_ri)) - (A1 * thrust::get<0>(B1_ri) + B1 * thrust::get<0>(A1_ri)));
							//(A1*B2_ri(1)+B2*A1_ri(1))-(A1*B1_ri(1)+B1*A1_ri(1));...
							
						CVec3 dcN1N2_ri_2 = thrust::make_tuple<double>(
							(B1 * thrust::get<1>(C2_ri) + C2 * thrust::get<1>(B1_ri)) - (B2 * thrust::get<1>(C1_ri) + C1 * thrust::get<1>(B2_ri)),	
							//(B1*C2_ri(2) + C2*B1_ri(2))-(B2*C1_ri(2) + C1*B2_ri(2)),
							-(A1 * thrust::get<1>(C2_ri) + C2 * thrust::get<1>(A1_ri)) + (A2 * thrust::get<1>(C1_ri) + C1 * thrust::get<1>(A2_ri)), //CHANGE(9/19): misplaced parenthesis
							// -(A1*C2_ri(2)+C2*A1_ri(2))+(A2*C1_ri(2)+C1*A2_ri(2)),
							(A1 * thrust::get<1>(B2_ri) + B2 * thrust::get<1>(A1_ri)) - (A1 * thrust::get<1>(B1_ri) + B1 * thrust::get<1>(A1_ri)));
							//  (A1*B2_ri(2)+B2*A1_ri(2))-(A1*B1_ri(2)+B1*A1_ri(2));...
							
						CVec3 dcN1N2_ri_3 = thrust::make_tuple<double>(
							(B1 * thrust::get<2>(C2_ri) + C2 * thrust::get<2>(B1_ri)) - (B2 * thrust::get<2>(C1_ri) + C1 * thrust::get<2>(B2_ri)),	
							//(B1*C2_ri(3) + C2*B1_ri(3))-(B2*C1_ri(3) + C1*B2_ri(3)),
							-(A1 * thrust::get<2>(C2_ri) + C2 * thrust::get<2>(A1_ri)) + (A2 * thrust::get<2>(C1_ri) + C1 * thrust::get<2>(A2_ri)) , //CHANGE(9/19): misplaced parenthesis
							// -(A1*C2_ri(3)+C2*A1_ri(3))+(A2*C1_ri(3)+C1*A2_ri(3)),
							(A1 * thrust::get<2>(B2_ri) + B2 * thrust::get<2>(A1_ri)) - (A1 * thrust::get<2>(B1_ri) + B1 * thrust::get<2>(A1_ri)));
							//(A1*B2_ri(3)+B2*A1_ri(3))-(A1*B1_ri(3)+B1*A1_ri(3))];
				
						Mat_3x3	dcN1N2_ri = thrust::make_tuple<CVec3>(dcN1N2_ri_1, dcN1N2_ri_2, dcN1N2_ri_3);
				
				
						CVec3 dcN1N2_rl_1 = thrust::make_tuple<double>(
							(B1 * thrust::get<0>(C2_rl) + C2 * thrust::get<0>(B1_rl)) - (B2 * thrust::get<0>(C1_rl) + C1 * thrust::get<0>(B2_rl)),
							//(B1*C2_rl(1) + C2*B1_rl(1))-(B2*C1_rl(1) + C1*B2_rl(1)), 
							-(A1 * thrust::get<0>(C2_rl) + C2 * thrust::get<0>(A1_rl)) + (A2 * thrust::get<0>(C1_rl) + C1 * thrust::get<0>(A2_rl)) , //CHANGE(9/19): misplaced parenthesis
							//-(A1*C2_rl(1)+C2*A1_rl(1))+(A2*C1_rl(1)+C1*A2_rl(1)), 
							(A1 * thrust::get<0>(B2_rl) + B2 * thrust::get<0>(A1_rl)) - (A1 * thrust::get<0>(B1_rl) + B1 * thrust::get<0>(A1_rl)));
							//(A1*B2_rl(1)+B2*A1_rl(1))-(A1*B1_rl(1)+B1*A1_rl(1));...
							
						CVec3 dcN1N2_rl_2 = thrust::make_tuple<double>(
							(B1 * thrust::get<1>(C2_rl) + C2 * thrust::get<1>(B1_rl)) - (B2 * thrust::get<1>(C1_rl) + C1 * thrust::get<1>(B2_rl)),
							//(B1*C2_rl(2) + C2*B1_rl(2))-(B2*C1_rl(2) + C1*B2_rl(2)), 
							-(A1 * thrust::get<1>(C2_rl) + C2 * thrust::get<1>(A1_rl)) + (A2 * thrust::get<1>(C1_rl) + C1 * thrust::get<1>(A2_rl)) , //CHANGE(9/19): misplaced parenthesis
							//-(A1*C2_rl(2)+C2*A1_rl(2))+(A2*C1_rl(2)+C1*A2_rl(2)), 
							(A1 * thrust::get<1>(B2_rl) + B2 * thrust::get<1>(A1_rl)) - (A1 * thrust::get<1>(B1_rl) + B1 * thrust::get<1>(A1_rl)));
							//(A1*B2_rl(2)+B2*A1_rl(2))-(A1*B1_rl(2)+B1*A1_rl(2));...
							
						CVec3 dcN1N2_rl_3 = thrust::make_tuple<double>(
							(B1 * thrust::get<2>(C2_rl) + C2 * thrust::get<2>(B1_rl)) - (B2 * thrust::get<2>(C1_rl) + C1 * thrust::get<2>(B2_rl)),
							//(B1*C2_rl(3) + C2*B1_rl(3))-(B2*C1_rl(3) + C1*B2_rl(3)), 
							-(A1 * thrust::get<2>(C2_rl) + C2 * thrust::get<2>(A1_rl)) + (A2 * thrust::get<2>(C1_rl) + C1 * thrust::get<2>(A2_rl)) , //CHANGE(9/19): misplaced parenthesis		
							//-(A1*C2_rl(3)+C2*A1_rl(3))+(A2*C1_rl(3)+C1*A2_rl(3)), 
							(A1 * thrust::get<2>(B2_rl) + B2 * thrust::get<2>(A1_rl)) - (A1 * thrust::get<2>(B1_rl) + B1 * thrust::get<2>(A1_rl)));			
							//(A1*B2_rl(3)+B2*A1_rl(3))-(A1*B1_rl(3)+B1*A1_rl(3))];
				
						Mat_3x3	dcN1N2_rl = thrust::make_tuple<CVec3>(dcN1N2_rl_1, dcN1N2_rl_2, dcN1N2_rl_3);
				
                std::cout<<"PnN1nN2_rj: "<< thrust::get<0>(PnN1nN2_rj)<<" "<< thrust::get<1>(PnN1nN2_rj)<<" "<< thrust::get<2>(PnN1nN2_rj)<<std::endl;
                std::cout<<"PnN1nN2_rk: "<< thrust::get<0>(PnN1nN2_rk)<<" "<< thrust::get<1>(PnN1nN2_rk)<<" "<< thrust::get<2>(PnN1nN2_rk)<<std::endl;
                std::cout<<"PnN1nN2_ri: "<< thrust::get<0>(PnN1nN2_ri)<<" "<< thrust::get<1>(PnN1nN2_ri)<<" "<< thrust::get<2>(PnN1nN2_ri)<<std::endl;
                std::cout<<"PnN1nN2_rl: "<< thrust::get<0>(PnN1nN2_rl)<<" "<< thrust::get<1>(PnN1nN2_rl)<<" "<< thrust::get<2>(PnN1nN2_rl)<<std::endl;
				
				std::cout<<"dcN1N2_rj_1: "<< thrust::get<0>(dcN1N2_rj_1)<<" "<< thrust::get<1>(dcN1N2_rj_1)<<" "<< thrust::get<2>(dcN1N2_rj_1)<<std::endl;
                std::cout<<"dcN1N2_rj_2: "<< thrust::get<0>(dcN1N2_rj_2)<<" "<< thrust::get<1>(dcN1N2_rj_2)<<" "<< thrust::get<2>(dcN1N2_rj_2)<<std::endl;
                std::cout<<"dcN1N2_rj_3: "<< thrust::get<0>(dcN1N2_rj_3)<<" "<< thrust::get<1>(dcN1N2_rj_3)<<" "<< thrust::get<2>(dcN1N2_rj_3)<<std::endl;
                
	
				std::cout<<"dcN1N2_rk_1: "<< thrust::get<0>(dcN1N2_rk_1)<<" "<< thrust::get<1>(dcN1N2_rk_1)<<" "<< thrust::get<2>(dcN1N2_rk_1)<<std::endl;
                std::cout<<"dcN1N2_rk_2: "<< thrust::get<0>(dcN1N2_rk_2)<<" "<< thrust::get<1>(dcN1N2_rk_2)<<" "<< thrust::get<2>(dcN1N2_rk_2)<<std::endl;
                std::cout<<"dcN1N2_rk_3: "<< thrust::get<0>(dcN1N2_rk_3)<<" "<< thrust::get<1>(dcN1N2_rk_3)<<" "<< thrust::get<2>(dcN1N2_rk_3)<<std::endl;
                
				std::cout<<"dcN1N2_ri_1: "<< thrust::get<0>(dcN1N2_ri_1)<<" "<< thrust::get<1>(dcN1N2_ri_1)<<" "<< thrust::get<2>(dcN1N2_ri_1)<<std::endl;
                std::cout<<"dcN1N2_ri_2: "<< thrust::get<0>(dcN1N2_ri_2)<<" "<< thrust::get<1>(dcN1N2_ri_2)<<" "<< thrust::get<2>(dcN1N2_ri_2)<<std::endl;
                std::cout<<"dcN1N2_ri_3: "<< thrust::get<0>(dcN1N2_ri_3)<<" "<< thrust::get<1>(dcN1N2_ri_3)<<" "<< thrust::get<2>(dcN1N2_ri_3)<<std::endl;
    
				std::cout<<"dcN1N2_rl_1: "<< thrust::get<0>(dcN1N2_rl_1)<<" "<< thrust::get<1>(dcN1N2_rl_1)<<" "<< thrust::get<2>(dcN1N2_rl_1)<<std::endl;
                std::cout<<"dcN1N2_rl_2: "<< thrust::get<0>(dcN1N2_rl_2)<<" "<< thrust::get<1>(dcN1N2_rl_2)<<" "<< thrust::get<2>(dcN1N2_rl_2)<<std::endl;
                std::cout<<"dcN1N2_rl_3: "<< thrust::get<0>(dcN1N2_rl_3)<<" "<< thrust::get<1>(dcN1N2_rl_3)<<" "<< thrust::get<2>(dcN1N2_rl_3)<<std::endl;
				
				
				double N1N2_nN1nN2 = CVec3_dot(N1,N2) / (nN1*nN2*nN1*nN2);
				CVec3 COSE_rj = CVec3_plus(CVec3_scalermult( 1/(nN1*nN2), DN1N2_rj), CVec3_scalermult( -1.0*N1N2_nN1nN2, PnN1nN2_rj));
					//CVec3_scalermult(nN1 * nN2, DN1N2_rj) , CVec3_scalermult( N1N2_nN1nN2 , PnN1nN2_rj ) );
					//CHANGE(9/14): rewritting the computation to match the matlab version.
	
					//COSE_rjx = ((nN1*nN2)*DN1N2_rj(1) - dot(N1,N2)*PnN1nN2_rj(1))/(nN1*nN2)^2; 
					//COSE_rjy = ((nN1*nN2)*DN1N2_rj(2) - dot(N1,N2)*PnN1nN2_rj(2))/(nN1*nN2)^2;
					//COSE_rjz = ((nN1*nN2)*DN1N2_rj(3) - dot(N1,N2)*PnN1nN2_rj(3))/(nN1*nN2)^2;
	
				CVec3 COSE_rk = CVec3_plus(CVec3_scalermult( 1/(nN1*nN2), DN1N2_rk), CVec3_scalermult( -1.0*N1N2_nN1nN2, PnN1nN2_rk));
					//CVec3_scalermult(nN1 * nN2, DN1N2_rk) , CVec3_scalermult( N1N2_nN1nN2 , PnN1nN2_rk ) );
					//CHANGE(9/14): rewritting the computation to match the matlab version.
	
					//COSE_rkx = ((nN1*nN2)*DN1N2_rk(1) - dot(N1,N2)*PnN1nN2_rk(1))/(nN1*nN2)^2;
					//COSE_rky = ((nN1*nN2)*DN1N2_rk(2) - dot(N1,N2)*PnN1nN2_rk(2))/(nN1*nN2)^2;
					//COSE_rkz = ((nN1*nN2)*DN1N2_rk(3) - dot(N1,N2)*PnN1nN2_rk(3))/(nN1*nN2)^2;
					
				CVec3 COSE_ri = CVec3_plus(CVec3_scalermult( 1/(nN1*nN2), DN1N2_ri), CVec3_scalermult( -1.0*N1N2_nN1nN2, PnN1nN2_ri));
					//CVec3_scalermult(nN1 * nN2, DN1N2_ri) , CVec3_scalermult( N1N2_nN1nN2 , PnN1nN2_ri ) );
					//CHANGE(9/14): rewritting the computation to match the matlab version.
					
					//COSE_rix = ((nN1*nN2)*DN1N2_ri(1) - dot(N1,N2)*PnN1nN2_ri(1))/(nN1*nN2)^2;
					//COSE_riy = ((nN1*nN2)*DN1N2_ri(2) - dot(N1,N2)*PnN1nN2_ri(2))/(nN1*nN2)^2;
					//COSE_riz = ((nN1*nN2)*DN1N2_ri(3) - dot(N1,N2)*PnN1nN2_ri(3))/(nN1*nN2)^2;
					
				CVec3 COSE_rl = CVec3_plus(CVec3_scalermult( 1/(nN1*nN2), DN1N2_rl), CVec3_scalermult( -1.0*N1N2_nN1nN2, PnN1nN2_rl));
					//CVec3_scalermult(nN1 * nN2, DN1N2_rl) , CVec3_scalermult( N1N2_nN1nN2 , PnN1nN2_rl ) );
					//CHANGE(9/14): rewritting the computation to match the matlab version
					
					//COSE_rlx = ((nN1*nN2)*DN1N2_rl(1) - dot(N1,N2)*PnN1nN2_rl(1))/(nN1*nN2)^2;
					//COSE_rly = ((nN1*nN2)*DN1N2_rl(2) - dot(N1,N2)*PnN1nN2_rl(2))/(nN1*nN2)^2;
					//COSE_rlz = ((nN1*nN2)*DN1N2_rl(3) - dot(N1,N2)*PnN1nN2_rl(3))/(nN1*nN2)^2;
					
					std::cout<<"N1N2_nN1nN2:" <<N1N2_nN1nN2<<std::endl;
					
					std::cout<<"nN2: " <<nN2<<std::endl; 
					
					std::cout<<"nN1: " <<nN1<<std::endl;
					std::cout<<"abc1: " <<A1<<"" <<B1<< " " <<C1<<std::endl;
					std::cout<<"abc2: " <<A2<<"" <<B2<< " " <<C2<<std::endl;
					
					std::cout<<"COSE_rj: "<< thrust::get<0>(COSE_rj)<<" "<< thrust::get<1>(COSE_rj)<<" "<< thrust::get<2>(COSE_rj)<<std::endl;
					std::cout<<"COSE_rk: "<< thrust::get<0>(COSE_rk)<<" "<< thrust::get<1>(COSE_rk)<<" "<< thrust::get<2>(COSE_rk)<<std::endl;
					std::cout<<"COSE_ri: "<< thrust::get<0>(COSE_ri)<<" "<< thrust::get<1>(COSE_ri)<<" "<< thrust::get<2>(COSE_ri)<<std::endl;
					std::cout<<"COSE_rl: "<< thrust::get<0>(COSE_rl)<<" "<< thrust::get<1>(COSE_rl)<<" "<< thrust::get<2>(COSE_rl)<<std::endl;
			
			double SINE_rjx = 1/(nN1*nN2) * (CVec3_dot( CVec3_cross(N1,N2) ,thrust::get<0>(dUD_rj))
								+ CVec3_dot( thrust::get<0>(dcN1N2_rj), unitDir ))
								- (1/(nN1*nN2*nN1*nN2)) * CVec3_dot( CVec3_cross(N1, N2), unitDir) * thrust::get<0>(PnN1nN2_rj);
								//(nN1*nN2*(dot(cross(N1,N2), dUD_rj(1,:)) + dot(dcN1N2_rj(1,:), UD)) - dot(cross(N1,N2),UD)*PnN1nN2_rj(1))/(nN1*nN2)^2;
			
			double SINE_rjy = 1/(nN1*nN2) * (CVec3_dot( CVec3_cross(N1,N2) ,thrust::get<1>(dUD_rj))
								+ CVec3_dot( thrust::get<1>(dcN1N2_rj), unitDir ))
								- (1/(nN1*nN2*nN1*nN2)) * CVec3_dot( CVec3_cross(N1, N2), unitDir) * thrust::get<1>(PnN1nN2_rj);			
								//SINE_rjy = (nN1*nN2*(dot(cross(N1,N2), dUD_rj(2,:)) + dot(dcN1N2_rj(2,:), UD)) - dot(cross(N1,N2),UD)*PnN1nN2_rj(2))/(nN1*nN2)^2;
			
			double SINE_rjz = 1.0/(nN1*nN2) * (CVec3_dot( CVec3_cross(N1,N2) ,thrust::get<2>(dUD_rj))
								+ CVec3_dot( thrust::get<2>(dcN1N2_rj), unitDir ))
								- (1.0/(nN1*nN2*nN1*nN2)) * CVec3_dot( CVec3_cross(N1, N2), unitDir) * thrust::get<2>(PnN1nN2_rj);
								//1/(nN1*nN2)*(dot(cross(N1,N2), dUD_rj(3,:)) + dot(dcN1N2_rj(3,:), UD))...
    							//- dot(cross(N1,N2),UD)*PnN1nN2_rj(3) / (nN1*nN2)^2;
			double SINE_rkx = 1.0/(nN1*nN2) * (CVec3_dot( CVec3_cross(N1,N2) ,thrust::get<0>(dUD_rk))
								+ CVec3_dot( thrust::get<0>(dcN1N2_rk), unitDir ))
								- (1.0/(nN1*nN2*nN1*nN2)) * CVec3_dot( CVec3_cross(N1, N2), unitDir) * thrust::get<0>(PnN1nN2_rk);
								//(nN1*nN2*(dot(cross(N1,N2), dUD_rk(1,:)) + dot(dcN1N2_rk(1,:), UD)) - dot(cross(N1,N2),UD)*PnN1nN2_rk(1))/(nN1*nN2)^2;

			double SINE_rky = 1.0/(nN1*nN2) * (CVec3_dot( CVec3_cross(N1,N2) ,thrust::get<1>(dUD_rk))
								+ CVec3_dot( thrust::get<1>(dcN1N2_rk), unitDir ))
								- (1/(nN1*nN2*nN1*nN2)) * CVec3_dot( CVec3_cross(N1, N2), unitDir) * thrust::get<1>(PnN1nN2_rk);
								//(nN1*nN2*(dot(cross(N1,N2), dUD_rk(2,:)) + dot(dcN1N2_rk(2,:), UD)) - dot(cross(N1,N2),UD)*PnN1nN2_rk(2))/(nN1*nN2)^2;
							
			double SINE_rkz = 1.0/(nN1*nN2) * (CVec3_dot( CVec3_cross(N1,N2) ,thrust::get<2>(dUD_rk))
								+ CVec3_dot( thrust::get<2>(dcN1N2_rk), unitDir ))
								- (1/(nN1*nN2*nN1*nN2)) * CVec3_dot( CVec3_cross(N1, N2), unitDir) * thrust::get<2>(PnN1nN2_rk);
								//(nN1*nN2*(dot(cross(N1,N2), dUD_rk(3,:)) + dot(dcN1N2_rk(3,:), UD)) - dot(cross(N1,N2),UD)*PnN1nN2_rk(3))/(nN1*nN2)^2;

			double SINE_rix = 1.0/(nN1*nN2) * (CVec3_dot( CVec3_cross(N1,N2) ,thrust::get<0>(dUD_ri))
								+ CVec3_dot( thrust::get<0>(dcN1N2_ri), unitDir ))
								- (1/(nN1*nN2*nN1*nN2)) * CVec3_dot( CVec3_cross(N1, N2), unitDir) * thrust::get<0>(PnN1nN2_ri);
								//(nN1*nN2*(dot(cross(N1,N2), dUD_ri(1,:)) + dot(dcN1N2_ri(1,:), UD)) - dot(cross(N1,N2),UD)*PnN1nN2_ri(1))/(nN1*nN2)^2;

			double SINE_riy = 1.0/(nN1*nN2) * (CVec3_dot( CVec3_cross(N1,N2) ,thrust::get<1>(dUD_ri))
								+ CVec3_dot( thrust::get<1>(dcN1N2_ri), unitDir ))
								- (1/(nN1*nN2*nN1*nN2)) * CVec3_dot( CVec3_cross(N1, N2), unitDir) * thrust::get<1>(PnN1nN2_ri);
								//(nN1*nN2*(dot(cross(N1,N2), dUD_ri(2,:)) + dot(dcN1N2_ri(2,:), UD)) - dot(cross(N1,N2),UD)*PnN1nN2_ri(2))/(nN1*nN2)^2;

			double SINE_riz = 1.0/(nN1*nN2) * (CVec3_dot( CVec3_cross(N1,N2) ,thrust::get<2>(dUD_ri))
								+ CVec3_dot( thrust::get<2>(dcN1N2_ri), unitDir ))
								- (1/(nN1*nN2*nN1*nN2)) * CVec3_dot( CVec3_cross(N1, N2), unitDir) * thrust::get<2>(PnN1nN2_ri);
								//(nN1*nN2*(dot(cross(N1,N2), dUD_ri(3,:)) + dot(dcN1N2_ri(3,:), UD)) - dot(cross(N1,N2),UD)*PnN1nN2_ri(3))/(nN1*nN2)^2;

			double SINE_rlx = 1.0/(nN1*nN2) * (CVec3_dot( CVec3_cross(N1,N2) ,thrust::get<0>(dUD_rl))
								+ CVec3_dot( thrust::get<0>(dcN1N2_rl), unitDir ))
								- (1/(nN1*nN2*nN1*nN2)) * CVec3_dot( CVec3_cross(N1, N2), unitDir) * thrust::get<0>(PnN1nN2_rl);
								//(nN1*nN2*(dot(cross(N1,N2), dUD_rl(1,:)) + dot(dcN1N2_rl(1,:), UD)) - dot(cross(N1,N2),UD)*PnN1nN2_rl(1))/(nN1*nN2)^2;

			double SINE_rly = 1.0/(nN1*nN2) * (CVec3_dot( CVec3_cross(N1,N2) ,thrust::get<1>(dUD_rl))
								+ CVec3_dot( thrust::get<1>(dcN1N2_rl), unitDir ))
								- (1/(nN1*nN2*nN1*nN2)) * CVec3_dot( CVec3_cross(N1, N2), unitDir) * thrust::get<1>(PnN1nN2_rl);
								//(nN1*nN2*(dot(cross(N1,N2), dUD_rl(2,:)) + dot(dcN1N2_rl(2,:), UD)) - dot(cross(N1,N2),UD)*PnN1nN2_rl(2))/(nN1*nN2)^2;
			
			double SINE_rlz = 1.0/(nN1*nN2) * (CVec3_dot( CVec3_cross(N1,N2) ,thrust::get<2>(dUD_rl))
								+ CVec3_dot( thrust::get<2>(dcN1N2_rl), unitDir ))
								- (1/(nN1*nN2*nN1*nN2)) * CVec3_dot( CVec3_cross(N1, N2), unitDir) * thrust::get<2>(PnN1nN2_rl);
								//(nN1*nN2*(dot(cross(N1,N2), dUD_rl(3,:)) + dot(dcN1N2_rl(3,:), UD)) - dot(cross(N1,N2),UD)*PnN1nN2_rl(3))/(nN1*nN2)^2;

			std::cout<<"Sine_rj: "<< (SINE_rjx)<<" "<< (SINE_rjy)<<" "<< (SINE_rjz)<<std::endl;
			std::cout<<"Sine_rk: "<< (SINE_rkx)<<" "<< (SINE_rky)<<" "<< (SINE_rkz)<<std::endl;
			std::cout<<"Sine_ri: "<< (SINE_rix)<<" "<< (SINE_riy)<<" "<< (SINE_riz)<<std::endl;
			std::cout<<"Sine_rl: "<< (SINE_rlx)<<" "<< (SINE_rly)<<" "<< (SINE_rlz)<<std::endl;
			
			double angle_0 = 1.5707963267/2.0;
			double spring_constant = bendingTriangleInfoVecs.spring_constant;
			double place_0_x = spring_constant * (cos(angle_0) * (thrust::get<0>(COSE_ri)) + spring_constant * sin(angle_0) * SINE_rix);
			double place_0_y = spring_constant * (cos(angle_0) * (thrust::get<1>(COSE_ri)) + spring_constant * sin(angle_0) * SINE_riy);
			double place_0_z = spring_constant * (cos(angle_0) * (thrust::get<2>(COSE_ri)) + spring_constant * sin(angle_0) * SINE_riz);

			double place_1_x = spring_constant * (cos(angle_0) * (thrust::get<0>(COSE_rj)) + spring_constant * sin(angle_0) * SINE_rjx);
			double place_1_y = spring_constant * (cos(angle_0) * (thrust::get<1>(COSE_rj)) + spring_constant * sin(angle_0) * SINE_rjy);
			double place_1_z = spring_constant * (cos(angle_0) * (thrust::get<2>(COSE_rj)) + spring_constant * sin(angle_0) * SINE_rjz);

			double place_2_x = spring_constant * (cos(angle_0) * (thrust::get<0>(COSE_rk)) + spring_constant * sin(angle_0) * SINE_rkx);
			double place_2_y = spring_constant * (cos(angle_0) * (thrust::get<1>(COSE_rk)) + spring_constant * sin(angle_0) * SINE_rky);
			double place_2_z = spring_constant * (cos(angle_0) * (thrust::get<2>(COSE_rk)) + spring_constant * sin(angle_0) * SINE_rkz);

			double place_3_x = spring_constant * (cos(angle_0) * (thrust::get<0>(COSE_rl)) + spring_constant * sin(angle_0) * SINE_rlx);
			double place_3_y = spring_constant * (cos(angle_0) * (thrust::get<1>(COSE_rl)) + spring_constant * sin(angle_0) * SINE_rly);
			double place_3_z = spring_constant * (cos(angle_0) * (thrust::get<2>(COSE_rl)) + spring_constant * sin(angle_0) * SINE_rlz);
			std::cout<<"Res i : " <<id_i << " " << (place_0_x)<<" "<< (place_0_y)<<" "<< (place_0_z)<<std::endl;
			std::cout<<"Res j : " <<id_j << " " << (place_1_x)<<" "<< (place_1_y)<<" "<< (place_1_z)<<std::endl;
			std::cout<<"Res k : " <<id_k << " " << (place_2_x)<<" "<< (place_2_y)<<" "<< (place_2_z)<<std::endl;
			std::cout<<"Res l : " <<id_l << " " << (place_3_x)<<" "<< (place_3_y)<<" "<< (place_3_z)<<std::endl;
			
												
        }
    
	}*/
    
    thrust::counting_iterator<unsigned> elemId(0); 

	//bendingTriangleInfoVecs.initial_angle = 1.5707963267/2.0;
	thrust::fill(bendingTriangleInfoVecs.tempNodeForceXReduced.begin(),bendingTriangleInfoVecs.tempNodeForceXReduced.end(),0.0);
	thrust::fill(bendingTriangleInfoVecs.tempNodeForceYReduced.begin(),bendingTriangleInfoVecs.tempNodeForceYReduced.end(),0.0);
	thrust::fill(bendingTriangleInfoVecs.tempNodeForceZReduced.begin(),bendingTriangleInfoVecs.tempNodeForceZReduced.end(),0.0);
	thrust::fill(bendingTriangleInfoVecs.tempNodeForceXUnreduced.begin(),bendingTriangleInfoVecs.tempNodeForceXUnreduced.end(),0.0);
	thrust::fill(bendingTriangleInfoVecs.tempNodeForceYUnreduced.begin(),bendingTriangleInfoVecs.tempNodeForceYUnreduced.end(),0.0);
	thrust::fill(bendingTriangleInfoVecs.tempNodeForceZUnreduced.begin(),bendingTriangleInfoVecs.tempNodeForceZUnreduced.end(),0.0);

    //apply force to temporary vectors.
    bendingTriangleInfoVecs.bending_triangle_energy= 
    thrust::transform_reduce(
        thrust::make_zip_iterator(
            thrust::make_tuple(
				elemId,
                coordInfoVecs.edges2Triangles_1.begin(),
                coordInfoVecs.edges2Triangles_2.begin(),
                coordInfoVecs.edges2Nodes_1.begin(),
                coordInfoVecs.edges2Nodes_2.begin())),
        thrust::make_zip_iterator(
            thrust::make_tuple(
				elemId, 
                coordInfoVecs.edges2Triangles_1.begin(),
                coordInfoVecs.edges2Triangles_2.begin(),
                coordInfoVecs.edges2Nodes_1.begin(),
                coordInfoVecs.edges2Nodes_2.begin())) + coordInfoVecs.num_edges,
        CosBendingFunctor(
            bendingTriangleInfoVecs.spring_constant,
            bendingTriangleInfoVecs.initial_angle,        
            thrust::raw_pointer_cast(coordInfoVecs.nodeLocX.data()),
            thrust::raw_pointer_cast(coordInfoVecs.nodeLocY.data()),
            thrust::raw_pointer_cast(coordInfoVecs.nodeLocZ.data()),
            
            thrust::raw_pointer_cast(bendingTriangleInfoVecs.tempNodeIdUnreduced.data()),
            thrust::raw_pointer_cast(bendingTriangleInfoVecs.tempNodeForceXUnreduced.data()),
            thrust::raw_pointer_cast(bendingTriangleInfoVecs.tempNodeForceYUnreduced.data()),
            thrust::raw_pointer_cast(bendingTriangleInfoVecs.tempNodeForceZUnreduced.data()),
            thrust::raw_pointer_cast(coordInfoVecs.triangles2Nodes_1.data()), 
            thrust::raw_pointer_cast(coordInfoVecs.triangles2Nodes_2.data()), 
            thrust::raw_pointer_cast(coordInfoVecs.triangles2Nodes_3.data())),
		0.0, thrust::plus<double>() );
	
/*	for (unsigned i = 0; i < bendingTriangleInfoVecs.tempNodeIdUnreduced.size(); i++) {

		std::cout<<"id: "<< bendingTriangleInfoVecs.tempNodeIdUnreduced[i]<<std::endl;
		std::cout<< "unreduced F_x: "<< bendingTriangleInfoVecs.tempNodeForceXUnreduced[i]<<std::endl;
		std::cout<< "unreduced F_y: "<< bendingTriangleInfoVecs.tempNodeForceYUnreduced[i]<<std::endl;
		std::cout<< "unreduced F_z: "<< bendingTriangleInfoVecs.tempNodeForceZUnreduced[i]<<std::endl;
	}*/
    //now we have un reduced forces. Sort by id and reduce. 
    //key, then value. Each vector returns sorted		
    thrust::sort_by_key(bendingTriangleInfoVecs.tempNodeIdUnreduced.begin(), bendingTriangleInfoVecs.tempNodeIdUnreduced.end(),
        thrust::make_zip_iterator(
            thrust::make_tuple(
                bendingTriangleInfoVecs.tempNodeForceXUnreduced.begin(),
                bendingTriangleInfoVecs.tempNodeForceYUnreduced.begin(),
                bendingTriangleInfoVecs.tempNodeForceZUnreduced.begin())), thrust::less<unsigned>());
    
    unsigned endKey = thrust::get<0>(
        thrust::reduce_by_key(
            bendingTriangleInfoVecs.tempNodeIdUnreduced.begin(), 
            bendingTriangleInfoVecs.tempNodeIdUnreduced.end(),
        thrust::make_zip_iterator(
            thrust::make_tuple(
                bendingTriangleInfoVecs.tempNodeForceXUnreduced.begin(),
                bendingTriangleInfoVecs.tempNodeForceYUnreduced.begin(),
                bendingTriangleInfoVecs.tempNodeForceZUnreduced.begin())),
			
			bendingTriangleInfoVecs.tempNodeIdReduced.begin(),
        thrust::make_zip_iterator(
            thrust::make_tuple(
                bendingTriangleInfoVecs.tempNodeForceXReduced.begin(),
                bendingTriangleInfoVecs.tempNodeForceYReduced.begin(),
                bendingTriangleInfoVecs.tempNodeForceZReduced.begin())),
		thrust::equal_to<unsigned>(), CVec3Add())) - bendingTriangleInfoVecs.tempNodeIdReduced.begin();//binary_pred, binary_op 
		
    /*	for (unsigned i = 0; i < bendingTriangleInfoVecs.tempNodeIdReduced.size(); i++) {

			std::cout<<"id: "<< bendingTriangleInfoVecs.tempNodeIdReduced[i]<<std::endl;
			std::cout<< "reduced F_x: "<< bendingTriangleInfoVecs.tempNodeForceXReduced[i]<<std::endl;
			std::cout<< "reduced F_y: "<< bendingTriangleInfoVecs.tempNodeForceYReduced[i]<<std::endl;
			std::cout<< "reduced F_z: "<< bendingTriangleInfoVecs.tempNodeForceZReduced[i]<<std::endl;
		}*/
     //apply reduced force to all nodes. 
    thrust::for_each(
        thrust::make_zip_iterator(//1st begin
            thrust::make_tuple(
                bendingTriangleInfoVecs.tempNodeIdReduced.begin(),
                bendingTriangleInfoVecs.tempNodeForceXReduced.begin(),
                bendingTriangleInfoVecs.tempNodeForceYReduced.begin(),
                bendingTriangleInfoVecs.tempNodeForceZReduced.begin())),
        thrust::make_zip_iterator(//1st end
            thrust::make_tuple(
                bendingTriangleInfoVecs.tempNodeIdReduced.begin(),
                bendingTriangleInfoVecs.tempNodeForceXReduced.begin(),
                bendingTriangleInfoVecs.tempNodeForceYReduced.begin(),
                bendingTriangleInfoVecs.tempNodeForceZReduced.begin())) + endKey,
        AddForceFunctor (
            thrust::raw_pointer_cast(coordInfoVecs.nodeForceX.data()),
            thrust::raw_pointer_cast(coordInfoVecs.nodeForceY.data()),
            thrust::raw_pointer_cast(coordInfoVecs.nodeForceZ.data())));

};

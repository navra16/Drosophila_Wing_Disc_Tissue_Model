#include "hip/hip_runtime.h"
#include "System.h"
#include "SystemStructures.h" 
#include "AreaTriangles.h"
#include "BendingTriangles.h"
#include "LinearSprings.h"
#include "LJSprings.h"
#include "NodeAdvance.h"
#include "Storage.h" 
#include "Edgeswap_test.h"

System::System() {};

void System::solveSystem(){

	Edgeswap edgeswap(coordInfoVecs);

	bool runSim = true; 
	while (runSim == true) { 
		
		for (unsigned i = 0; i < generalParams.solve_time; i++) {
			generalParams.iteration = i;
			thrust::fill(coordInfoVecs.nodeForceX.begin(), coordInfoVecs.nodeForceX.end(), 0.0);
			thrust::fill(coordInfoVecs.nodeForceY.begin(), coordInfoVecs.nodeForceY.end(), 0.0);
			thrust::fill(coordInfoVecs.nodeForceZ.begin(), coordInfoVecs.nodeForceZ.end(), 0.0);
	
			ComputeLinearSprings( 
				generalParams, 
				coordInfoVecs,
				linearSpringInfoVecs, 
				ljInfoVecs);
 
			ComputeAreaTriangleSprings(
				generalParams,
				coordInfoVecs,
				areaTriangleInfoVecs);
			
			ComputeCosTriangleSprings(
				generalParams,
				coordInfoVecs,  
				bendingTriangleInfoVecs); 
			
			ComputeLJSprings(
				coordInfoVecs,
				ljInfoVecs,
				generalParams);

			//now forces are computed, move nodes.
			AdvancePositions(
				coordInfoVecs,
				generalParams,
				domainParams);
				

			AdvanceLJParticle(
				generalParams,
				coordInfoVecs,
				ljInfoVecs);
			
		
		} 
		runSim = false;
		//storage->storeVariables();
		std::cout<<"lj points "<< ljInfoVecs.LJ_PosX<< " "<<  ljInfoVecs.LJ_PosY << " "<<  ljInfoVecs.LJ_PosZ << std::endl;
		std::cout<<"lj force "<< ljInfoVecs.forceX<< " "<<  ljInfoVecs.forceY << " "<<  ljInfoVecs.forceZ << std::endl;

		storage->print_VTK_File();
		
		for (unsigned edge = 2; edge < coordInfoVecs.num_edges; edge++  ) {
			edgeswap.edge_swap(
				edge,
				generalParams,
				coordInfoVecs,
				linearSpringInfoVecs,
				bendingTriangleInfoVecs);
		}
		storage->print_VTK_File();
		
		/*for (unsigned i = 0; i < bendingTriangleInfoVecs.tempNodeForceZUnreduced.size(); i++) {
			std::cout<< "1: "<< bendingTriangleInfoVecs.tempNodeForceZUnreduced[i]<<std::endl;
	std::cout<< "2 "<< bendingTriangleInfoVecs.tempNodeForceXUnreduced[i]<<std::endl;
	std::cout<< "2 "<< bendingTriangleInfoVecs.tempNodeForceYUnreduced[i]<<std::endl;
			std::cout<< "2 "<< bendingTriangleInfoVecs.tempNodeForceZUnreduced[i]<<std::endl;
		}*/
				//Test code first. 



		/*std::cout<<" node 11 loc: "<< coordInfoVecs.nodeLocX[11]<< " "<< coordInfoVecs.nodeLocY[11]<< " "<< coordInfoVecs.nodeLocZ[11]<< std::endl;
		std::cout<<" node 11 force: "<< coordInfoVecs.nodeForceX[11]<< " "<< coordInfoVecs.nodeForceY[11]<< " "<< coordInfoVecs.nodeForceZ[11]<< std::endl;
		std::cout<<" node 17 loc: "<< coordInfoVecs.nodeLocX[17]<< " "<< coordInfoVecs.nodeLocY[17]<< " "<< coordInfoVecs.nodeLocZ[17]<< std::endl;
		std::cout<<" node 17 force: "<< coordInfoVecs.nodeForceX[17]<< " "<< coordInfoVecs.nodeForceY[17]<< " "<< coordInfoVecs.nodeForceZ[17]<< std::endl;
		std::cout<<" node 39 loc: "<< coordInfoVecs.nodeLocX[39]<< " "<< coordInfoVecs.nodeLocY[39]<< " "<< coordInfoVecs.nodeLocZ[39]<< std::endl;
		std::cout<<" node 39 force: "<< coordInfoVecs.nodeForceX[39]<< " "<< coordInfoVecs.nodeForceY[39]<< " "<< coordInfoVecs.nodeForceZ[39]<< std::endl;
		std::cout<<" node 40 loc: "<< coordInfoVecs.nodeLocX[40]<< " "<< coordInfoVecs.nodeLocY[40]<< " "<< coordInfoVecs.nodeLocZ[40]<< std::endl;
		std::cout<<" node 40 force: "<< coordInfoVecs.nodeForceX[40]<< " "<< coordInfoVecs.nodeForceY[40]<< " "<< coordInfoVecs.nodeForceZ[40]<< std::endl;
		*/

	}

	
};





void System::assignStorage(std::shared_ptr<Storage> _storage) {
	storage = _storage;
}

//initialize memory for thrust vectors and set coordInfoVecs vals from input. 
void System::initializeSystem(HostSetInfoVecs& hostSetInfoVecs) {
	std::cout<<"Initializing"<<std::endl;

	generalParams.maxNodeCount = hostSetInfoVecs.hostNodeLocX.size();
	coordInfoVecs.num_edges = hostSetInfoVecs.hostEdges2Nodes_1.size();
	coordInfoVecs.num_triangles = hostSetInfoVecs.hostTriangles2Nodes_1.size();

	std::cout<<"num nodes: "<< generalParams.maxNodeCount << std::endl;
	std::cout<<"num edges: "<< coordInfoVecs.num_edges << std::endl;
	std::cout<<"num elems: "<< coordInfoVecs.num_triangles << std::endl;
	//allocate memory
	coordInfoVecs.isNodeFixed.resize(hostSetInfoVecs.hostNodeLocX.size());
	coordInfoVecs.prevNodeLocX.resize(hostSetInfoVecs.hostNodeLocX.size());
	coordInfoVecs.prevNodeLocY.resize(hostSetInfoVecs.hostNodeLocX.size());
	coordInfoVecs.prevNodeLocZ.resize(hostSetInfoVecs.hostNodeLocX.size());

	coordInfoVecs.prevNodeForceX.resize(hostSetInfoVecs.hostNodeLocX.size());
	coordInfoVecs.prevNodeForceY.resize(hostSetInfoVecs.hostNodeLocX.size());
	coordInfoVecs.prevNodeForceZ.resize(hostSetInfoVecs.hostNodeLocX.size());
	
	coordInfoVecs.nodeLocX.resize(hostSetInfoVecs.hostNodeLocX.size());
	coordInfoVecs.nodeLocY.resize(hostSetInfoVecs.hostNodeLocX.size());
	coordInfoVecs.nodeLocZ.resize(hostSetInfoVecs.hostNodeLocX.size());

	coordInfoVecs.nodeForceX.resize(hostSetInfoVecs.hostNodeLocX.size());
	coordInfoVecs.nodeForceY.resize(hostSetInfoVecs.hostNodeLocX.size());
	coordInfoVecs.nodeForceZ.resize(hostSetInfoVecs.hostNodeLocX.size());

	coordInfoVecs.triangles2Nodes_1.resize( coordInfoVecs.num_triangles );
	coordInfoVecs.triangles2Nodes_2.resize( coordInfoVecs.num_triangles );
	coordInfoVecs.triangles2Nodes_3.resize( coordInfoVecs.num_triangles );
	
	coordInfoVecs.triangles2Edges_1.resize( coordInfoVecs.num_triangles );
	coordInfoVecs.triangles2Edges_2.resize( coordInfoVecs.num_triangles );
	coordInfoVecs.triangles2Edges_3.resize( coordInfoVecs.num_triangles );

	coordInfoVecs.edges2Nodes_1.resize( coordInfoVecs.num_edges );
	coordInfoVecs.edges2Nodes_2.resize( coordInfoVecs.num_edges );
	
	coordInfoVecs.edges2Triangles_1.resize( coordInfoVecs.num_edges );
	coordInfoVecs.edges2Triangles_2.resize( coordInfoVecs.num_edges );



	//copy info to GPU
	std::cout<<"Copying"<<std::endl;
	thrust::copy(hostSetInfoVecs.hostIsNodeFixed.begin(),hostSetInfoVecs.hostIsNodeFixed.end(), coordInfoVecs.isNodeFixed.begin());
	thrust::fill(coordInfoVecs.nodeForceX.begin(), coordInfoVecs.nodeForceX.end(), 0.0);
	thrust::fill(coordInfoVecs.nodeForceY.begin(), coordInfoVecs.nodeForceY.end(), 0.0);
	thrust::fill(coordInfoVecs.nodeForceZ.begin(), coordInfoVecs.nodeForceZ.end(), 0.0);

	thrust::fill(coordInfoVecs.prevNodeForceX.begin(), coordInfoVecs.prevNodeForceX.end(), 0.0);
	thrust::fill(coordInfoVecs.prevNodeForceY.begin(), coordInfoVecs.prevNodeForceY.end(), 0.0);
	thrust::fill(coordInfoVecs.prevNodeForceZ.begin(), coordInfoVecs.prevNodeForceZ.end(), 0.0);
	
	thrust::copy(hostSetInfoVecs.hostNodeLocX.begin(), hostSetInfoVecs.hostNodeLocX.end(), coordInfoVecs.prevNodeLocX.begin() );
	thrust::copy(hostSetInfoVecs.hostNodeLocY.begin(), hostSetInfoVecs.hostNodeLocY.end(), coordInfoVecs.prevNodeLocY.begin() );
	thrust::copy(hostSetInfoVecs.hostNodeLocZ.begin(), hostSetInfoVecs.hostNodeLocZ.end(), coordInfoVecs.prevNodeLocZ.begin() );
	
	thrust::copy(hostSetInfoVecs.hostNodeLocX.begin(), hostSetInfoVecs.hostNodeLocX.end(), coordInfoVecs.nodeLocX.begin() );
	thrust::copy(hostSetInfoVecs.hostNodeLocY.begin(), hostSetInfoVecs.hostNodeLocY.end(), coordInfoVecs.nodeLocY.begin() );
	thrust::copy(hostSetInfoVecs.hostNodeLocZ.begin(), hostSetInfoVecs.hostNodeLocZ.end(), coordInfoVecs.nodeLocZ.begin() );
	
	thrust::copy(hostSetInfoVecs.hostTriangles2Nodes_1.begin(), hostSetInfoVecs.hostTriangles2Nodes_1.end(), coordInfoVecs.triangles2Nodes_1.begin() );
	thrust::copy(hostSetInfoVecs.hostTriangles2Nodes_2.begin(), hostSetInfoVecs.hostTriangles2Nodes_2.end(), coordInfoVecs.triangles2Nodes_2.begin() );
	thrust::copy(hostSetInfoVecs.hostTriangles2Nodes_3.begin(), hostSetInfoVecs.hostTriangles2Nodes_3.end(), coordInfoVecs.triangles2Nodes_3.begin() );
	
	thrust::copy(hostSetInfoVecs.hostTriangles2Edges_1.begin(), hostSetInfoVecs.hostTriangles2Edges_1.end(), coordInfoVecs.triangles2Edges_1.begin() );
	thrust::copy(hostSetInfoVecs.hostTriangles2Edges_2.begin(), hostSetInfoVecs.hostTriangles2Edges_2.end(), coordInfoVecs.triangles2Edges_2.begin() );
	thrust::copy(hostSetInfoVecs.hostTriangles2Edges_3.begin(), hostSetInfoVecs.hostTriangles2Edges_3.end(), coordInfoVecs.triangles2Edges_3.begin() );

	thrust::copy(hostSetInfoVecs.hostEdges2Nodes_1.begin(), hostSetInfoVecs.hostEdges2Nodes_1.end(), coordInfoVecs.edges2Nodes_1.begin() );
	thrust::copy(hostSetInfoVecs.hostEdges2Nodes_2.begin(), hostSetInfoVecs.hostEdges2Nodes_2.end(), coordInfoVecs.edges2Nodes_2.begin() );
	
	thrust::copy(hostSetInfoVecs.hostEdges2Triangles_1.begin(), hostSetInfoVecs.hostEdges2Triangles_1.end(), coordInfoVecs.edges2Triangles_1.begin() );
	thrust::copy(hostSetInfoVecs.hostEdges2Triangles_2.begin(), hostSetInfoVecs.hostEdges2Triangles_2.end(), coordInfoVecs.edges2Triangles_2.begin() );

 
	//allocate memory for other data structures.   

	//area triangle info vec
	//number of area springs is the number of triangles
	std::cout<<"Mem"<<std::endl;
	areaTriangleInfoVecs.tempNodeIdUnreduced.resize(areaTriangleInfoVecs.factor * coordInfoVecs.num_triangles);
	areaTriangleInfoVecs.tempNodeForceXUnreduced.resize(areaTriangleInfoVecs.factor * coordInfoVecs.num_triangles);
	areaTriangleInfoVecs.tempNodeForceYUnreduced.resize(areaTriangleInfoVecs.factor * coordInfoVecs.num_triangles);
	areaTriangleInfoVecs.tempNodeForceZUnreduced.resize(areaTriangleInfoVecs.factor * coordInfoVecs.num_triangles);
	
	areaTriangleInfoVecs.tempNodeIdReduced.resize(areaTriangleInfoVecs.factor * coordInfoVecs.num_triangles);
	areaTriangleInfoVecs.tempNodeForceXReduced.resize(areaTriangleInfoVecs.factor * coordInfoVecs.num_triangles);
	areaTriangleInfoVecs.tempNodeForceYReduced.resize(areaTriangleInfoVecs.factor * coordInfoVecs.num_triangles);
	areaTriangleInfoVecs.tempNodeForceZReduced.resize(areaTriangleInfoVecs.factor * coordInfoVecs.num_triangles);

	//beinding triangle info vec
	//num bending springs is the number of times each edge is between two triangles. 
	bendingTriangleInfoVecs.numBendingSprings = coordInfoVecs.edges2Triangles_1.size();

	bendingTriangleInfoVecs.tempNodeIdUnreduced.resize(bendingTriangleInfoVecs.factor * bendingTriangleInfoVecs.numBendingSprings);
	bendingTriangleInfoVecs.tempNodeForceXUnreduced.resize(bendingTriangleInfoVecs.factor * bendingTriangleInfoVecs.numBendingSprings);
	bendingTriangleInfoVecs.tempNodeForceYUnreduced.resize(bendingTriangleInfoVecs.factor * bendingTriangleInfoVecs.numBendingSprings);
	bendingTriangleInfoVecs.tempNodeForceZUnreduced.resize(bendingTriangleInfoVecs.factor * bendingTriangleInfoVecs.numBendingSprings);
	
	bendingTriangleInfoVecs.tempNodeIdReduced.resize(bendingTriangleInfoVecs.factor * bendingTriangleInfoVecs.numBendingSprings);
	bendingTriangleInfoVecs.tempNodeForceXReduced.resize(bendingTriangleInfoVecs.factor * bendingTriangleInfoVecs.numBendingSprings);
	bendingTriangleInfoVecs.tempNodeForceYReduced.resize(bendingTriangleInfoVecs.factor * bendingTriangleInfoVecs.numBendingSprings);
	bendingTriangleInfoVecs.tempNodeForceZReduced.resize(bendingTriangleInfoVecs.factor * bendingTriangleInfoVecs.numBendingSprings);

	//linear springs
	linearSpringInfoVecs.tempNodeIdUnreduced.resize(linearSpringInfoVecs.factor * coordInfoVecs.num_edges);
	linearSpringInfoVecs.tempNodeForceXUnreduced.resize(linearSpringInfoVecs.factor * coordInfoVecs.num_edges);
	linearSpringInfoVecs.tempNodeForceYUnreduced.resize(linearSpringInfoVecs.factor * coordInfoVecs.num_edges);
	linearSpringInfoVecs.tempNodeForceZUnreduced.resize(linearSpringInfoVecs.factor * coordInfoVecs.num_edges);
	
	linearSpringInfoVecs.tempNodeIdReduced.resize(linearSpringInfoVecs.factor * coordInfoVecs.num_edges);
	linearSpringInfoVecs.tempNodeForceXReduced.resize(linearSpringInfoVecs.factor * coordInfoVecs.num_edges);
	linearSpringInfoVecs.tempNodeForceYReduced.resize(linearSpringInfoVecs.factor * coordInfoVecs.num_edges);
	linearSpringInfoVecs.tempNodeForceZReduced.resize(linearSpringInfoVecs.factor * coordInfoVecs.num_edges);
	
	linearSpringInfoVecs.edge_initial_length.resize(coordInfoVecs.num_edges);
	
	thrust::copy(hostSetInfoVecs.hostEdge_initial_length.begin(), hostSetInfoVecs.hostEdge_initial_length.end(), linearSpringInfoVecs.edge_initial_length.begin() );
	std::cout<<"initial lengths: "<< linearSpringInfoVecs.edge_initial_length.size()<<std::endl;

	std::cout<<"System Ready"<<std::endl;

	//Generate LJ particle list. and set LJ particle midpoint.
	double maxX_lj = *(thrust::max_element(coordInfoVecs.nodeLocX.begin(),coordInfoVecs.nodeLocX.end()));
	double minX_lj = *(thrust::min_element(coordInfoVecs.nodeLocX.begin(),coordInfoVecs.nodeLocX.end()));
	double maxY_lj = *(thrust::max_element(coordInfoVecs.nodeLocY.begin(),coordInfoVecs.nodeLocY.end()));
	double minY_lj = *(thrust::min_element(coordInfoVecs.nodeLocY.begin(),coordInfoVecs.nodeLocY.end()));
	
	ljInfoVecs.LJ_PosX = (maxX_lj + minX_lj)/2.0;
	ljInfoVecs.LJ_PosY = (maxY_lj + minY_lj)/2.0;


	//currently unused
	thrust::host_vector<unsigned> tempIds;
	for (unsigned i = 0; i < hostSetInfoVecs.hostNodeLocX.size(); i++ ) {
		double xLoc = hostSetInfoVecs.hostNodeLocX[i];
		double yLoc = hostSetInfoVecs.hostNodeLocY[i];
		double zLoc = hostSetInfoVecs.hostNodeLocZ[i];
		
		double xDist = ljInfoVecs.LJ_PosX - xLoc;
		double yDist = ljInfoVecs.LJ_PosY - yLoc;
		double zDist = ljInfoVecs.LJ_PosZ - zLoc;

		double dist = std::sqrt(xDist*xDist + yDist*yDist + zDist*zDist);
		//just test all poitns for now. Optimize later.
		if (dist < 100 * ljInfoVecs.Rcutoff) {
			tempIds.push_back(i);
		}
	}
	ljInfoVecs.node_id_close.resize( tempIds.size() );
	thrust::copy(tempIds.begin(), tempIds.end(), ljInfoVecs.node_id_close.begin());
	std::cout<<"lj nodes: "<< ljInfoVecs.node_id_close.size() << std::endl;


};



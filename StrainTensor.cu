#include "hip/hip_runtime.h"
/*
      AUTHOR: Navaira Sherwani (Active Shape Programming paper - by Furhman et. al.)
      
      SPONTANEOUS STRAIN TENSOR
      
      lambda = | ?11 ?12 ?13 |
          | ?21 ?22 ?23 |
          | ?31 ?32 ?33 |
          
      lambda = lambda_11 (e1 x e1) + lambda_22 (e2 x e2) + lambda_33 (e3 x e3)   -- e1 and e2 are surface tangents. 
      
      lambda_11 = lambda_iso * lambda_aniso
      lambda_22 = lambda_iso/lambda_aniso
      
      lambda = lambda_iso*lambda_aniso (e1 x e1) + lambda_iso/lambda_aniso (e2 x e2) + (e3 x e3) <- THIS IS OUR LAMBDA FIELD. 
      
      1. So start by taking user values for ?_iso and ?_aniso
      
      2. Create the strain field. 
      
      3. apply the strain field to the vertices of the edge in question so:
                
                
                lambda_a = 0.5 * [ lambda*(X_a) + lambda*(X_b)] (a and b are vertices of the spring a)
                
      4. Calculate the length of the spring and store in L0. 
             
                da = || X_a - X_b || = del_Xa -> spring length
      
      5. Apply the decomposed lambda_a to (not the scalar) spring length.  
                
                da_F = || lambda_a . del_Xa ||
                
                This should be then stored in edge_current_length. 
                
                The reason this is not edge_final_length is because the strain tensor is divided
                into segments so that the model can be relaxed incrementally.                                                                                                 
                
                       

*/
#include "StrainTensor.h"
#include "SystemStructures.h"
#include "System.h"
#include <hip/hip_runtime.h>
#include <cmath>

#define BLOCK_SZ 256


// tuple helpers 

template<int I>  __host__ __device__
inline double c(const CVec3& v) { return thrust::get<I>(v); }

__host__ __device__
inline Mat_3x3 outer(const CVec3& a, const CVec3& b)
{
    return Mat_3x3(
        CVec3( c<0>(a)*c<0>(b), c<0>(a)*c<1>(b), c<0>(a)*c<2>(b) ),
        CVec3( c<1>(a)*c<0>(b), c<1>(a)*c<1>(b), c<1>(a)*c<2>(b) ),
        CVec3( c<2>(a)*c<0>(b), c<2>(a)*c<1>(b), c<2>(a)*c<2>(b) )
    );
}


// scaled add: C+= s * A (element-wise)

__host__ __device__
inline void axpy(double s, const Mat_3x3& A, Mat_3x3& C)
{
    /* row 0 */
    CVec3 r0 = thrust::get<0>(C);
    CVec3 a0 = thrust::get<0>(A);
    thrust::get<0>(r0) += s * c<0>(a0);
    thrust::get<1>(r0) += s * c<1>(a0);
    thrust::get<2>(r0) += s * c<2>(a0);
    thrust::get<0>(C)   = r0;

    /* row 1 */
    CVec3 r1 = thrust::get<1>(C);
    CVec3 a1 = thrust::get<1>(A);
    thrust::get<0>(r1) += s * c<0>(a1);
    thrust::get<1>(r1) += s * c<1>(a1);
    thrust::get<2>(r1) += s * c<2>(a1);
    thrust::get<1>(C)   = r1;

    /* row 2 */
    CVec3 r2 = thrust::get<2>(C);
    CVec3 a2 = thrust::get<2>(A);
    thrust::get<0>(r2) += s * c<0>(a2);
    thrust::get<1>(r2) += s * c<1>(a2);
    thrust::get<2>(r2) += s * c<2>(a2);
    thrust::get<2>(C)   = r2;
}



/* 3×3 · 3×1  ------------------------------------------------- */
__host__ __device__
inline CVec3 matVec(const Mat_3x3& M, const CVec3& v)
{
    return CVec3(
        c<0>( thrust::get<0>(M) )*c<0>(v) + c<1>( thrust::get<0>(M) )*c<1>(v) + c<2>( thrust::get<0>(M) )*c<2>(v),
        c<0>( thrust::get<1>(M) )*c<0>(v) + c<1>( thrust::get<1>(M) )*c<1>(v) + c<2>( thrust::get<1>(M) )*c<2>(v),
        c<0>( thrust::get<2>(M) )*c<0>(v) + c<1>( thrust::get<2>(M) )*c<1>(v) + c<2>( thrust::get<2>(M) )*c<2>(v)
    );
}


/* ?v?  ------------------------------------------------------- */
__host__ __device__
double norm3(const CVec3& v)
{
    return sqrt(thrust::get<0>(v)*thrust::get<0>(v) +
                thrust::get<1>(v)*thrust::get<1>(v) +
                thrust::get<2>(v)*thrust::get<2>(v)) + 1e-14;
}



/*----------------------------------------------------------------------------------
1. Function to build basis vectors
-----------------------------------------------------------------------------------*/

__global__
void k_buildBasis(int N, const double* x, const double *y, const double *z,
                  double c_dx, double c_dy, double c_dz, // projected center of full sphere 
                  double cx, double cy, double cz,// center of layer being computed
                  CVec3* e_h,// double* e_h_y, double* e_h_z, 
                  CVec3* e_R,// double* e_R_y, double* e_R_z, 
                  CVec3* e_phi)//, double* e_phi_y, double* e_phi_z)
{
                  
    int i = blockIdx.x*blockDim.x + threadIdx.x;
                      
    if (i>=N) return;
    
    // position and normal - eh vector
    double px = x[i], py = y[i], pz = z[i]; // c_d is the center of the completed sphere. 
    double nrm = sqrt(px*px + py*py + pz*pz) + 1e-14;
    double hx = px/nrm, hy = py/nrm, hz = pz/nrm;
    e_h[i] = CVec3(hx, hy, hz);  //hx; e_h_y[i] = hy; e_h_z[i] = hz;  
    
    // vector OA - where O is center of disc (not center of sphere) 
    double ox = x[i] - cx, oy = y[i] - cy, oz = z[i] - cz;
    double on = sqrt(ox*ox + oy*oy + oz*oz) +1e-14;
    ox/=on; oy/=on; oz/=on;
    
    // R = OA - (h.OA) h
    double dot_h_OA = hx*ox + hy*oy + hz*oz;
    double rx = ox - dot_h_OA*hx;
    double ry = oy - dot_h_OA*hy;
    double rz = oz - dot_h_OA*hz;
    double rn = sqrt(rx*rx + ry*ry + rz*rz) +1e-14;
    rx/=rn; ry/=rn; rz/=rn;
    
    e_R[i] = CVec3(rx, ry, rz);// rx; e_R_y[i] = ry; e_R_z[i] = rz;
    
    // e_phi = h x R
    double phix = hy*rz - hz*ry; 
    double phiy = hz*rx - hx*rz;
    double phiz = hx*ry - hy*rx;
    double phi_n = sqrt(phix*phix + phiy*phiy + phiz*phiz) +1e-14;
    e_phi[i] = CVec3(phix/phi_n, phiy/phi_n, phiz/phi_n);
                  
                  
}
                  

/* ----------------------------------------------------------------------------
   1.  build lambda at vertices  (one thread per vertex)                           */
__global__
void k_buildLambda(int    N,
                   const double *x,
                   const double *y,
                   const double *z,
                   double cx, double cy, double cz,          // mesh centre (GP)
                   double lam_iso_outDV_center, double lam_iso_outDV_edge, double lam_aniso_outDV_center, double lam_aniso_outDV_edge, double disc_radius, double *rho,   // Strain tensor field parameters from System.h generalParams +disc Radius 
                   double tFrac,
                   double *lam_rr,
                   double *lam_pp,
                   double *lam_ss, 
                   const CVec3* e_R, const CVec3* e_phi, const CVec3* e_h, 
                   Mat_3x3* lam_alpha, const int *layerFlag)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid >= N) return;
    
    //if (layerFlag == 0 || layerflag == 1) return;
    
    /*----  axi-symmetric basis vectors (flat sheet approximation)  ----*/
    double dx = x[tid] - cx;
    double dy = y[tid] - cy;
    double dz = z[tid] - cz;
    
    double r = sqrt(dx*dx + dy*dy) + 1e-14;
     
    rho[tid] = r/disc_radius;
                
    (void)r; // not used here but left for consistency / extensions

    /* linear schedule  ?(t) = I + e t                                          */
    double lamIso = (lam_iso_outDV_center + (lam_iso_outDV_edge - lam_iso_outDV_center)*(rho[tid]*rho[tid]));// * tFrac;   // radial   (_rr)
    double lamAni = (lam_aniso_outDV_center + (lam_aniso_outDV_edge - lam_aniso_outDV_center)*(rho[tid]*rho[tid])); //* tFrac;   // circumf. (_ff)
    
    //bool isBasal = (layerFlag[tid] < 0);
    //if (isBasal) lamAni = 1.0/lamAni;
    
    lam_rr[tid] = (lamIso*lamAni);
    lam_pp[tid] = (lamIso/lamAni);
    lam_ss[tid] = 1.0;                     // no through-thickness growth
    
    
    // Tensor at v_i
    
    Mat_3x3 L = Mat_3x3{
        CVec3(0,0,0), CVec3(0,0,0), CVec3(0,0,0),
    };
    
    axpy(lam_rr[tid], outer(e_R [tid], e_R [tid]), L);
    axpy(lam_pp[tid], outer(e_phi[tid], e_phi[tid]), L);
    axpy(lam_ss[tid], outer(e_h  [tid], e_h  [tid]), L);

    lam_alpha[tid] = L;   // store (row-major) 3×3 tensor
    
}



/* ===== Full projection lambda:e_e ============================ */
__global__
void k_edgeRestProj(int    E,
                    const int    *e2n1,  const int *e2n2,
                    const double *x,     const double *y,   const double *z,
                    const Mat_3x3 *lam_alpha,          // NEW  ? one ? per vertex
                    double *L0,                  // ? original, still here
                    double *Lstar,
                    const int *edgeLayerFlags, int targetLayer)
{
    int eid = blockIdx.x * blockDim.x + threadIdx.x;
    if (eid >= E) return;
    
    // skip edges if not in the desired layer
    if(edgeLayerFlags[eid]==2 ) return;

    int a = e2n1[eid]; 
    int b = e2n2[eid];
    
    CVec3 dX = CVec3(x[a] - x[b], y[a] - y[b], z[a] - z[b]);
    
    // store initial stretch in edge_initial_length
    L0[eid] = norm3(dX);
    
    Mat_3x3 La = lam_alpha[a];
    Mat_3x3 Lb = lam_alpha[b];
   Mat_3x3 Lp;
thrust::get<0>(Lp) = CVec3(
    0.5*( c<0>( thrust::get<0>(La) ) + c<0>( thrust::get<0>(Lb) )),
    0.5*( c<1>( thrust::get<0>(La) ) + c<1>( thrust::get<0>(Lb) )),
    0.5*( c<2>( thrust::get<0>(La) ) + c<2>( thrust::get<0>(Lb) )) );

thrust::get<1>(Lp) = CVec3(
    0.5*( c<0>( thrust::get<1>(La) ) + c<0>( thrust::get<1>(Lb) )),
    0.5*( c<1>( thrust::get<1>(La) ) + c<1>( thrust::get<1>(Lb) )),
    0.5*( c<2>( thrust::get<1>(La) ) + c<2>( thrust::get<1>(Lb) )) );

thrust::get<2>(Lp) = CVec3(
    0.5*( c<0>( thrust::get<2>(La) ) + c<0>( thrust::get<2>(Lb) )),
    0.5*( c<1>( thrust::get<2>(La) ) + c<1>( thrust::get<2>(Lb) )),
    0.5*( c<2>( thrust::get<2>(La) ) + c<2>( thrust::get<2>(Lb) )) );

     /* ---- stretched edge vector ----------------------------- */
    CVec3 dX_stretch = matVec(Lp, dX);

    /* ---- new rest length ----------------------------------- */
    Lstar[eid] = norm3(dX_stretch);


/* ----------------------------------------------------------------------------
   ====   PUBLIC WRAPPERS   ================================================== */
namespace StrainTensorGPU {

void buildVertexLambda(GeneralParams& gp,
                       CoordInfoVecs& coord,
                       LambdaField&         field,
                       double               tFrac)
{
    if (field.lam_rr.size() != coord.nodeLocX.size())
        field.resize(coord.nodeLocX.size());
    if (gp.rho.size() != coord.nodeLocX.size())
        gp.rho.resize(coord.nodeLocX.size());

    int N = static_cast<int>(coord.nodeLocX.size());
    dim3 grid((N + BLOCK_SZ - 1) / BLOCK_SZ);

    
    // build basis vectors
    k_buildBasis<<<grid,BLOCK_SZ>>>(
        N,
        thrust::raw_pointer_cast(coord.nodeLocX.data()), thrust::raw_pointer_cast(coord.nodeLocY.data()), thrust::raw_pointer_cast(coord.nodeLocZ.data()),
        gp.c_dx, gp.c_dy, gp.c_dz,
        gp.centerX, gp.centerY, gp.centerZ,
        thrust::raw_pointer_cast(field.e_h.data()), thrust::raw_pointer_cast(field.e_R.data()), thrust::raw_pointer_cast(field.e_phi.data()));
        
        hipDeviceSynchronize();
    
    // build lambda field 
    k_buildLambda<<<grid,BLOCK_SZ>>>(
        N,
        thrust::raw_pointer_cast(coord.nodeLocX.data()), thrust::raw_pointer_cast(coord.nodeLocY.data()), thrust::raw_pointer_cast(coord.nodeLocZ.data()),
        gp.centerX, gp.centerY, gp.centerZ,
        gp.lambda_iso_center_outDV, gp.lambda_iso_edge_outDV,
        gp.lambda_aniso_center_outDV, gp.lambda_aniso_edge_outDV,
        gp.disc_radius,
        thrust::raw_pointer_cast(gp.rho.data()),
        tFrac,
        thrust::raw_pointer_cast(field.lam_rr.data()), thrust::raw_pointer_cast(field.lam_pp.data()), thrust::raw_pointer_cast(field.lam_ss.data()),
        thrust::raw_pointer_cast(field.e_R.data()), thrust::raw_pointer_cast(field.e_phi.data()), thrust::raw_pointer_cast(field.e_h.data()),
        thrust::raw_pointer_cast(field.lam_alpha.data()),
        thrust::raw_pointer_cast(gp.edges_in_upperhem.data()));
    hipDeviceSynchronize();
}

/* ------------------------------------------------------------------------- */
void updateEdgeRestLengths(CoordInfoVecs&  coord,
                           GeneralParams& gp,
                           const LambdaField&    field,
                           LinearSpringInfoVecs& lsInfo, int targetLayer)
{
    int E = static_cast<int>(coord.num_edges);
    dim3 grid((E + BLOCK_SZ - 1) / BLOCK_SZ); 


  /* full lambda:e_e projection */
    k_edgeRestProj<<<grid,BLOCK_SZ>>>(
        E,
        thrust::raw_pointer_cast(coord.edges2Nodes_1.data()), thrust::raw_pointer_cast(coord.edges2Nodes_2.data()),
        thrust::raw_pointer_cast(coord.nodeLocX.data()),      thrust::raw_pointer_cast(coord.nodeLocY.data()),      thrust::raw_pointer_cast(coord.nodeLocZ.data()),
        thrust::raw_pointer_cast(field.lam_alpha.data()),        // NEW  whole tensor array
        thrust::raw_pointer_cast(lsInfo.edge_initial_length.data()),
        thrust::raw_pointer_cast(lsInfo.edge_final_length.data()),
        thrust::raw_pointer_cast(gp.edges_in_upperhem.data()),
        targetLayer );

    hipDeviceSynchronize();
}
